#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 *  phi_force.c
 *
 *  Computes the force on the fluid from the thermodynamic sector
 *  via the divergence of the chemical stress. Its calculation as
 *  a divergence ensures momentum is conserved.
 *
 *  Note that the stress may be asymmetric.
 *
 *  $Id: phi_force.c 1728 2012-07-18 08:41:51Z agray3 $
 *
 *  Edinburgh Soft Matter and Statistical Physics Group and
 *  Edinburgh Parallel Computing Centre
 *
 *  Kevin Stratford (kevin@epcc.ed.ac.uk)
 *  (c) 2011 The University of Edinburgh
 *
 *****************************************************************************/

#include <assert.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h> 

#define INCLUDING_FROM_GPU_SOURCE
#include "phi_force_gpu.h"

#include "pe.h"
//#include "coords.h"
#include "lattice.h"
#include "phi.h"
#include "site_map.h"
#include "leesedwards.h"
#include "free_energy.h"
#include "wall.h"
#include "phi_force_stress.h"

// FROM util.c
#include "util.h"
//static const double r3_ = (1.0/3.0);


__constant__ double electric_cd[3];
__constant__ int nop_cd;
__constant__ int N_cd[3];
__constant__ int Nall_cd[3];
__constant__ int nhalo_cd;
__constant__ int nsites_cd;
__constant__ double redshift_cd;
__constant__ double rredshift_cd;
__constant__ double q0_cd;
__constant__ double a0_cd;
__constant__ double kappa0_cd;
__constant__ double kappa1_cd;
__constant__ double xi_cd;
__constant__ double zeta_cd;
__constant__ double gamma_cd;
__constant__ double epsilon_cd;
__constant__ double r3_cd;
__constant__ double d_cd[3][3];
__constant__ double e_cd[3][3][3];


extern "C" void checkCUDAError(const char *msg);

/*****************************************************************************
 *
 *  phi_force_calculation
 *
 *  Driver routine to compute the body force on fluid from phi sector.
 *
 *****************************************************************************/

void phi_force_calculation_gpu(void) {

  int N[3],nhalo,Nall[3];
  
  nhalo = coords_nhalo();
  coords_nlocal(N); 


  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;
  
  int nsites=Nall[X]*Nall[Y]*Nall[Z];
 

  

  // FROM blue_phase.c
  double q0_;        /* Pitch = 2pi / q0_ */
  double a0_;        /* Bulk free energy parameter A_0 */
  double gamma_;     /* Controls magnitude of order */
  double kappa0_;    /* Elastic constant \kappa_0 */
  double kappa1_;    /* Elastic constant \kappa_1 */
  
  double xi_;        /* effective molecular aspect ratio (<= 1.0) */
  double redshift_;  /* redshift parameter */
  double rredshift_; /* reciprocal */
  double zeta_;      /* Apolar activity parameter \zeta */
  
  double epsilon_; /* Dielectric anisotropy (e/12pi) */
  
  double electric_[3]; /* Electric field */
  


  redshift_ = blue_phase_redshift(); 
  rredshift_ = blue_phase_rredshift(); 
  q0_=blue_phase_q0();
  a0_=blue_phase_a0();
  kappa0_=blue_phase_kappa0();
  kappa1_=blue_phase_kappa1();
  xi_=blue_phase_get_xi();
  zeta_=blue_phase_get_zeta();
  gamma_=blue_phase_gamma();
  blue_phase_get_electric_field(electric_);
  epsilon_=blue_phase_get_dielectric_anisotropy();

  //hipMemcpy(electric_d, electric_, 3*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpyToSymbol(HIP_SYMBOL(electric_cd), electric_, 3*sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(N_cd), N, 3*sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(Nall_cd), Nall, 3*sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(nhalo_cd), &nhalo, sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(nsites_cd), &nsites, sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(redshift_cd), &redshift_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(rredshift_cd), &rredshift_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(q0_cd), &q0_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(a0_cd), &a0_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(kappa0_cd), &kappa0_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(kappa1_cd), &kappa1_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(xi_cd), &xi_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(zeta_cd), &zeta_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(gamma_cd), &gamma_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(epsilon_cd), &epsilon_, sizeof(double), 0, hipMemcpyHostToDevice);
 hipMemcpyToSymbol(HIP_SYMBOL(r3_cd), &r3_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(d_cd), d_, 3*3*sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(e_cd), e_, 3*3*3*sizeof(double), 0, hipMemcpyHostToDevice); 

 
  checkCUDAError("phi_force hipMemcpyToSymbol");

  //if (force_required_ == 0) return;

  //if (le_get_nplane_total() > 0 || wall_present()) {
    /* Must use the flux method for LE planes */
    /* Also convenient for plane walls */
    //phi_force_flux();
  //}
  //else {
  //if (force_divergence_) {

      int nblocks=(N[X]*N[Y]*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;

      phi_force_calculation_fluid_gpu_d<<<nblocks,DEFAULT_TPB>>>
	(le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d,force_d);
      
      hipDeviceSynchronize();
      checkCUDAError("phi_force_calculation_fluid_gpu_d");

      //}
      //else {
     //hi_force_fluid_phi_gradmu();
      //}
      //}

  return;
}



/*****************************************************************************
 *
 *  blue_phase_compute_h
 *
 *  Compute the molcular field h from q, the q gradient tensor dq, and
 *  the del^2 q tensor.
 *
 *****************************************************************************/


__device__ void blue_phase_compute_h_gpu_d(double q[3][3], double dq[3][3][3],
			      double dsq[3][3], double h[3][3]) {
  int ia, ib, ic, id;

  double q0;              /* Redshifted value */
  double kappa0, kappa1;  /* Redshifted values */
  double q2;
  double e2;
  double eq;
  double sum;

  q0 = q0_cd*rredshift_cd;
  kappa0 = kappa0_cd*redshift_cd*redshift_cd;
  kappa1 = kappa1_cd*redshift_cd*redshift_cd;

  /* From the bulk terms in the free energy... */

  q2 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      q2 += q[ia][ib]*q[ia][ib];
    }
  }

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sum = 0.0;
      for (ic = 0; ic < 3; ic++) {
	sum += q[ia][ic]*q[ib][ic];
      }
      h[ia][ib] = -a0_cd*(1.0 - r3_cd*gamma_cd)*q[ia][ib]
	+ a0_cd*gamma_cd*(sum - r3_cd*q2*d_cd[ia][ib]) - a0_cd*gamma_cd*q2*q[ia][ib];
    }
  }

  /* From the gradient terms ... */
  /* First, the sum e_abc d_b Q_ca. With two permutations, we
   * may rewrite this as e_bca d_b Q_ca */

  eq = 0.0;
  for (ib = 0; ib < 3; ib++) {
    for (ic = 0; ic < 3; ic++) {
      for (ia = 0; ia < 3; ia++) {
	eq += e_cd[ib][ic][ia]*dq[ib][ic][ia];
      }
    }
  }

  /* d_c Q_db written as d_c Q_bd etc */
  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sum = 0.0;
      for (ic = 0; ic < 3; ic++) {
	for (id = 0; id < 3; id++) {
	  sum +=
	    (e_cd[ia][ic][id]*dq[ic][ib][id] + e_cd[ib][ic][id]*dq[ic][ia][id]);
	}
      }
      h[ia][ib] += kappa0*dsq[ia][ib]
	- 2.0*kappa1*q0*sum + 4.0*r3_cd*kappa1*q0*eq*d_cd[ia][ib]
	- 4.0*kappa1*q0*q0*q[ia][ib];
    }
  }

  /* Electric field term */

  e2 = 0.0;
  for (ia = 0; ia < 3; ia++) {
    e2 += electric_cd[ia]*electric_cd[ia];
  }

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      h[ia][ib] +=  epsilon_cd*(electric_cd[ia]*electric_cd[ib] - r3_cd*d_cd[ia][ib]*e2);
    }
  }

  return;
}



/*****************************************************************************
 *
 *  blue_phase_compute_fed
 *
 *  Compute the free energy density as a function of q and the q gradient
 *  tensor dq.
 *
 *****************************************************************************/

__device__ double blue_phase_compute_fed_gpu_d(double q[3][3], double dq[3][3][3]){

  int ia, ib, ic, id;
  double q0;
  double kappa0, kappa1;
  double q2, q3;
  double dq0, dq1;
  double sum;
  double efield;
 
  q0 = q0_cd*rredshift_cd;
  kappa0 = kappa0_cd*redshift_cd*redshift_cd;
  kappa1 = kappa1_cd*redshift_cd*redshift_cd;

  q2 = 0.0;

  /* Q_ab^2 */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      q2 += q[ia][ib]*q[ia][ib];
    }
  }

  /* Q_ab Q_bc Q_ca */

  q3 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      for (ic = 0; ic < 3; ic++) {
	/* We use here the fact that q[ic][ia] = q[ia][ic] */
	q3 += q[ia][ib]*q[ib][ic]*q[ia][ic];
      }
    }
  }

  /* (d_b Q_ab)^2 */

  dq0 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    sum = 0.0;
    for (ib = 0; ib < 3; ib++) {
      sum += dq[ib][ia][ib];
    }
    dq0 += sum*sum;
  }

  /* (e_acd d_c Q_db + 2q_0 Q_ab)^2 */
  /* With symmetric Q_db write Q_bd */

  dq1 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sum = 0.0;
      for (ic = 0; ic < 3; ic++) {
	for (id = 0; id < 3; id++) {
	  sum += e_cd[ia][ic][id]*dq[ic][ib][id];
	}
      }
      sum += 2.0*q0*q[ia][ib];
      dq1 += sum*sum;
    }
  }

  /* Electric field term (epsilon_ includes the factor 1/12pi) */

  efield = 0.0;
  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      efield += electric_cd[ia]*q[ia][ib]*electric_cd[ib];
    }
  }

  sum = 0.5*a0_cd*(1.0 - r3_cd*gamma_cd)*q2 - r3_cd*a0_cd*gamma_cd*q3 +
    0.25*a0_cd*gamma_cd*q2*q2 + 0.5*kappa0*dq0 + 0.5*kappa1*dq1 - epsilon_cd*efield;;

  return sum;
}


/*****************************************************************************
 *
 *  blue_phase_compute_stress
 *
 *  Compute the stress as a function of the q tensor, the q tensor
 *  gradient and the molecular field.
 *
 *  Note the definition here has a minus sign included to allow
 *  computation of the force as minus the divergence (which often
 *  appears as plus in the liquid crystal literature). This is a
 *  separate operation at the end to avoid confusion.
 *
 *****************************************************************************/

__device__ void blue_phase_compute_stress_gpu_d(double q[3][3], double dq[3][3][3],
				   double h[3][3], double sth[3][3]){
  int ia, ib, ic, id, ie;
  double q0;
  double kappa0;
  double kappa1;
  double qh;
  double p0;

  q0 = q0_cd*rredshift_cd;
  kappa0 = kappa0_cd*redshift_cd*redshift_cd;
  kappa1 = kappa1_cd*redshift_cd*redshift_cd;
  
  /* We have ignored the rho T term at the moment, assumed to be zero
   * (in particular, it has no divergence if rho = const). */

  p0 = 0.0 - blue_phase_compute_fed_gpu_d(q, dq);

  /* The contraction Q_ab H_ab */

  qh = 0.0;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      qh += q[ia][ib]*h[ia][ib];
    }
  }

  /* The term in the isotropic pressure, plus that in qh */
  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sth[ia][ib] = -p0*d_cd[ia][ib] + 2.0*xi_cd*(q[ia][ib] + r3_cd*d_cd[ia][ib])*qh;
    }
  }

  /* Remaining two terms in xi and molecular field */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      for (ic = 0; ic < 3; ic++) {
	sth[ia][ib] +=
	  -xi_cd*h[ia][ic]*(q[ib][ic] + r3_cd*d_cd[ib][ic])
	  -xi_cd*(q[ia][ic] + r3_cd*d_cd[ia][ic])*h[ib][ic];
      }
    }
  }

  /* Dot product term d_a Q_cd . dF/dQ_cd,b */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {

      for (ic = 0; ic < 3; ic++) {
	for (id = 0; id < 3; id++) {
	  sth[ia][ib] +=
	    - kappa0*dq[ia][ib][ic]*dq[id][ic][id]
	    - kappa1*dq[ia][ic][id]*dq[ib][ic][id]
	    + kappa1*dq[ia][ic][id]*dq[ic][ib][id];

	  for (ie = 0; ie < 3; ie++) {
	    sth[ia][ib] +=
	     -2.0*kappa1*q0*dq[ia][ic][id]*e_cd[ib][ic][ie]*q[id][ie];
	  }
	}
      }
    }
  }

  /* The antisymmetric piece q_ac h_cb - h_ac q_cb. We can
   * rewrite it as q_ac h_bc - h_ac q_bc. */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      for (ic = 0; ic < 3; ic++) {
	sth[ia][ib] += q[ia][ic]*h[ib][ic] - h[ia][ic]*q[ib][ic];
      }
    }
  }

  /* This is the minus sign. */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sth[ia][ib] = -sth[ia][ib];
    }
  }

  return;
}

/*****************************************************************************
 *
 *  blue_phase_chemical_stress
 *
 *  Return the stress sth[3][3] at lattice site index.
 *
 *****************************************************************************/

__device__ void blue_phase_chemical_stress_gpu_d(int index, 
						 double *phi_site_d,
						 double *grad_phi_site_d,
						 double *delsq_phi_site_d,
						 double sth[3][3]){


  int ia;

  double q[3][3];
  double h[3][3];
  double dq[3][3][3];
  double dsq[3][3];

  //  if(threadIdx.x==0 && blockIdx.x==0) printf("in BPCS\n");

/*   phi_get_q_tensor(index, q); */
/*   phi_gradients_tensor_gradient(index, dq); */
/*   phi_gradients_tensor_delsq(index, dsq); */


  /* load phi */

  q[X][X] = phi_site_d[nsites_cd*XX+index];
  q[X][Y] = phi_site_d[nsites_cd*XY+index];
  q[X][Z] = phi_site_d[nsites_cd*XZ+index];
  q[Y][X] = q[X][Y];
  q[Y][Y] = phi_site_d[nsites_cd*YY+index];
  q[Y][Z] = phi_site_d[nsites_cd*YZ+index];
  q[Z][X] = q[X][Z];
  q[Z][Y] = q[Y][Z];
  q[Z][Z] = 0.0 - q[X][X] - q[Y][Y];


  /* load grad phi */
  for (ia = 0; ia < 3; ia++) {
    dq[ia][X][X] = grad_phi_site_d[ia*nsites_cd*5 + XX*nsites_cd + index];
    dq[ia][X][Y] = grad_phi_site_d[ia*nsites_cd*5 + XY*nsites_cd + index];
    dq[ia][X][Z] = grad_phi_site_d[ia*nsites_cd*5 + XZ*nsites_cd + index];
    dq[ia][Y][X] = dq[ia][X][Y];
    dq[ia][Y][Y] = grad_phi_site_d[ia*nsites_cd*5 + YY*nsites_cd + index];
    dq[ia][Y][Z] = grad_phi_site_d[ia*nsites_cd*5 + YZ*nsites_cd + index];
    dq[ia][Z][X] = dq[ia][X][Z];
    dq[ia][Z][Y] = dq[ia][Y][Z];
    dq[ia][Z][Z] = 0.0 - dq[ia][X][X] - dq[ia][Y][Y];
  }


    /* load delsq phi */
  dsq[X][X] = delsq_phi_site_d[XX*nsites_cd+index];
  dsq[X][Y] = delsq_phi_site_d[XY*nsites_cd+index];
  dsq[X][Z] = delsq_phi_site_d[XZ*nsites_cd+index];
  dsq[Y][X] = dsq[X][Y];
  dsq[Y][Y] = delsq_phi_site_d[YY*nsites_cd+index];
  dsq[Y][Z] = delsq_phi_site_d[YZ*nsites_cd+index];
  dsq[Z][X] = dsq[X][Z];
  dsq[Z][Y] = dsq[Y][Z];
  dsq[Z][Z] = 0.0 - dsq[X][X] - dsq[Y][Y];


  blue_phase_compute_h_gpu_d(q, dq, dsq, h);
  blue_phase_compute_stress_gpu_d(q, dq, h, sth);

  return;
}

/*****************************************************************************
 *
 *  phi_force_calculation_fluid
 *
 *  Compute force from thermodynamic sector via
 *    F_alpha = nalba_beta Pth_alphabeta
 *  using a simple six-point stencil.
 *
 *  Side effect: increments the force at each local lattice site in
 *  preparation for the collision stage.
 *
 *****************************************************************************/


__global__ void phi_force_calculation_fluid_gpu_d(int * le_index_real_to_buffer_d,
						  double *phi_site_d,
						  double *grad_phi_site_d,
						  double *delsq_phi_site_d,
						  double *force_d
					    ) {

  int ia, ic, jc, kc, icm1, icp1;
  int index, index1;
  double pth0[3][3];
  double pth1[3][3];
  double force[3];

  //void (* chemical_stress)(const int index, double s[3][3]);

  //coords_nlocal(nlocal);

  //phi_force_stress_allocate();
  //phi_force_stress_compute();

  //chemical_stress = phi_force_stress;


/*   for (ic = 1; ic <= nlocal[X]; ic++) { */
/*     icm1 = le_index_real_to_buffer(ic, -1); */
/*     icp1 = le_index_real_to_buffer(ic, +1); */
/*     for (jc = 1; jc <= nlocal[Y]; jc++) { */
/*       for (kc = 1; kc <= nlocal[Z]; kc++) { */


  int threadIndex,ii, jj, kk;

 /* CUDA thread index */
 threadIndex = blockIdx.x*blockDim.x+threadIdx.x;
 
 /* Avoid going beyond problem domain */
 if (threadIndex < N_cd[X]*N_cd[Y]*N_cd[Z])
    {


      /* calculate index from CUDA thread index */

      get_coords_from_index_gpu_d(&ii,&jj,&kk,threadIndex,N_cd);
      index = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd,kk+nhalo_cd,Nall_cd);      
      

      /* icm1 = le_index_real_to_buffer(ic, -1); */
      /* icp1 = le_index_real_to_buffer(ic, +1); */
      /*le_index_real_to_buffer_d holds -1 then +1 translation values */
      icm1=le_index_real_to_buffer_d[ii+nhalo_cd];
      icp1=le_index_real_to_buffer_d[Nall_cd[X]+ii+nhalo_cd];      
      


	//printf("inphiforce\n");

	//index = coords_index(ic, jc, kc);


	/* Compute pth at current point */
      blue_phase_chemical_stress_gpu_d(index,phi_site_d,grad_phi_site_d,delsq_phi_site_d,pth0);

	/* Compute differences */
	
	//index1 = coords_index(icp1, jc, kc);
	index1 = get_linear_index_gpu_d(icp1,jj+nhalo_cd,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,phi_site_d,grad_phi_site_d,delsq_phi_site_d,pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] = -0.5*(pth1[ia][X] + pth0[ia][X]);
	}

	//index1 = coords_index(icm1, jc, kc);
	index1 = get_linear_index_gpu_d(icm1,jj+nhalo_cd,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,phi_site_d,grad_phi_site_d,delsq_phi_site_d,pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] += 0.5*(pth1[ia][X] + pth0[ia][X]);
	}

	
	//index1 = coords_index(ic, jc+1, kc);
	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd+1,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] -= 0.5*(pth1[ia][Y] + pth0[ia][Y]);
	}
	//index1 = coords_index(ic, jc-1, kc);
	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd-1,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] += 0.5*(pth1[ia][Y] + pth0[ia][Y]);
	}
	
	//index1 = coords_index(ic, jc, kc+1);
	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd,kk+nhalo_cd+1,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] -= 0.5*(pth1[ia][Z] + pth0[ia][Z]);
	}
	//index1 = coords_index(ic, jc, kc-1);
	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd,kk+nhalo_cd-1,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] += 0.5*(pth1[ia][Z] + pth0[ia][Z]);
	}

	/* Store the force on lattice */

/* 	hydrodynamics_add_force_local(index, force); */
	for (ia=0;ia<3;ia++)
	  force_d[index*3+ia]+=force[ia];


/* 	/\* Next site *\/ */
/*       } */
/*     } */
/*   } */


    }

  //phi_force_stress_free();

  return;
}


/* get linear index from 3d coordinates */
 __device__ static int get_linear_index_gpu_d(int ii,int jj,int kk,int N_d[3])
{
  
  int yfac = N_d[Z];
  int xfac = N_d[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}

/* get 3d coordinates from the index on the accelerator */
__device__ static void get_coords_from_index_gpu_d(int *ii,int *jj,int *kk,int index,int N_d[3])

{
  
  int yfac = N_d[Z];
  int xfac = N_d[Y]*yfac;
  
  *ii = index/xfac;
  *jj = ((index-xfac*(*ii))/yfac);
  *kk = (index-(*ii)*xfac-(*jj)*yfac);

  return;

}
