#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 *  phi_force.c
 *
 *  Computes the force on the fluid from the thermodynamic sector
 *  via the divergence of the chemical stress. Its calculation as
 *  a divergence ensures momentum is conserved.
 *
 *  Note that the stress may be asymmetric.
 *
 *  $Id: phi_force.c 1728 2012-07-18 08:41:51Z agray3 $
 *
 *  Edinburgh Soft Matter and Statistical Physics Group and
 *  Edinburgh Parallel Computing Centre
 *
 *  Kevin Stratford (kevin@epcc.ed.ac.uk)
 *  (c) 2011 The University of Edinburgh
 *
 *****************************************************************************/

#include <assert.h>
#include <stdlib.h>
#include <math.h>
#include <stdio.h> 

#define INCLUDING_FROM_GPU_SOURCE
#include "phi_force_gpu.h"

#include "pe.h"
//#include "coords.h"
#include "lattice.h"
#include "phi.h"
#include "site_map.h"
#include "leesedwards.h"
#include "free_energy.h"
#include "wall.h"
#include "phi_force_stress.h"

// FROM util.c
#include "util.h"
//static const double r3_ = (1.0/3.0);


__constant__ double electric_cd[3];
__constant__ int nop_cd;
//__constant__ int N_cd[3];
//__constant__ int Nall_cd[3];
//__constant__ int nhalo_cd;
//__constant__ int nsites_cd;
__constant__ double redshift_cd;
__constant__ double rredshift_cd;
__constant__ double q0shift_cd;
__constant__ double a0_cd;
__constant__ double kappa0shift_cd;
__constant__ double kappa1shift_cd;
__constant__ double xi_cd;
__constant__ double zeta_cd;
__constant__ double gamma_cd;
__constant__ double epsilon_cd;
__constant__ double r3_cd;
__constant__ double d_cd[3][3];
__constant__ double e_cd[3][3][3];


extern "C" void checkCUDAError(const char *msg);

/*****************************************************************************
 *
 *  phi_force_calculation
 *
 *  Driver routine to compute the body force on fluid from phi sector.
 *
 *****************************************************************************/

void phi_force_calculation_gpu(void) {

  int N[3],nhalo,Nall[3];
  
  nhalo = coords_nhalo();
  coords_nlocal(N); 


  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;
  
  int nsites=Nall[X]*Nall[Y]*Nall[Z];
 

  

  // FROM blue_phase.c
  double q0_;        /* Pitch = 2pi / q0_ */
  double a0_;        /* Bulk free energy parameter A_0 */
  double gamma_;     /* Controls magnitude of order */
  double kappa0_;    /* Elastic constant \kappa_0 */
  double kappa1_;    /* Elastic constant \kappa_1 */
  
  double xi_;        /* effective molecular aspect ratio (<= 1.0) */
  double redshift_;  /* redshift parameter */
  double rredshift_; /* reciprocal */
  double zeta_;      /* Apolar activity parameter \zeta */
  
  double epsilon_; /* Dielectric anisotropy (e/12pi) */
  
  double electric_[3]; /* Electric field */
  


  redshift_ = blue_phase_redshift(); 
  rredshift_ = blue_phase_rredshift(); 
  q0_=blue_phase_q0();
  a0_=blue_phase_a0();
  kappa0_=blue_phase_kappa0();
  kappa1_=blue_phase_kappa1();
  xi_=blue_phase_get_xi();
  zeta_=blue_phase_get_zeta();
  gamma_=blue_phase_gamma();
  blue_phase_get_electric_field(electric_);
  epsilon_=blue_phase_get_dielectric_anisotropy();

 q0_ = q0_*rredshift_;
 kappa0_ = kappa0_*redshift_*redshift_;
 kappa1_ = kappa1_*redshift_*redshift_;


  //hipMemcpy(electric_d, electric_, 3*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpyToSymbol(HIP_SYMBOL(electric_cd), electric_, 3*sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(N_cd), N, 3*sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(Nall_cd), Nall, 3*sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(nhalo_cd), &nhalo, sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(nsites_cd), &nsites, sizeof(int), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(redshift_cd), &redshift_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(rredshift_cd), &rredshift_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(q0shift_cd), &q0_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(a0_cd), &a0_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(kappa0shift_cd), &kappa0_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(kappa1shift_cd), &kappa1_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(xi_cd), &xi_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(zeta_cd), &zeta_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(gamma_cd), &gamma_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(epsilon_cd), &epsilon_, sizeof(double), 0, hipMemcpyHostToDevice);
 hipMemcpyToSymbol(HIP_SYMBOL(r3_cd), &r3_, sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(d_cd), d_, 3*3*sizeof(double), 0, hipMemcpyHostToDevice); 
 hipMemcpyToSymbol(HIP_SYMBOL(e_cd), e_, 3*3*3*sizeof(double), 0, hipMemcpyHostToDevice); 

 
  checkCUDAError("phi_force hipMemcpyToSymbol");

  //if (force_required_ == 0) return;

  //if (le_get_nplane_total() > 0 || wall_present()) {
    /* Must use the flux method for LE planes */
    /* Also convenient for plane walls */
    //phi_force_flux();
  //}
  //else {
  //if (force_divergence_) {


  #define TPB 256
      int nblocks=(N[X]*N[Y]*N[Z]+TPB-1)/TPB;

      phi_force_calculation_fluid_gpu_d<<<nblocks,TPB>>>
	(le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d,force_d);
      
      hipDeviceSynchronize();
      checkCUDAError("phi_force_calculation_fluid_gpu_d");

      //}
      //else {
     //hi_force_fluid_phi_gradmu();
      //}
      //}

  return;
}



/*****************************************************************************
 *
 *  blue_phase_compute_h
 *
 *  Compute the molcular field h from q, the q gradient tensor dq, and
 *  the del^2 q tensor.
 *
 *****************************************************************************/


__device__ void blue_phase_compute_h_gpu_d(double q[3][3], double dq[3][3][3],
			      double dsq[3][3], double h[3][3]) {
  int ia, ib, ic, id;

  double q2;
  double e2;
  double eq;
  double sum;


  /* From the bulk terms in the free energy... */

  /* q2 = 0.0; */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      q2 += q[ia][ib]*q[ia][ib];
    }
  }

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sum = 0.0;
      for (ic = 0; ic < 3; ic++) {
  	sum += q[ia][ic]*q[ib][ic];
      }
      h[ia][ib] = -a0_cd*(1.0 - r3_cd*gamma_cd)*q[ia][ib]
  	+ a0_cd*gamma_cd*(sum - r3_cd*q2*d_cd[ia][ib]) - a0_cd*gamma_cd*q2*q[ia][ib];
    }
  }

  /* From the gradient terms ... */
  /* First, the sum e_abc d_b Q_ca. With two permutations, we
   * may rewrite this as e_bca d_b Q_ca */

  eq = 0.0;
  for (ib = 0; ib < 3; ib++) {
    for (ic = 0; ic < 3; ic++) {
      for (ia = 0; ia < 3; ia++) {
  	eq += e_cd[ib][ic][ia]*dq[ib][ic][ia];
      }
    }
  }


  /* d_c Q_db written as d_c Q_bd etc */
  //for (ia = 0; ia < 3; ia++) {
    //for (ib = 0; ib < 3; ib++) {
      /* sum = 0.0; */
      /* for (ic = 0; ic < 3; ic++) { */
      /* 	for (id = 0; id < 3; id++) { */
      /* 	  sum += */
      /* 	    (e_cd[ia][ic][id]*dq[ic][ib][id] + e_cd[ib][ic][id]*dq[ic][ia][id]); */
      /* 	} */
      /* } */
      /* h[ia][ib] +=  kappa0shift_cd*dsq[ia][ib] */
      /* 	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[ia][ib] */
      /* 	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[ia][ib]; */

      // }
  //}

      sum = 0.0;
      sum += (e_cd[0][0][0]*dq[0][0][0] + e_cd[0][0][0]*dq[0][0][0]);
      sum += (e_cd[0][0][1]*dq[0][0][1] + e_cd[0][0][1]*dq[0][0][1]);
      sum += (e_cd[0][0][2]*dq[0][0][2] + e_cd[0][0][2]*dq[0][0][2]);
      sum += (e_cd[0][1][0]*dq[1][0][0] + e_cd[0][1][0]*dq[1][0][0]);
      sum += (e_cd[0][1][1]*dq[1][0][1] + e_cd[0][1][1]*dq[1][0][1]);
      sum += (e_cd[0][1][2]*dq[1][0][2] + e_cd[0][1][2]*dq[1][0][2]);
      sum += (e_cd[0][2][0]*dq[2][0][0] + e_cd[0][2][0]*dq[2][0][0]);
      sum += (e_cd[0][2][1]*dq[2][0][1] + e_cd[0][2][1]*dq[2][0][1]);
      sum += (e_cd[0][2][2]*dq[2][0][2] + e_cd[0][2][2]*dq[2][0][2]);

      h[0][0] +=  kappa0shift_cd*dsq[0][0]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[0][0]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[0][0];

      sum = 0.0;
      sum += (e_cd[0][0][0]*dq[0][1][0] + e_cd[1][0][0]*dq[0][0][0]);
      sum += (e_cd[0][0][1]*dq[0][1][1] + e_cd[1][0][1]*dq[0][0][1]);
      sum += (e_cd[0][0][2]*dq[0][1][2] + e_cd[1][0][2]*dq[0][0][2]);
      sum += (e_cd[0][1][0]*dq[1][1][0] + e_cd[1][1][0]*dq[1][0][0]);
      sum += (e_cd[0][1][1]*dq[1][1][1] + e_cd[1][1][1]*dq[1][0][1]);
      sum += (e_cd[0][1][2]*dq[1][1][2] + e_cd[1][1][2]*dq[1][0][2]);
      sum += (e_cd[0][2][0]*dq[2][1][0] + e_cd[1][2][0]*dq[2][0][0]);
      sum += (e_cd[0][2][1]*dq[2][1][1] + e_cd[1][2][1]*dq[2][0][1]);
      sum += (e_cd[0][2][2]*dq[2][1][2] + e_cd[1][2][2]*dq[2][0][2]);

      h[0][1] +=  kappa0shift_cd*dsq[0][1]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[0][1]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[0][1];

      sum = 0.0;
      sum += (e_cd[0][0][0]*dq[0][2][0] + e_cd[2][0][0]*dq[0][0][0]);
      sum += (e_cd[0][0][1]*dq[0][2][1] + e_cd[2][0][1]*dq[0][0][1]);
      sum += (e_cd[0][0][2]*dq[0][2][2] + e_cd[2][0][2]*dq[0][0][2]);
      sum += (e_cd[0][1][0]*dq[1][2][0] + e_cd[2][1][0]*dq[1][0][0]);
      sum += (e_cd[0][1][1]*dq[1][2][1] + e_cd[2][1][1]*dq[1][0][1]);
      sum += (e_cd[0][1][2]*dq[1][2][2] + e_cd[2][1][2]*dq[1][0][2]);
      sum += (e_cd[0][2][0]*dq[2][2][0] + e_cd[2][2][0]*dq[2][0][0]);
      sum += (e_cd[0][2][1]*dq[2][2][1] + e_cd[2][2][1]*dq[2][0][1]);
      sum += (e_cd[0][2][2]*dq[2][2][2] + e_cd[2][2][2]*dq[2][0][2]);

      h[0][2] +=  kappa0shift_cd*dsq[0][2]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[0][2]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[0][2];

      ////////
      sum = 0.0;
      sum += (e_cd[1][0][0]*dq[0][0][0] + e_cd[0][0][0]*dq[0][1][0]);
      sum += (e_cd[1][0][1]*dq[0][0][1] + e_cd[0][0][1]*dq[0][1][1]);
      sum += (e_cd[1][0][2]*dq[0][0][2] + e_cd[0][0][2]*dq[0][1][2]);
      sum += (e_cd[1][1][0]*dq[1][0][0] + e_cd[0][1][0]*dq[1][1][0]);
      sum += (e_cd[1][1][1]*dq[1][0][1] + e_cd[0][1][1]*dq[1][1][1]);
      sum += (e_cd[1][1][2]*dq[1][0][2] + e_cd[0][1][2]*dq[1][1][2]);
      sum += (e_cd[1][2][0]*dq[2][0][0] + e_cd[0][2][0]*dq[2][1][0]);
      sum += (e_cd[1][2][1]*dq[2][0][1] + e_cd[0][2][1]*dq[2][1][1]);
      sum += (e_cd[1][2][2]*dq[2][0][2] + e_cd[0][2][2]*dq[2][1][2]);

      h[1][0] +=  kappa0shift_cd*dsq[1][0]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[1][0]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[1][0];

      sum = 0.0;
      sum += (e_cd[1][0][0]*dq[0][1][0] + e_cd[1][0][0]*dq[0][1][0]);
      sum += (e_cd[1][0][1]*dq[0][1][1] + e_cd[1][0][1]*dq[0][1][1]);
      sum += (e_cd[1][0][2]*dq[0][1][2] + e_cd[1][0][2]*dq[0][1][2]);
      sum += (e_cd[1][1][0]*dq[1][1][0] + e_cd[1][1][0]*dq[1][1][0]);
      sum += (e_cd[1][1][1]*dq[1][1][1] + e_cd[1][1][1]*dq[1][1][1]);
      sum += (e_cd[1][1][2]*dq[1][1][2] + e_cd[1][1][2]*dq[1][1][2]);
      sum += (e_cd[1][2][0]*dq[2][1][0] + e_cd[1][2][0]*dq[2][1][0]);
      sum += (e_cd[1][2][1]*dq[2][1][1] + e_cd[1][2][1]*dq[2][1][1]);
      sum += (e_cd[1][2][2]*dq[2][1][2] + e_cd[1][2][2]*dq[2][1][2]);

      h[1][1] +=  kappa0shift_cd*dsq[1][1]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[1][1]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[1][1];

      sum = 0.0;
      sum += (e_cd[1][0][0]*dq[0][2][0] + e_cd[2][0][0]*dq[0][1][0]);
      sum += (e_cd[1][0][1]*dq[0][2][1] + e_cd[2][0][1]*dq[0][1][1]);
      sum += (e_cd[1][0][2]*dq[0][2][2] + e_cd[2][0][2]*dq[0][1][2]);
      sum += (e_cd[1][1][0]*dq[1][2][0] + e_cd[2][1][0]*dq[1][1][0]);
      sum += (e_cd[1][1][1]*dq[1][2][1] + e_cd[2][1][1]*dq[1][1][1]);
      sum += (e_cd[1][1][2]*dq[1][2][2] + e_cd[2][1][2]*dq[1][1][2]);
      sum += (e_cd[1][2][0]*dq[2][2][0] + e_cd[2][2][0]*dq[2][1][0]);
      sum += (e_cd[1][2][1]*dq[2][2][1] + e_cd[2][2][1]*dq[2][1][1]);
      sum += (e_cd[1][2][2]*dq[2][2][2] + e_cd[2][2][2]*dq[2][1][2]);

      h[1][2] +=  kappa0shift_cd*dsq[1][2]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[1][2]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[1][2];

      /////
      sum = 0.0;
      sum += (e_cd[2][0][0]*dq[0][0][0] + e_cd[0][0][0]*dq[0][2][0]);
      sum += (e_cd[2][0][1]*dq[0][0][1] + e_cd[0][0][1]*dq[0][2][1]);
      sum += (e_cd[2][0][2]*dq[0][0][2] + e_cd[0][0][2]*dq[0][2][2]);
      sum += (e_cd[2][1][0]*dq[1][0][0] + e_cd[0][1][0]*dq[1][2][0]);
      sum += (e_cd[2][1][1]*dq[1][0][1] + e_cd[0][1][1]*dq[1][2][1]);
      sum += (e_cd[2][1][2]*dq[1][0][2] + e_cd[0][1][2]*dq[1][2][2]);
      sum += (e_cd[2][2][0]*dq[2][0][0] + e_cd[0][2][0]*dq[2][2][0]);
      sum += (e_cd[2][2][1]*dq[2][0][1] + e_cd[0][2][1]*dq[2][2][1]);
      sum += (e_cd[2][2][2]*dq[2][0][2] + e_cd[0][2][2]*dq[2][2][2]);

      h[2][0] +=  kappa0shift_cd*dsq[2][0]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[2][0]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[2][0];

      sum = 0.0;
      sum += (e_cd[2][0][0]*dq[0][1][0] + e_cd[1][0][0]*dq[0][2][0]);
      sum += (e_cd[2][0][1]*dq[0][1][1] + e_cd[1][0][1]*dq[0][2][1]);
      sum += (e_cd[2][0][2]*dq[0][1][2] + e_cd[1][0][2]*dq[0][2][2]);
      sum += (e_cd[2][1][0]*dq[1][1][0] + e_cd[1][1][0]*dq[1][2][0]);
      sum += (e_cd[2][1][1]*dq[1][1][1] + e_cd[1][1][1]*dq[1][2][1]);
      sum += (e_cd[2][1][2]*dq[1][1][2] + e_cd[1][1][2]*dq[1][2][2]);
      sum += (e_cd[2][2][0]*dq[2][1][0] + e_cd[1][2][0]*dq[2][2][0]);
      sum += (e_cd[2][2][1]*dq[2][1][1] + e_cd[1][2][1]*dq[2][2][1]);
      sum += (e_cd[2][2][2]*dq[2][1][2] + e_cd[1][2][2]*dq[2][2][2]);

      h[2][1] +=  kappa0shift_cd*dsq[2][1]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[2][1]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[2][1];

      sum = 0.0;
      sum += (e_cd[2][0][0]*dq[0][2][0] + e_cd[2][0][0]*dq[0][2][0]);
      sum += (e_cd[2][0][1]*dq[0][2][1] + e_cd[2][0][1]*dq[0][2][1]);
      sum += (e_cd[2][0][2]*dq[0][2][2] + e_cd[2][0][2]*dq[0][2][2]);
      sum += (e_cd[2][1][0]*dq[1][2][0] + e_cd[2][1][0]*dq[1][2][0]);
      sum += (e_cd[2][1][1]*dq[1][2][1] + e_cd[2][1][1]*dq[1][2][1]);
      sum += (e_cd[2][1][2]*dq[1][2][2] + e_cd[2][1][2]*dq[1][2][2]);
      sum += (e_cd[2][2][0]*dq[2][2][0] + e_cd[2][2][0]*dq[2][2][0]);
      sum += (e_cd[2][2][1]*dq[2][2][1] + e_cd[2][2][1]*dq[2][2][1]);
      sum += (e_cd[2][2][2]*dq[2][2][2] + e_cd[2][2][2]*dq[2][2][2]);

      h[2][2] +=  kappa0shift_cd*dsq[2][2]
      	- 2.0*kappa1shift_cd*q0shift_cd*sum + 4.0*r3_cd*kappa1shift_cd*q0shift_cd*eq*d_cd[2][2]
      	- 4.0*kappa1shift_cd*q0shift_cd*q0shift_cd*q[2][2];






  /* Electric field term */

  e2 = 0.0;
  for (ia = 0; ia < 3; ia++) {
    e2 += electric_cd[ia]*electric_cd[ia];
  }

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      h[ia][ib] +=  epsilon_cd*(electric_cd[ia]*electric_cd[ib] - r3_cd*d_cd[ia][ib]*e2);
    }
  }

  return;
}



/*****************************************************************************
 *
 *  blue_phase_compute_fed
 *
 *  Compute the free energy density as a function of q and the q gradient
 *  tensor dq.
 *
 *****************************************************************************/

__device__ double blue_phase_compute_fed_gpu_d(double q[3][3], double dq[3][3][3]){

  int ia, ib, ic, id;
  double q2, q3;
  double dq0, dq1;
  double sum;
  double efield;
 
  q2 = 0.0;

  /* Q_ab^2 */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      q2 += q[ia][ib]*q[ia][ib];
    }
  }

  /* Q_ab Q_bc Q_ca */

  q3 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      for (ic = 0; ic < 3; ic++) {
	/* We use here the fact that q[ic][ia] = q[ia][ic] */
	q3 += q[ia][ib]*q[ib][ic]*q[ia][ic];
      }
    }
  }

  /* (d_b Q_ab)^2 */

  dq0 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    sum = 0.0;
    for (ib = 0; ib < 3; ib++) {
      sum += dq[ib][ia][ib];
    }
    dq0 += sum*sum;
  }

  /* (e_acd d_c Q_db + 2q_0 Q_ab)^2 */
  /* With symmetric Q_db write Q_bd */

  dq1 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sum = 0.0;
      for (ic = 0; ic < 3; ic++) {
	for (id = 0; id < 3; id++) {
	  sum += e_cd[ia][ic][id]*dq[ic][ib][id];
	}
      }
      sum += 2.0*q0shift_cd*q[ia][ib];
      dq1 += sum*sum;
    }
  }

  /* Electric field term (epsilon_ includes the factor 1/12pi) */

  efield = 0.0;
  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      efield += electric_cd[ia]*q[ia][ib]*electric_cd[ib];
    }
  }

  sum = 0.5*a0_cd*(1.0 - r3_cd*gamma_cd)*q2 - r3_cd*a0_cd*gamma_cd*q3 +
    0.25*a0_cd*gamma_cd*q2*q2 + 0.5*kappa0shift_cd*dq0 + 0.5*kappa1shift_cd*dq1 - epsilon_cd*efield;;

  return sum;
}


/*****************************************************************************
 *
 *  blue_phase_compute_stress
 *
 *  Compute the stress as a function of the q tensor, the q tensor
 *  gradient and the molecular field.
 *
 *  Note the definition here has a minus sign included to allow
 *  computation of the force as minus the divergence (which often
 *  appears as plus in the liquid crystal literature). This is a
 *  separate operation at the end to avoid confusion.
 *
 *****************************************************************************/

__device__ void blue_phase_compute_stress_gpu_d(double q[3][3], double dq[3][3][3],
				   double h[3][3], double sth[3][3]){
  int ia, ib, ic, id, ie;

  double tmpdbl,tmpdbl2;
  
  /* We have ignored the rho T term at the moment, assumed to be zero
   * (in particular, it has no divergence if rho = const). */

  tmpdbl = 0.0 - blue_phase_compute_fed_gpu_d(q, dq);

  /* The contraction Q_ab H_ab */

  tmpdbl2 = 0.0;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      tmpdbl2 += q[ia][ib]*h[ia][ib];
    }
  }

  /* The term in the isotropic pressure, plus that in qh */
  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      sth[ia][ib] = -tmpdbl*d_cd[ia][ib] + 2.0*xi_cd*(q[ia][ib] + r3_cd*d_cd[ia][ib])*tmpdbl2;
    }
  }

  /* Remaining two terms in xi and molecular field */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      tmpdbl=0.;
      for (ic = 0; ic < 3; ic++) {
	tmpdbl+=
  	  -xi_cd*h[ia][ic]*(q[ib][ic] + r3_cd*d_cd[ib][ic])
  	  -xi_cd*(q[ia][ic] + r3_cd*d_cd[ia][ic])*h[ib][ic];
      }
      sth[ia][ib] += tmpdbl;
    }
  }

  /* Dot product term d_a Q_cd . dF/dQ_cd,b */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      tmpdbl=0.;
      for (ic = 0; ic < 3; ic++) {
	for (id = 0; id < 3; id++) {
	  tmpdbl +=
	    - kappa0shift_cd*dq[ia][ib][ic]*dq[id][ic][id]
	    - kappa1shift_cd*dq[ia][ic][id]*dq[ib][ic][id]
	    + kappa1shift_cd*dq[ia][ic][id]*dq[ic][ib][id];
	  
	  tmpdbl2= -2.0*kappa1shift_cd*q0shift_cd*dq[ia][ic][id];
	  for (ie = 0; ie < 3; ie++) {
	    tmpdbl +=
	     tmpdbl2*e_cd[ib][ic][ie]*q[id][ie];
	  }
	}
      }
      sth[ia][ib]+=tmpdbl;
    }
  }

  /* The antisymmetric piece q_ac h_cb - h_ac q_cb. We can
   * rewrite it as q_ac h_bc - h_ac q_bc. */

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      tmpdbl=0.;
      for (ic = 0; ic < 3; ic++) {
  	 tmpdbl += q[ia][ic]*h[ib][ic] - h[ia][ic]*q[ib][ic];
      }
      sth[ia][ib]+=tmpdbl;
      /* This is the minus sign. */
      sth[ia][ib] = -sth[ia][ib];
    }
  }


  return;
}

/*****************************************************************************
 *
 *  blue_phase_chemical_stress
 *
 *  Return the stress sth[3][3] at lattice site index.
 *
 *****************************************************************************/

__device__ void blue_phase_chemical_stress_gpu_d(int index,
						 int *le_index_real_to_buffer_d,
						 double *phi_site_d,
						 double *grad_phi_site_d,
						 double *delsq_phi_site_d,
						 double sth[3][3]){


  int ia;

  double q[3][3];
  double h[3][3];
  double dq[3][3][3];
  double dsq[3][3];

  //  if(threadIdx.x==0 && blockIdx.x==0) printf("in BPCS\n");

/*   phi_get_q_tensor(index, q); */
/*   phi_gradients_tensor_gradient(index, dq); */
/*   phi_gradients_tensor_delsq(index, dsq); */


  /* load phi */

  q[X][X] = phi_site_d[nsites_cd*XX+index];
  q[X][Y] = phi_site_d[nsites_cd*XY+index];
  q[X][Z] = phi_site_d[nsites_cd*XZ+index];
  q[Y][X] = q[X][Y];
  q[Y][Y] = phi_site_d[nsites_cd*YY+index];
  q[Y][Z] = phi_site_d[nsites_cd*YZ+index];
  q[Z][X] = q[X][Z];
  q[Z][Y] = q[Y][Z];
  q[Z][Z] = 0.0 - q[X][X] - q[Y][Y];


  /* load grad phi */
  for (ia = 0; ia < 3; ia++) {
    dq[ia][X][X] = grad_phi_site_d[ia*nsites_cd*5 + XX*nsites_cd + index];
    dq[ia][X][Y] = grad_phi_site_d[ia*nsites_cd*5 + XY*nsites_cd + index];
    dq[ia][X][Z] = grad_phi_site_d[ia*nsites_cd*5 + XZ*nsites_cd + index];
    dq[ia][Y][X] = dq[ia][X][Y];
    dq[ia][Y][Y] = grad_phi_site_d[ia*nsites_cd*5 + YY*nsites_cd + index];
    dq[ia][Y][Z] = grad_phi_site_d[ia*nsites_cd*5 + YZ*nsites_cd + index];
    dq[ia][Z][X] = dq[ia][X][Z];
    dq[ia][Z][Y] = dq[ia][Y][Z];
    dq[ia][Z][Z] = 0.0 - dq[ia][X][X] - dq[ia][Y][Y];
  }


    /* load delsq phi */
  dsq[X][X] = delsq_phi_site_d[XX*nsites_cd+index];
  dsq[X][Y] = delsq_phi_site_d[XY*nsites_cd+index];
  dsq[X][Z] = delsq_phi_site_d[XZ*nsites_cd+index];
  dsq[Y][X] = dsq[X][Y];
  dsq[Y][Y] = delsq_phi_site_d[YY*nsites_cd+index];
  dsq[Y][Z] = delsq_phi_site_d[YZ*nsites_cd+index];
  dsq[Z][X] = dsq[X][Z];
  dsq[Z][Y] = dsq[Y][Z];
  dsq[Z][Z] = 0.0 - dsq[X][X] - dsq[Y][Y];


  //DEV
 
  int i,j,k,icm1,icp1,indexm1,indexp1;
  get_coords_from_index_gpu_d(&i,&j,&k,index,Nall_cd);
  icm1=le_index_real_to_buffer_d[i];
  icp1=le_index_real_to_buffer_d[Nall_cd[X]+i];      

  indexm1 = get_linear_index_gpu_d(icm1,j,k,Nall_cd);
  indexp1 = get_linear_index_gpu_d(icp1,j,k,Nall_cd);

     dq[0][X][X]
       = 0.5*(phi_site_d[nsites_cd*XX+indexp1] - phi_site_d[nsites_cd*XX+indexm1]);
     dq[0][X][Y]
       = 0.5*(phi_site_d[nsites_cd*XY+indexp1] - phi_site_d[nsites_cd*XY+indexm1]);
     dq[0][X][Z]
       = 0.5*(phi_site_d[nsites_cd*XZ+indexp1] - phi_site_d[nsites_cd*XZ+indexm1]);
     dq[0][Y][X]
       = 0.5*(phi_site_d[nsites_cd*XY+indexp1] - phi_site_d[nsites_cd*XY+indexm1]);
     dq[0][Y][Y]
       = 0.5*(phi_site_d[nsites_cd*YY+indexp1] - phi_site_d[nsites_cd*YY+indexm1]);
     dq[0][Y][Z]
       = 0.5*(phi_site_d[nsites_cd*YZ+indexp1] - phi_site_d[nsites_cd*YZ+indexm1]);
     dq[0][Z][X]
       = 0.5*(phi_site_d[nsites_cd*XZ+indexp1] - phi_site_d[nsites_cd*XZ+indexm1]);
     dq[0][Z][Y]
       = 0.5*(phi_site_d[nsites_cd*YZ+indexp1] - phi_site_d[nsites_cd*YZ+indexm1]);
     dq[0][Z][Z]
       = 0. - 0.5*(phi_site_d[nsites_cd*XX+indexp1] - phi_site_d[nsites_cd*XX+indexm1])
       - 0.5*(phi_site_d[nsites_cd*YY+indexp1] - phi_site_d[nsites_cd*YY+indexm1]);

 //END DEV


     blue_phase_compute_h_gpu_d(q, dq, dsq, h);
     //blue_phase_compute_h_gpu_d_test2(q, dq, dsq, h);
  blue_phase_compute_stress_gpu_d(q, dq, h, sth);

  return;
}

/*****************************************************************************
 *
 *  phi_force_calculation_fluid
 *
 *  Compute force from thermodynamic sector via
 *    F_alpha = nalba_beta Pth_alphabeta
 *  using a simple six-point stencil.
 *
 *  Side effect: increments the force at each local lattice site in
 *  preparation for the collision stage.
 *
 *****************************************************************************/


__global__ void phi_force_calculation_fluid_gpu_d(int * le_index_real_to_buffer_d,
						  double *phi_site_d,
						  double *grad_phi_site_d,
						  double *delsq_phi_site_d,
						  double *force_d
					    ) {

  int ia, icm1, icp1;
  int index, index1;
  double pth0[3][3];
  double pth1[3][3];
  double force[3];
  int threadIndex,ii, jj, kk;

 /* CUDA thread index */
 threadIndex = blockIdx.x*blockDim.x+threadIdx.x;
 
 /* Avoid going beyond problem domain */
 if (threadIndex < N_cd[X]*N_cd[Y]*N_cd[Z])
    {


      /* calculate index from CUDA thread index */

      get_coords_from_index_gpu_d(&ii,&jj,&kk,threadIndex,N_cd);
      index = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd,kk+nhalo_cd,Nall_cd);      
      icm1=le_index_real_to_buffer_d[ii+nhalo_cd];
      icp1=le_index_real_to_buffer_d[Nall_cd[X]+ii+nhalo_cd];      
      

	/* Compute pth at current point */
      blue_phase_chemical_stress_gpu_d(index,le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d,pth0);

	/* Compute differences */
	index1 = get_linear_index_gpu_d(icp1,jj+nhalo_cd,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d,pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] = -0.5*(pth1[ia][X] + pth0[ia][X]);
	}

	index1 = get_linear_index_gpu_d(icm1,jj+nhalo_cd,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d,pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] += 0.5*(pth1[ia][X] + pth0[ia][X]);
	}

	
	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd+1,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] -= 0.5*(pth1[ia][Y] + pth0[ia][Y]);
	}

	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd-1,kk+nhalo_cd,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] += 0.5*(pth1[ia][Y] + pth0[ia][Y]);
	}
	

	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd,kk+nhalo_cd+1,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] -= 0.5*(pth1[ia][Z] + pth0[ia][Z]);
	}

	index1 = get_linear_index_gpu_d(ii+nhalo_cd,jj+nhalo_cd,kk+nhalo_cd-1,Nall_cd);
	blue_phase_chemical_stress_gpu_d(index1,le_index_real_to_buffer_d,phi_site_d,grad_phi_site_d,delsq_phi_site_d, pth1);
	for (ia = 0; ia < 3; ia++) {
	  force[ia] += 0.5*(pth1[ia][Z] + pth0[ia][Z]);
	}

	/* Store the force on lattice */
	for (ia=0;ia<3;ia++)
	  force_d[ia*nsites_cd+index]+=force[ia];

    }


  return;
}


/* get linear index from 3d coordinates */
 __device__ static int get_linear_index_gpu_d(int ii,int jj,int kk,int N_d[3])
{
  
  int yfac = N_d[Z];
  int xfac = N_d[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}

/* get 3d coordinates from the index on the accelerator */
__device__ static void get_coords_from_index_gpu_d(int *ii,int *jj,int *kk,int index,int N_d[3])

{
  
  int yfac = N_d[Z];
  int xfac = N_d[Y]*yfac;
  
  *ii = index/xfac;
  *jj = ((index-xfac*(*ii))/yfac);
  *kk = (index-(*ii)*xfac-(*jj)*yfac);

  return;

}
