#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 * phi_datamgmt_gpu.cu
 *  
 * Phi data management for GPU adaptation of Ludwig
 * Alan Gray 
 *
 *****************************************************************************/

#include <assert.h>
#include <stdio.h>
#include <math.h>

#include "pe.h"
#include "utilities_gpu.h"
#include "phi_datamgmt_gpu.h"
#include "util.h"
#include "model.h"
#include "timer.h"


/* edge and halo buffers on accelerator */

double * edgeXLOW_d;
double * edgeXHIGH_d;
double * edgeYLOW_d;
double * edgeYHIGH_d;
double * edgeZLOW_d;
double * edgeZHIGH_d;
double * haloXLOW_d;
double * haloXHIGH_d;
double * haloYLOW_d;
double * haloYHIGH_d;
double * haloZLOW_d;
double * haloZHIGH_d;


/* host memory address pointers for temporary staging of data */
double * phi_site_temp;
double * grad_phi_site_temp;
double * delsq_phi_site_temp;


/* edge and  halo buffers on host */

double * edgeXLOW;
double * edgeXHIGH;
double * edgeYLOW;
double * edgeYHIGH;
double * edgeZLOW;
double * edgeZHIGH;
double * haloXLOW;
double * haloXHIGH;
double * haloYLOW;
double * haloYHIGH;
double * haloZLOW;
double * haloZHIGH;


/* pointers to data resident on accelerator */

extern int * N_d;
double * phi_site_d;
double * phi_site_full_d;
double * h_site_d;
double * stress_site_d;
double * grad_phi_site_d;
double * delsq_phi_site_d;
double * le_index_real_to_buffer_d;

int * le_index_real_to_buffer_temp;

//extern int Nall_cd[3];
//extern int *nsites_cd;


/* data size variables */
static int nhalo;
static int nsites;
static int nop;
static  int N[3];
static  int Nall[3];
static int nhalodataX;
static int nhalodataY;
static int nhalodataZ;
static int nlexbuf;

/* handles for CUDA streams (for ovelapping)*/
static hipStream_t streamX,streamY, streamZ;


void init_phi_gpu(){

  int ic;

  calculate_phi_data_sizes();
  allocate_phi_memory_on_gpu();
  

  /* get le_index_to_real_buffer values */
/*le_index_real_to_buffer holds -1 then +1 translation values */
  for (ic=0; ic<Nall[X]; ic++)
    {

      le_index_real_to_buffer_temp[ic]=
	le_index_real_to_buffer(ic,-1);

      le_index_real_to_buffer_temp[Nall[X]+ic]=
le_index_real_to_buffer(ic,+1);

    }

  hipMemcpy(le_index_real_to_buffer_d, le_index_real_to_buffer_temp, 
	     nlexbuf*sizeof(int),hipMemcpyHostToDevice);

  /* create CUDA streams (for ovelapping)*/
  hipStreamCreate(&streamX);
  hipStreamCreate(&streamY);
  hipStreamCreate(&streamZ);


}


void finalise_phi_gpu()
{
  free_phi_memory_on_gpu();

}

/* calculate sizes of data - needed for memory copies to accelerator */
static void calculate_phi_data_sizes()
{
  coords_nlocal(N);  
  nhalo = coords_nhalo();  

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsites = Nall[X]*Nall[Y]*Nall[Z];

  nop = phi_nop();

  nhalodataX = N[Y] * N[Z] * nhalo * nop ;
  nhalodataY = Nall[X] * N[Z] * nhalo * nop;
  nhalodataZ = Nall[X] * Nall[Y] * nhalo * nop;



  //nlexbuf = le_get_nxbuffer();
/*for holding le buffer index translation */
/* -1 then +1 values */
  nlexbuf = 2*Nall[X]; 



}





/* Allocate memory on accelerator */
static void allocate_phi_memory_on_gpu()
{

  /* temp arrays for staging data on  host */
  phi_site_temp = (double *) malloc(nsites*nop*sizeof(double));
  grad_phi_site_temp = (double *) malloc(nsites*nop*3*sizeof(double));
  delsq_phi_site_temp = (double *) malloc(nsites*nop*sizeof(double));
  le_index_real_to_buffer_temp = (int *) malloc(nlexbuf*sizeof(int));

  hipHostAlloc( (void **)&edgeXLOW, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeXHIGH, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeYLOW, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeYHIGH, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeZLOW, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeZHIGH, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);


  hipHostAlloc( (void **)&haloXLOW, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloXHIGH, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloYLOW, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloYHIGH, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloZLOW, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloZHIGH, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);
  

  hipMalloc((void **) &edgeXLOW_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &edgeXHIGH_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &edgeYLOW_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &edgeYHIGH_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &edgeZLOW_d, nhalodataZ*sizeof(double));
  hipMalloc((void **) &edgeZHIGH_d, nhalodataZ*sizeof(double));
  
  hipMalloc((void **) &haloXLOW_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &haloXHIGH_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &haloYLOW_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &haloYHIGH_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &haloZLOW_d, nhalodataZ*sizeof(double));
  hipMalloc((void **) &haloZHIGH_d, nhalodataZ*sizeof(double));
  
  hipMalloc((void **) &phi_site_d, nsites*nop*sizeof(double));
  hipMalloc((void **) &phi_site_full_d, nsites*9*sizeof(double));
  hipMalloc((void **) &h_site_d, nsites*9*sizeof(double));
  hipMalloc((void **) &stress_site_d, nsites*9*sizeof(double));
  hipMalloc((void **) &delsq_phi_site_d, nsites*nop*sizeof(double));
  hipMalloc((void **) &grad_phi_site_d, nsites*3*nop*sizeof(double));
  hipMalloc((void **) &le_index_real_to_buffer_d, nlexbuf*sizeof(int));


     checkCUDAError("allocate_phi_memory_on_gpu");

}


/* Free memory on accelerator */
static void free_phi_memory_on_gpu()
{

  /* free temp memory on host */
  free(phi_site_temp);
  free(grad_phi_site_temp);
  free(delsq_phi_site_temp);
  free(le_index_real_to_buffer_temp);

  hipHostFree(edgeXLOW);
  hipHostFree(edgeXHIGH);
  hipHostFree(edgeYLOW);
  hipHostFree(edgeYHIGH);
  hipHostFree(edgeZLOW);
  hipHostFree(edgeZHIGH);

  hipHostFree(haloXLOW);
  hipHostFree(haloXHIGH);
  hipHostFree(haloYLOW);
  hipHostFree(haloYHIGH);
  hipHostFree(haloZLOW);
  hipHostFree(haloZHIGH);


  /* free memory on accelerator */

  hipFree(edgeXLOW_d);
  hipFree(edgeXHIGH_d);
  hipFree(edgeYLOW_d);
  hipFree(edgeYHIGH_d);
  hipFree(edgeZLOW_d);
  hipFree(edgeZHIGH_d);

  hipFree(haloXLOW_d);
  hipFree(haloXHIGH_d);
  hipFree(haloYLOW_d);
  hipFree(haloYHIGH_d);
  hipFree(haloZLOW_d);
  hipFree(haloZHIGH_d);

  hipFree(phi_site_d);
  hipFree(phi_site_full_d);
  hipFree(h_site_d);
  hipFree(stress_site_d);
  hipFree(delsq_phi_site_d);
  hipFree(grad_phi_site_d);
  hipFree(le_index_real_to_buffer_d);

}




/* copy phi from host to accelerator */
void put_phi_on_gpu()
{

  int index, ic, jc, kc, iop;
	      

  /* get temp host copies of arrays */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {

	      index = get_linear_index(ic, jc, kc, Nall); 

	      for (iop=0; iop<nop; iop++)
		{

		  phi_site_temp[iop*nsites+index]=phi_op_get_phi_site(index,iop);
		}
	    }
	}
    }


  /* copy data from CPU to accelerator */
  hipMemcpy(phi_site_d, phi_site_temp, nsites*nop*sizeof(double), \
	     hipMemcpyHostToDevice);

  checkCUDAError("put_phi_on_gpu");

}

/* copy grad phi from host to accelerator */
void put_grad_phi_on_gpu()
{

  int index, i, ic, jc, kc, iop;
  double grad_phi[3];
	      

  /* get temp host copies of arrays */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {

	      index = get_linear_index(ic, jc, kc, Nall); 

	      for (iop=0; iop<nop; iop++)
		{
		  phi_gradients_grad_n(index, iop, grad_phi);

		  for (i=0;i<3;i++)
		    {
		      grad_phi_site_temp[i*nsites*nop+iop*nsites+index]=grad_phi[i];
		    }
		}
	    }
	}
    }


  /* copy data from CPU to accelerator */
  hipMemcpy(grad_phi_site_d, grad_phi_site_temp, nsites*nop*3*sizeof(double), \
	     hipMemcpyHostToDevice);


  checkCUDAError("put_grad_phi_on_gpu");

}

/* copy grad phi from accelerator to host*/
void get_grad_phi_from_gpu()
{

  int index, i, ic, jc, kc, iop;
  double grad_phi[3];
	      

  /* copy data from accelerator to CPU */
  hipMemcpy(grad_phi_site_temp, grad_phi_site_d, nsites*nop*3*sizeof(double), \
	     hipMemcpyDeviceToHost);


  /* set grad phi */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {

	      index = get_linear_index(ic, jc, kc, Nall); 

	      for (iop=0; iop<nop; iop++)
		{

		  for (i=0;i<3;i++)
		    {
		      grad_phi[i]=grad_phi_site_temp[i*nsites*nop+iop*nsites+index];
		    }

		  phi_gradients_set_grad_n(index, iop, grad_phi);

		}
	    }
	}
    }



  checkCUDAError("get_grad_phi_from_gpu");

}

/* copy phi from host to accelerator */
void put_delsq_phi_on_gpu()
{

  int index, ic, jc, kc, iop;

  /* get temp host copies of arrays */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {

	      index = get_linear_index(ic, jc, kc, Nall); 

	      for (iop=0; iop<nop; iop++){

		delsq_phi_site_temp[iop*nsites+index] = phi_gradients_delsq_n(index,iop);
	      	      
	      }
	    }
	}
    }


  /* copy data from CPU to accelerator */
  hipMemcpy(delsq_phi_site_d, delsq_phi_site_temp, nsites*nop*sizeof(double), \
	     hipMemcpyHostToDevice);

  checkCUDAError("put_delsq_phi_on_gpu");

}

/* copy delsq phi from accelerator to host*/
void get_delsq_phi_from_gpu()
{

  int index, ic, jc, kc, iop;



  /* copy data from CPU to accelerator */
  hipMemcpy(delsq_phi_site_temp, delsq_phi_site_d, nsites*nop*sizeof(double), \
	     hipMemcpyDeviceToHost);

  /* get temp host copies of arrays */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {

	      index = get_linear_index(ic, jc, kc, Nall); 

	      for (iop=0; iop<nop; iop++){

		phi_gradients_set_delsq_n(index,iop,delsq_phi_site_temp[iop*nsites+index]);
	      	      
	      }
	    }
	}
    }


  checkCUDAError("get_delsq_phi_from_gpu");

}



/* copy phi from accelerator to host */
void get_phi_from_gpu()
{

  int index, ic, jc, kc, iop;
	      

  /* copy data from accelerator to host */
  hipMemcpy(phi_site_temp, phi_site_d, nsites*nop*sizeof(double),	\
         hipMemcpyDeviceToHost);

  /* set phi */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {

	      index = get_linear_index(ic, jc, kc, Nall); 

	      for (iop=0; iop<nop; iop++)
		{

		
		  phi_op_set_phi_site(index,iop,phi_site_temp[iop*nsites+index]);

		}
	    }
	}
    }

  //checkCUDAError("get_phi_site_from_gpu");

}

__global__ void expand_phi_on_gpu_d(double* phi_site_d,double* phi_site_full_d)
{
  
  int index = blockIdx.x*blockDim.x+threadIdx.x;
  
  /* Avoid going beyond problem domain */
  if (index < Nall_cd[X]*Nall_cd[Y]*Nall_cd[Z])
    {
      
      
      /* calculate index from CUDA thread index */
      
      phi_site_full_d[3*X*nsites_cd+X*nsites_cd+index]
	= phi_site_d[nsites_cd*XX+index];
      phi_site_full_d[3*X*nsites_cd+Y*nsites_cd+index]
	= phi_site_d[nsites_cd*XY+index];
      phi_site_full_d[3*X*nsites_cd+Z*nsites_cd+index]
	= phi_site_d[nsites_cd*XZ+index];
      phi_site_full_d[3*Y*nsites_cd+X*nsites_cd+index]
	=  phi_site_full_d[3*X*nsites_cd+Y*nsites_cd+index];
      phi_site_full_d[3*Y*nsites_cd+Y*nsites_cd+index]
	= phi_site_d[nsites_cd*YY+index];
      phi_site_full_d[3*Y*nsites_cd+Z*nsites_cd+index]
	= phi_site_d[nsites_cd*YZ+index];
      phi_site_full_d[3*Z*nsites_cd+X*nsites_cd+index]
	= phi_site_full_d[3*X*nsites_cd+Z*nsites_cd+index];
      phi_site_full_d[3*Z*nsites_cd+Y*nsites_cd+index]
	= phi_site_full_d[3*Y*nsites_cd+Z*nsites_cd+index];
      phi_site_full_d[3*Z*nsites_cd+Z*nsites_cd+index]
	= 0.0 -  phi_site_full_d[3*X*nsites_cd+X*nsites_cd+index]
	-  phi_site_full_d[3*Y*nsites_cd+Y*nsites_cd+index];


    }

}
void expand_phi_on_gpu()
{
  int N[3],nhalo,Nall[3];
  nhalo = coords_nhalo();
  coords_nlocal(N);
  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;
  int nsites=Nall[X]*Nall[Y]*Nall[Z];
  
  hipMemcpyToSymbol(HIP_SYMBOL(N_cd), N, 3*sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(Nall_cd), Nall, 3*sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(nhalo_cd), &nhalo, sizeof(int), 0, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(nsites_cd), &nsites, sizeof(int), 0, hipMemcpyHostToDevice); 
  
  int nblocks=(Nall[X]*Nall[Y]*Nall[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
  
  expand_phi_on_gpu_d<<<nblocks,DEFAULT_TPB>>>
    (phi_site_d,phi_site_full_d);

 
}

void phi_halo_gpu(){

  halo_gpu(1,nop,0,phi_site_d);

}

extern double * velocity_d;
void velocity_halo_gpu(){

  halo_gpu(1,3,0,velocity_d);

}
