/*****************************************************************************
 *
 * utilities_gpu.cu
 *  
 * Data management and other utilities for GPU adaptation of Ludwig
 * Alan Gray/ Alan Richardson 
 *
 *****************************************************************************/

#include <assert.h>
#include <stdio.h>
#include <math.h>

#include "pe.h"
#include "utilities_gpu.h"
#include "phi_datamgmt_gpu.h"
#include "dist_datamgmt_gpu.h"
#include "util.h"
#include "model.h"
#include "timer.h"

//#define GPUS_PER_NODE 4
#define GPUS_PER_NODE 1

/* external pointers to data on host*/
extern const double ma_[NVEL][NVEL];
extern const double mi_[NVEL][NVEL];
extern const double wv[NVEL];
extern const int cv[NVEL][3];
extern const double q_[NVEL][3][3];

extern double * fluxe;
extern double * fluxw;
extern double * fluxy;
extern double * fluxz;

double * ma_d;
double * mi_d;
int * cv_d;
double * q_d;
double * wv_d;
char * site_map_status_d;
double * force_d;
double * velocity_d;
int * N_d;
double * force_global_d;

double * r3_d;
double * d_d;
double * e_d;

double * electric_d;

double * fluxe_d;
double * fluxw_d;
double * fluxy_d;
double * fluxz_d;


/* host memory address pointers for temporary staging of data */

char * site_map_status_temp;
double * force_temp;
double * velocity_temp;

/* data size variables */
static int nhalo;
static int nsites;
static int nop;
static  int N[3];
static  int Nall[3];



/* Perform tasks necessary to initialise accelerator */
void initialise_gpu()
{

  double force_global[3];


  int devicenum=cart_rank()%GPUS_PER_NODE;

  //FERMI0 hack
  //if (devicenum ==1 ) devicenum=4;
  //devicenum=0;

  hipSetDevice(devicenum);

  //hipGetDevice(&devicenum);
  //printf("rank %d running on device %d\n",cart_rank(),devicenum);
  
  calculate_data_sizes();
  allocate_memory_on_gpu();

  /* get global force from physics module */
  fluid_body_force(force_global);

  put_site_map_on_gpu();

  /* copy data from host to accelerator */
  hipMemcpy(N_d, N, 3*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(ma_d, ma_, NVEL*NVEL*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mi_d, mi_, NVEL*NVEL*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(cv_d, cv, NVEL*3*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(wv_d, wv, NVEL*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(q_d, q_, NVEL*3*3*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(force_global_d, force_global, 3*sizeof(double), \
	     hipMemcpyHostToDevice);

  hipMemcpy(r3_d, &r3_, sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(d_d, d_, 3*3*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(e_d, e_, 3*3*3*sizeof(double), hipMemcpyHostToDevice); 

  /* int N[3],nhalo,Nall[3]; */
  
  /* nhalo = coords_nhalo(); */
  /* coords_nlocal(N);  */


  /* Nall[X]=N[X]+2*nhalo; */
  /* Nall[Y]=N[Y]+2*nhalo; */
  /* Nall[Z]=N[Z]+2*nhalo; */
  
  /* int nsites=Nall[X]*Nall[Y]*Nall[Z]; */

  /* printf("XXXX %d\n",nsites); */
   hipMemcpyToSymbol(HIP_SYMBOL(N_cd), N, 3*sizeof(int), 0, hipMemcpyHostToDevice);  
  /* hipMemcpyToSymbol(HIP_SYMBOL(Nall_cd), Nall, 3*sizeof(int), 0, hipMemcpyHostToDevice);  */
  /* hipMemcpyToSymbol(HIP_SYMBOL(nhalo_cd), &nhalo, sizeof(int), 0, hipMemcpyHostToDevice);  */
  /* hipMemcpyToSymbol(HIP_SYMBOL(nsites_cd), &nsites, sizeof(int), 0, hipMemcpyHostToDevice) */; 
 

  

  init_dist_gpu();
  init_phi_gpu();


  checkCUDAError("Init GPU");  


}

/* Perform tasks necessary to finalise accelerator */
void finalise_gpu()
{


  free_memory_on_gpu();
  finalise_dist_gpu();
  //finalise_phi_gpu();
 

  checkCUDAError("Finalise GPU");


}




/* calculate sizes of data - needed for memory copies to accelerator */
static void calculate_data_sizes()
{
  coords_nlocal(N);  
  nhalo = coords_nhalo();  

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsites = Nall[X]*Nall[Y]*Nall[Z];
  nop = phi_nop();


}





/* Allocate memory on accelerator */
static void allocate_memory_on_gpu()
{

  /* temp arrays for staging data on  host */
  force_temp = (double *) malloc(nsites*3*sizeof(double));
  velocity_temp = (double *) malloc(nsites*3*sizeof(double));
  site_map_status_temp = (char *) malloc(nsites*sizeof(char));
  
  hipMalloc((void **) &site_map_status_d, nsites*sizeof(char));
  hipMalloc((void **) &ma_d, NVEL*NVEL*sizeof(double));
  hipMalloc((void **) &mi_d, NVEL*NVEL*sizeof(double));
  hipMalloc((void **) &cv_d, NVEL*3*sizeof(int));
  hipMalloc((void **) &wv_d, NVEL*sizeof(double));
  hipMalloc((void **) &q_d, NVEL*3*3*sizeof(double));
  hipMalloc((void **) &force_d, nsites*3*sizeof(double));
  hipMalloc((void **) &velocity_d, nsites*3*sizeof(double));

  hipMalloc((void **) &fluxe_d, nop*nsites*sizeof(double));
  hipMalloc((void **) &fluxw_d, nop*nsites*sizeof(double));
  hipMalloc((void **) &fluxy_d, nop*nsites*sizeof(double));
  hipMalloc((void **) &fluxz_d, nop*nsites*sizeof(double));
 
  

  hipMalloc((void **) &N_d, sizeof(int)*3);
  hipMalloc((void **) &force_global_d, sizeof(double)*3);

  hipMalloc((void **) &r3_d, sizeof(double));
  hipMalloc((void **) &d_d, sizeof(double)*3*3);
  hipMalloc((void **) &e_d, sizeof(double)*3*3*3);

  hipMalloc((void **) &electric_d, sizeof(double)*3);

  checkCUDAError("allocate_memory_on_gpu");

}


/* Free memory on accelerator */
static void free_memory_on_gpu()
{

  /* free temp memory on host */
  free(force_temp);
  free(velocity_temp);
  free(site_map_status_temp);

  hipFree(ma_d);
  hipFree(mi_d);
  hipFree(cv_d);
  hipFree(wv_d);
  hipFree(q_d);
  hipFree(site_map_status_d);
  hipFree(force_d);
  hipFree(velocity_d);
  hipFree(N_d);
  hipFree(force_global_d);

  hipFree(fluxe_d);
  hipFree(fluxw_d);
  hipFree(fluxy_d);
  hipFree(fluxz_d);

  hipFree(r3_d);
  hipFree(d_d);
  hipFree(e_d);

  hipFree(electric_d);

  checkCUDAError("free_memory_on_gpu");
}


/* copy site map from host to accelerator */
void put_site_map_on_gpu()
{

  int index, ic, jc, kc;
	      

  /* get temp host copies of arrays */
  for (ic=1; ic<=N[X]; ic++)
    {
      for (jc=1; jc<=N[Y]; jc++)
	{
	  for (kc=1; kc<=N[Z]; kc++)
	    {
	      index = coords_index(ic, jc, kc); 

	      site_map_status_temp[index] = site_map_get_status(ic, jc, kc);

	    }
	}
    }

  /* copy data from CPU to accelerator */
  hipMemcpy(site_map_status_d, site_map_status_temp, nsites*sizeof(char), \
	     hipMemcpyHostToDevice);

  checkCUDAError("put_site_map_on_gpu");

}



void zero_force_on_gpu()
{

  int zero=0;
  hipMemset(force_d,zero,nsites*3*sizeof(double));
  checkCUDAError("zero_force_on_gpu");
}



/* copy force from host to accelerator */
void put_force_on_gpu()
{

  int index, i, ic, jc, kc;
  double force[3];
	      

  /* get temp host copies of arrays */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {
	      index = coords_index(ic, jc, kc);

	      hydrodynamics_get_force_local(index,force);
	      	      
	      for (i=0;i<3;i++)
		{
		  force_temp[i*nsites+index]=force[i];
		}
	    }
	}
    }


  /* copy data from CPU to accelerator */
  hipMemcpy(force_d, force_temp, nsites*3*sizeof(double), \
	     hipMemcpyHostToDevice);

  checkCUDAError("put_force_on_gpu");

}

/* copy force from accelerator to host */
void get_force_from_gpu()
{

  int index, i, ic, jc, kc;
  double force[3];
	      
  /* copy data from accelerator to CPU */
  hipMemcpy(force_temp, force_d, nsites*3*sizeof(double), \
	     hipMemcpyDeviceToHost);

  
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {
	      index = coords_index(ic, jc, kc);

	      for (i=0;i<3;i++)
		{
		  force[i]=force_temp[i*nsites+index];
		}

	      hydrodynamics_set_force_local(index,force);
	      	      
	    }
	}
    }



  checkCUDAError("get_force_from_gpu");

}

void get_velocity_from_gpu()
{
  int index,i, ic,jc,kc;
  double velocity[3];

  hipMemcpy(velocity_temp, velocity_d, nsites*3*sizeof(double),
	    hipMemcpyDeviceToHost);

  /* copy velocity from temporary array back to hydrodynamics module */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {
	      index = coords_index(ic, jc, kc);
	      for (i=0;i<3;i++)
		{
		  velocity[i]=velocity_temp[index*3+i];
		}
	      hydrodynamics_set_velocity(index,velocity);
	    }
	}
    }

  checkCUDAError("get_velocity_from_gpu");

}

void put_velocity_on_gpu()
{
  int index,i, ic,jc,kc;
  double velocity[3];


  /* copy velocity from temporary array back to hydrodynamics module */
  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {

	      index = coords_index(ic, jc, kc);
	      hydrodynamics_get_velocity(index,velocity);

	      for (i=0;i<3;i++)
		{
		  velocity_temp[index*3+i]=velocity[i];
		}
	    }
	}
    }

  hipMemcpy(velocity_d, velocity_temp, nsites*3*sizeof(double),
	    hipMemcpyHostToDevice);

  checkCUDAError("put_velocity_on_gpu");


}


void put_fluxes_on_gpu(){

  hipMemcpy(fluxe_d, fluxe, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);
  hipMemcpy(fluxw_d, fluxw, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);
  hipMemcpy(fluxy_d, fluxy, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);
  hipMemcpy(fluxz_d, fluxz, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);


}

void get_fluxes_from_gpu(){

  hipMemcpy(fluxe, fluxe_d, nsites*nop*sizeof(double),
	    hipMemcpyDeviceToHost);
  hipMemcpy(fluxw, fluxw_d, nsites*nop*sizeof(double),
	    hipMemcpyDeviceToHost);
  hipMemcpy(fluxy, fluxy_d, nsites*nop*sizeof(double),
	    hipMemcpyDeviceToHost);
  hipMemcpy(fluxz, fluxz_d, nsites*nop*sizeof(double),
	    hipMemcpyDeviceToHost);


}


/* get linear index from 3d coordinates (host) */
int get_linear_index(int ii,int jj,int kk,int N[3])

{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}


/* check for CUDA errors */
void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
				hipGetErrorString( err) );
		fflush(stdout);
		fflush(stderr);
		exit(EXIT_FAILURE);
	}                         
}
