#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 * utilities_gpu.cu
 *  
 * Alan Gray
 *
 *****************************************************************************/

#include <assert.h>
#include <stdio.h>
#include <math.h>

#include "common_gpu.h"
#include "pe.h"
#include "utilities_gpu.h"
#include "utilities_internal_gpu.h"
#include "field_datamgmt_gpu.h"
#include "comms_gpu.h"
#include "util.h"
#include "model.h"
#include "timer.h"

/* external pointers to data on host*/
extern const double ma_[NVEL][NVEL];
extern const double mi_[NVEL][NVEL];
extern const double wv[NVEL];
extern const int cv[NVEL][3];
extern const double q_[NVEL][3][3];

extern double * fluxe;
extern double * fluxw;
extern double * fluxy;
extern double * fluxz;

double * ma_d;
double * mi_d;
int * cv_d;
double * q_d;
double * wv_d;
char * site_map_status_d;
char * colloid_map_d;
double * colloid_r_d;
int * N_d;
double * force_global_d;
double * tmpscal1_d;
double * tmpscal2_d;

double * r3_d;
double * d_d;
double * e_d;

double * electric_d;

double * fluxe_d;
double * fluxw_d;
double * fluxy_d;
double * fluxz_d;


/* host memory address pointers for temporary staging of data */

char * site_map_status_temp;
char * colloid_map_temp;

/* data size variables */
static int nhalo;
static int nsites;
static int nop;
static  int N[3];
static  int Nall[3];

extern double * colloid_force_d;


/* Perform tasks necessary to initialise accelerator */
void initialise_gpu()
{

  double force_global[3];


  int devicenum=cart_rank()%GPUS_PER_NODE;

  hipSetDevice(devicenum);

  if (cart_rank()==0){
    hipGetDevice(&devicenum);
    printf("master rank running on device %d\n",devicenum);
  }

  calculate_data_sizes();
  allocate_memory_on_gpu();

  /* get global force from physics module */
  fluid_body_force(force_global);

  put_site_map_on_gpu();

  /* copy data from host to accelerator */
  hipMemcpy(N_d, N, 3*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(ma_d, ma_, NVEL*NVEL*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mi_d, mi_, NVEL*NVEL*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(cv_d, cv, NVEL*3*sizeof(int), hipMemcpyHostToDevice); 
  hipMemcpy(wv_d, wv, NVEL*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(q_d, q_, NVEL*3*3*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(force_global_d, force_global, 3*sizeof(double), \
	     hipMemcpyHostToDevice);

  hipMemcpy(r3_d, &r3_, sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(d_d, d_, 3*3*sizeof(double), hipMemcpyHostToDevice); 
  hipMemcpy(e_d, e_, 3*3*3*sizeof(double), hipMemcpyHostToDevice); 

  

  init_comms_gpu();
  init_field_gpu();


  checkCUDAError("Init GPU");  


}

/* Perform tasks necessary to finalise accelerator */
void finalise_gpu()
{


  free_memory_on_gpu();
  finalise_field_gpu();
  //finalise_phi_gpu();
 

  checkCUDAError("Finalise GPU");


}




/* calculate sizes of data - needed for memory copies to accelerator */
static void calculate_data_sizes()
{
  coords_nlocal(N);  
  nhalo = coords_nhalo();  

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsites = Nall[X]*Nall[Y]*Nall[Z];
  nop = phi_nop();


}





/* Allocate memory on accelerator */
static void allocate_memory_on_gpu()
{

  /* temp arrays for staging data on  host */
  site_map_status_temp = (char *) malloc(nsites*sizeof(char));
  colloid_map_temp = (char *) malloc(nsites*sizeof(char));
  char minusone=-1;
  memset(colloid_map_temp,minusone,nsites*sizeof(char));
  
  hipMalloc((void **) &site_map_status_d, nsites*sizeof(char));
  hipMalloc((void **) &colloid_map_d, nsites*sizeof(char));
  hipMalloc((void **) &colloid_r_d, MAX_COLLOIDS*3*sizeof(double));
  hipMalloc((void **) &ma_d, NVEL*NVEL*sizeof(double));
  hipMalloc((void **) &mi_d, NVEL*NVEL*sizeof(double));
  hipMalloc((void **) &cv_d, NVEL*3*sizeof(int));
  hipMalloc((void **) &wv_d, NVEL*sizeof(double));
  hipMalloc((void **) &q_d, NVEL*3*3*sizeof(double));
  hipMalloc((void **) &tmpscal1_d, nsites*sizeof(double));
  hipMalloc((void **) &tmpscal2_d, nsites*sizeof(double));

  hipMalloc((void **) &fluxe_d, nop*nsites*sizeof(double));
  hipMalloc((void **) &fluxw_d, nop*nsites*sizeof(double));
  hipMalloc((void **) &fluxy_d, nop*nsites*sizeof(double));
  hipMalloc((void **) &fluxz_d, nop*nsites*sizeof(double));
  
  hipMalloc((void **) &N_d, sizeof(int)*3);
  hipMalloc((void **) &force_global_d, sizeof(double)*3);



  hipMalloc((void **) &r3_d, sizeof(double));
  hipMalloc((void **) &d_d, sizeof(double)*3*3);
  hipMalloc((void **) &e_d, sizeof(double)*3*3*3);

  hipMalloc((void **) &electric_d, sizeof(double)*3);

  checkCUDAError("allocate_memory_on_gpu");

}


/* Free memory on accelerator */
static void free_memory_on_gpu()
{

  /* free temp memory on host */
  free(site_map_status_temp);
  free(colloid_map_temp);

  hipFree(ma_d);
  hipFree(mi_d);
  hipFree(cv_d);
  hipFree(wv_d);
  hipFree(q_d);
  hipFree(site_map_status_d);
  hipFree(colloid_map_d);
  hipFree(colloid_r_d);
  hipFree(N_d);
  hipFree(force_global_d);

  hipFree(tmpscal1_d);
  hipFree(tmpscal2_d);

  hipFree(fluxe_d);
  hipFree(fluxw_d);
  hipFree(fluxy_d);
  hipFree(fluxz_d);
 
  hipFree(r3_d);
  hipFree(d_d);
  hipFree(e_d);

  hipFree(electric_d);

  checkCUDAError("free_memory_on_gpu");
}

__global__ void printsitemap4421(char * site_map_status_d){

  printf("PPP %d\n",site_map_status_d[4421]);

}

/* copy site map from host to accelerator */
void put_site_map_on_gpu()
{

  int index, ic, jc, kc;
	      

  for (ic=0; ic<Nall[X]; ic++)
    {
      for (jc=0; jc<Nall[Y]; jc++)
	{
	  for (kc=0; kc<Nall[Z]; kc++)
	    {
	      

	      index = get_linear_index(ic, jc, kc, Nall); 
	      site_map_status_temp[index] = site_map_get_status_index(index);

	    }
	}
    }


  /* copy data from CPU to accelerator */
  hipMemcpy(site_map_status_d, site_map_status_temp, nsites*sizeof(char), \
	     hipMemcpyHostToDevice);


  checkCUDAError("put_site_map_on_gpu");

}


colloid_t* colloid_list[MAX_COLLOIDS];
double colloid_r[MAX_COLLOIDS*3];

int build_colloid_list()
{

  int index, icolloid;
  colloid_t *p_c;
  int ncolloids=0;

  // build list of colloids, one entry for each, stored as memory addresses
  for (index=0;index<nsites;index++){
    
    p_c=colloid_at_site_index(index);  
    if(p_c){

      //printf("HHH %f\n", p_c->s.r[0]);
      int match=0;
      for (icolloid=0;icolloid<ncolloids;icolloid++){
	
	if(p_c==colloid_list[icolloid]){
	  match=1;
	  continue;
	}
	
      }
      if (match==0)
	{
	  colloid_list[ncolloids]=p_c;
	  ncolloids++;
	}
      
    }
    
  }

  return ncolloids;

}


/* copy colloid map from host to accelerator */
void put_colloid_map_on_gpu()
{
  
  int index;
  
  colloid_t *p_c;
  int icolloid;
  int ncolloids=build_colloid_list();

  for (index=0;index<nsites;index++){
    
    p_c=colloid_at_site_index(index);  
    if(p_c){
      
      //find out which colloid
      for (icolloid=0;icolloid<ncolloids;icolloid++){
	if(p_c==colloid_list[icolloid])	  break;
      }
      colloid_map_temp[index]=icolloid;
      
      //printf("%d %d\n", index,colloid_map_temp[index]);
    }

  }
  
  //  for (icolloid=0;icolloid<ncolloids;icolloid++)printf("colloid %d %d\n",icolloid,colloid_list[icolloid]);

  /* copy data from CPU to accelerator */
    hipMemcpy(colloid_map_d, colloid_map_temp, nsites*sizeof(char),	\
  	     hipMemcpyHostToDevice);


  checkCUDAError("put_colloid_map_on_gpu");

}

/* copy colloid map from host to accelerator */
void put_colloid_properties_on_gpu()
{
  
  int ia;
  colloid_t *p_c;
  int icolloid;
  int ncolloids=build_colloid_list();
      
   for (icolloid=0;icolloid<ncolloids;icolloid++){
    
     p_c=(colloid_t*) colloid_list[icolloid]; 
     
     //printf("NNN %f\n", p_c->s.r[0]);
     for (ia=0; ia<3; ia++)
       colloid_r[3*icolloid+ia]=p_c->s.r[ia]; 

   } 
  
  /* copy data from CPU to accelerator */
  hipMemcpy(colloid_r_d, colloid_r, ncolloids*3*sizeof(double), \
  	     hipMemcpyHostToDevice);


  checkCUDAError("put_colloid_map_on_gpu");

}





void zero_colloid_force_on_gpu()
{

  int zero=0;
  hipMemset(colloid_force_d,zero,nsites*6*3*sizeof(double));
  checkCUDAError("zero_colloid_force_on_gpu");
}




extern double * ftmp;
void put_fluxes_on_gpu(){

  int nop=phi_nop();
  int index,n;

  //transpose
  for (index=0;index<nsites;index++){
    for (n=0;n<nop;n++){
      ftmp[n*nsites+index]=fluxe[nop*index+n];
	}
  }
  hipMemcpy(fluxe_d, ftmp, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);



  for (index=0;index<nsites;index++){
    for (n=0;n<nop;n++){
      ftmp[n*nsites+index]=fluxw[nop*index+n];
	}
  }
  hipMemcpy(fluxw_d, ftmp, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);


  for (index=0;index<nsites;index++){
    for (n=0;n<nop;n++){
      ftmp[n*nsites+index]=fluxy[nop*index+n];
	}
  }
  hipMemcpy(fluxy_d, ftmp, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);



  for (index=0;index<nsites;index++){
    for (n=0;n<nop;n++){
      ftmp[n*nsites+index]=fluxz[nop*index+n];
	}
  }
  hipMemcpy(fluxz_d, ftmp, nsites*nop*sizeof(double),
	    hipMemcpyHostToDevice);


  /* hipMemcpy(fluxw_d, fluxw, nsites*nop*sizeof(double), */
  /* 	    hipMemcpyHostToDevice); */
  /* hipMemcpy(fluxy_d, fluxy, nsites*nop*sizeof(double), */
  /* 	    hipMemcpyHostToDevice); */
  /* hipMemcpy(fluxz_d, fluxz, nsites*nop*sizeof(double), */
  /* 	    hipMemcpyHostToDevice); */


}

/* void get_fluxes_from_gpu(){ */

/*   hipMemcpy(fluxe, fluxe_d, nsites*nop*sizeof(double), */
/* 	    hipMemcpyDeviceToHost); */
/*   hipMemcpy(fluxw, fluxw_d, nsites*nop*sizeof(double), */
/* 	    hipMemcpyDeviceToHost); */
/*   hipMemcpy(fluxy, fluxy_d, nsites*nop*sizeof(double), */
/* 	    hipMemcpyDeviceToHost); */
/*   hipMemcpy(fluxz, fluxz_d, nsites*nop*sizeof(double), */
/* 	    hipMemcpyDeviceToHost); */


/* } */



__global__ void printgpuint(int *array_d, int index){

  printf("GPU array [%d] = %d \n",index,array_d[index]);

}

__global__ void printgpudouble(double *array_d, int index){

  printf("GPU array [%d] = %e \n",index,array_d[index]);

}


/* get linear index from 3d coordinates (host) */
int get_linear_index(int ii,int jj,int kk,int N[3])

{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}


/* check for CUDA errors */
void checkCUDAError(const char *msg)
{
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) 
	{
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
				hipGetErrorString( err) );
		fflush(stdout);
		fflush(stderr);
		exit(EXIT_FAILURE);
	}                         
}
