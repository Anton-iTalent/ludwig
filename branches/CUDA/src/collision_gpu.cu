#include "hip/hip_runtime.h"
 /*****************************************************************************
 *
 *  collision_gpu.c
 *
 *  GPU implementation of collision functionality
 * 
 *  Alan Gray
 * 
 *****************************************************************************/

#include <assert.h>
#include <stdio.h>
#include <math.h>


/* below define needed to stop repeated declaration of distribution_ndist */
#define INCLUDING_FROM_GPU_SOURCE 

#include "model.h"
#include "site_map.h"
#include "collision_gpu.h"
#include "collision_internal_gpu.h"
#include "collision.h"
#include "fluctuations.h"

//#include "free_energy.h"
//#include "phi_cahn_hilliard.h"

#include "control.h"
#include "propagation_ode.h"

static int nrelax_ = RELAXATION_M10;     /* [RELAXATION_M10|TRT|BGK] */
                                         /* Default is M10 */
static int isothermal_fluctuations_ = 0; /* Flag for noise. */

static double rtau_shear;       /* Inverse relaxation time for shear modes */
static double rtau_bulk;        /* Inverse relaxation time for bulk modes */
static double var_shear;        /* Variance for shear mode fluctuations */
static double var_bulk;         /* Variance for bulk mode fluctuations */
static double rtau_[NVEL];      /* Inverse relaxation times */
static double noise_var[NVEL];  /* Noise variances */

static double rtau2;

extern const double d_[3][3];

void collide_gpu(void) {

  int ndist,nhalo;
  double mobility;
  int N[3];

  ndist = distribution_ndist();
  nhalo = coords_nhalo();
  coords_nlocal(N); 

  collision_relaxation_times_set_gpu();

  mobility = phi_cahn_hilliard_mobility();
  rtau2 = 2.0 / (1.0 + 2.0*mobility);


/* copy constants to accelerator (constant on-chip read-only memory) */
  copy_constants_to_gpu();
  
  /* set up CUDA grid */
  /* 1D decomposition - use x grid and block dimension only */ 
  int nblocks=(N[X]*N[Y]*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;

  if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_M10)

    {

   collision_multirelaxation_gpu_d<<<nblocks,DEFAULT_TPB>>>(ndist,nhalo, 
    		  N_d,force_global_d, f_d, site_map_status_d, 
    			force_d, velocity_d);

    }

  if  (ndist == 2 && is_propagation_ode() == 0) 
    { 

      collision_binary_lb_gpu_d<<<nblocks,DEFAULT_TPB>>>(ndist, nhalo, N_d, 					      force_global_d, 
					      f_d, 
					      site_map_status_d, 
					       phi_site_d,		
					       grad_phi_site_d,	
					       delsq_phi_site_d,	
							 force_d, 
							 velocity_d);   

    }

  if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_BGK)
    {
      printf("Error, KGK collision not supported yet in GPU version\n");
      exit(1);

    }

  
  hipDeviceSynchronize();

  return;
}

/*****************************************************************************
 *
 *  collision_multirelaxation
 *
 *  Collision with (potentially) different relaxation times for each
 *  different mode.
 *
 *  The matrices ma_ and mi_ project the distributions onto the
 *  modes, and vice-versa, respectively, for the current LB model.
 *
 *  The collision conserves density, and momentum (to within any
 *  body force present). The stress modes, and ghost modes, are
 *  relaxed toward their equilibrium values.
 *
 *  If ghost modes are not required, nmodes_ can be set equal to
 *  the number of hydrodynamic modes. Otherwise nmodes_ = NVEL.
 * 
 *  Adapted to run on GPU: Alan Gray / Alan Richardson  
 *
 *****************************************************************************/
__global__ void collision_multirelaxation_gpu_d(int ndist, int nhalo, int N[3], 
					      const double* __restrict__ force_global_d, 
					      double* __restrict__ f_d, 
					      const char* __restrict__ site_map_status_d, 
					      const double* __restrict__ force_d, 
    			       		      double* __restrict__ velocity_d
					      )
{

  int       index;       /* site indices */
  int       p, m;                    /* velocity index */
  int       ia, ib;                  /* indices ("alphabeta") */

  double    mode[NVEL];              /* Modes; hydrodynamic + ghost */
  double    rho, rrho;               /* Density, reciprocal density */
  double    u[3];                    /* Velocity */
  double    s[3][3];                 /* Stress */
  double    seq[3][3];               /* Equilibrium stress */
  double    shat[3][3];              /* random stress */
  double    ghat[NVEL];              /* noise for ghosts */
  double    rdim;                    /* 1 / dimension */

  double    force[3];                /* External force */
  double    tr_s, tr_seq;

  double    force_local[3];

  int threadIndex, nsite, Nall[3], ii, jj, kk, xfac, yfac;

  
  fluctuations_off_gpu_d(shat, ghat);

  rdim = 1.0/NDIM;

  for (ia = 0; ia < 3; ia++) {
    u[ia] = 0.0;
  }

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsite = Nall[X]*Nall[Y]*Nall[Z];

  /* CUDA thread index */
  threadIndex = blockIdx.x*blockDim.x+threadIdx.x;

  /* Avoid going beyond problem domain */
  if (threadIndex < N[X]*N[Y]*N[Z])
    {

      /* calculate index from CUDA thread index */
      yfac = N[Z];
      xfac = N[Y]*yfac;
      
      ii = threadIndex/xfac;
      jj = ((threadIndex-xfac*ii)/yfac);
      kk = (threadIndex-ii*xfac-jj*yfac);
      
      index = get_linear_index_gpu_d(ii+nhalo,jj+nhalo,kk+nhalo,Nall);
      
      if (site_map_status_d[index] == FLUID)
	{
	  
	  
	  
	  /* Compute all the modes */
	  
	  for (m = 0; m < NVEL; m++) {
	   double dtmp = 0.;
	  for (p = 0; p < NVEL; p++) {
	    dtmp += f_d[nsite*p + index]*ma_cd[m][p];
	  }
	  mode[m] = dtmp;
	  }
	  

	  /* For convenience, write out the physical modes, that is,
	   * rho, NDIM components of velocity, independent components
	   * of stress (upper triangle), and lower triangle. */
	  
	  rho = mode[0];
	  for (ia = 0; ia < NDIM; ia++) {
	    u[ia] = mode[1 + ia];
	  }
	  
	  
	  m = 0;
	  for (ia = 0; ia < NDIM; ia++) {
	    for (ib = ia; ib < NDIM; ib++) {
	      s[ia][ib] = mode[1 + NDIM + m++];
	    }
	  }
	  
	  for (ia = 1; ia < NDIM; ia++) {
	    for (ib = 0; ib < ia; ib++) {
	      s[ia][ib] = s[ib][ia];
	    }
	  }
	  
	  
	  
	  /* Compute the local velocity, taking account of any body force */
	  
	  rrho = 1.0/rho;
	  /* hydrodynamics_get_force_local(index, force_local); */
	  for (ia = 0; ia < 3; ia++) {
	    force_local[ia] = force_d[ia*nsite+index];
	  }
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    force[ia] = (force_global_d[ia] + force_local[ia]);
	    u[ia] = rrho*(u[ia] + 0.5*force[ia]);
	  }
	  
	  /* hydrodynamics_set_velocity(index, u); */
	  for (ia = 0; ia < 3; ia++) {
	    velocity_d[ia*nsite+index] = u[ia];
	  }
	  
	  /* Relax stress with different shear and bulk viscosity */
	  
	  tr_s   = 0.0;
	  tr_seq = 0.0;
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    /* Set equilibrium stress */
	    for (ib = 0; ib < NDIM; ib++) {
	      seq[ia][ib] = rho*u[ia]*u[ib];
	    }
	    /* Compute trace */
	    tr_s   += s[ia][ia];
	    tr_seq += seq[ia][ia];
	  }
	  
	  /* Form traceless parts */
	  for (ia = 0; ia < NDIM; ia++) {
	    s[ia][ia]   -= rdim*tr_s;
	    seq[ia][ia] -= rdim*tr_seq;
	  }
	  
	  /* Relax each mode */
	  tr_s = tr_s - rtau_bulk_d*(tr_s - tr_seq);
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    for (ib = 0; ib < NDIM; ib++) {
	      s[ia][ib] -= rtau_shear_d*(s[ia][ib] - seq[ia][ib]);
	      s[ia][ib] += d_cd[ia][ib]*rdim*tr_s;
	      
	      /* Correction from body force (assumes equal relaxation times) */
	      
	      s[ia][ib] += (2.0-rtau_shear_d)*(u[ia]*force[ib] + force[ia]*u[ib]);
	    }
	  }
	  
	  //if (isothermal_fluctuations_) 	    collision_fluctuations(index, shat, ghat);
	  
	  /* Now reset the hydrodynamic modes to post-collision values:
	   * rho is unchanged, velocity unchanged if no force,
	   * independent components of stress, and ghosts. */
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    mode[1 + ia] += force[ia];
	  }

	  
	  m = 0;
	  for (ia = 0; ia < NDIM; ia++) {
	    for (ib = ia; ib < NDIM; ib++) {
	      mode[1 + NDIM + m++] = s[ia][ib] + shat[ia][ib];
	    }
	  }
	  
	  /* Ghost modes are relaxed toward zero equilibrium. */
	  
	  for (m = NHYDRO; m < NVEL; m++) {
	    mode[m] = mode[m] - rtau_d[m]*(mode[m] - 0.0) + ghat[m];
	  }
	  
	  /* Project post-collision modes back onto the distribution */
	  
	  for (p = 0; p < NVEL; p++) {
	   double dtmp = 0.;
	  for (m = 0; m < NVEL; m++) {
	    dtmp += mi_cd[p][m]*mode[m];
	  }
	  f_d[nsite*p + index] = dtmp;
	  }
	  
	}
      
    }   
  
  
  return;
}

/*****************************************************************************
 *
 *  collision_binary_lb_gpu_d
 *
 *  Binary LB collision stage (here we are progressing toward
 *  decoupled version).
 *
 *  This follows the single fluid version above, with the addition
 *  that the equilibrium stress includes the thermodynamic term
 *  following Swift etal.
 *
 *  We also have to update the second distribution g from the
 *  order parameter modes phi, jphi[3], sphi[3][3].
 *
 *  There are two choices:
 *    1. relax jphi[i] toward equilibrium phi*u[i] at rate rtau2
 *       AND
 *       fix sphi[i][j] = phi*u[i]*u[j] + mu*d_[i][j]
 *       so the mobility enters through rtau2 (J. Stat. Phys. 2005).
 *    2.
 *       fix jphi[i] = phi*u[i] (i.e. relaxation time == 1.0)
 *       AND
 *       fix sphi[i][j] = phi*u[i]*u[j] + mobility*mu*d_[i][j]
 *       so the mobility enters with chemical potential (Kendon etal 2001).
 *
 *   As there seems to be little to choose between the two in terms of
 *   results, I prefer 2, as it avoids the calculation of jphi[i] from
 *   from the distributions g. However, keep 1 so tests don't break!
 *
 *   However, for asymmetric quenches version 1 may be preferred.
 *
 *   The reprojection of g moves phi (mostly) into the non-propagating
 *   distribution following J. Stat. Phys. (2005).
 *
 *  Adapted to run on GPU: Alan Gray / Alan Richardson  
 *
 *****************************************************************************/


__global__ void collision_binary_lb_gpu_d(int ndist, int nhalo, int N[3], 
					  const double* __restrict__ force_global_d, 
					  double* __restrict__ f_d, 
					  const char* __restrict__ site_map_status_d, 
					  const double* __restrict__ phi_site_d,		
					  const double* __restrict__ grad_phi_site_d,	
					  const double* __restrict__ delsq_phi_site_d,	
					  const double* __restrict__ force_d, 
					  double* __restrict__ velocity_d) 
{

  int       index;                   /* site indices */
  int       p, m;                    /* velocity index */
  int       i, j;                    /* summed over indices ("alphabeta") */

  double    mode[NVEL];              /* Modes; hydrodynamic + ghost */
  double    rho, rrho;               /* Density, reciprocal density */
  double    u[3];                    /* Velocity */
  double    s[3][3];                 /* Stress */
  double    seq[3][3];               /* equilibrium stress */
  double    shat[3][3];              /* random stress */
  double    ghat[NVEL];              /* noise for ghosts */

  double    force[3];                /* External force */
  double    tr_s, tr_seq;

  double    force_local[3];

  double f_loc[2*NVEL]; /* thread local copy of f_ data */

  const double   r3     = (1.0/3.0);

  double    phi, jdotc, sphidotq;    /* modes */
  double    jphi[3];
  double    sth[3][3], sphi[3][3];
  double    mu;                      /* Chemical potential */
  const double r2rcs4 = 4.5;         /* The constant 1 / 2 c_s^4 */


  int threadIndex, nsite, Nall[3], ii, jj, kk, xfac, yfac;

  /* ndist is always 2 in this routine. Use of hash define may help compiler */
#define NDIST 2


  fluctuations_off_gpu_d(shat, ghat); 

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsite = Nall[X]*Nall[Y]*Nall[Z];

  /* CUDA thread index */
  threadIndex = blockIdx.x*blockDim.x+threadIdx.x;

  /* Avoid going beyond problem domain */
  if (threadIndex < N[X]*N[Y]*N[Z])
    {
      
      /* calculate index from CUDA thread index */
      yfac = N[Z];
      xfac = N[Y]*yfac;
      
      ii = threadIndex/xfac;
      jj = ((threadIndex-xfac*ii)/yfac);
      kk = (threadIndex-ii*xfac-jj*yfac);
      
      index = get_linear_index_gpu_d(ii+nhalo,jj+nhalo,kk+nhalo,Nall);
      
/*       if (site_map_status_d[index] == FLUID) */
/* 	{ */
	  
	  
	  /* load data into registers */
	  for(p = 0; p < NVEL; p++) {
	    for(m = 0; m < NDIST; m++) {
	      f_loc[NVEL*m+p] = f_d[nsite*NDIST*p + nsite*m + index];
	    }
	  }
	  
	  
	  /* Compute all the modes */
	  for (m = 0; m < NVEL; m++) {
	    double mode_tmp = 0.0;
	    for (p = 0; p < NVEL; p++) {
	      mode_tmp += f_loc[p]*ma_cd[m][p];
	    }
	    mode[m] = mode_tmp;
	  }

	  
	  /* For convenience, write out the physical modes. */
	  
	  rho = mode[0];
	  for (i = 0; i < 3; i++) {
	    u[i] = mode[1 + i];
	  }
	  s[X][X] = mode[4];
	  s[X][Y] = mode[5];
	  s[X][Z] = mode[6];
	  s[Y][X] = s[X][Y];
	  s[Y][Y] = mode[7];
	  s[Y][Z] = mode[8];
	  s[Z][X] = s[X][Z];
	  s[Z][Y] = s[Y][Z];
	  s[Z][Z] = mode[9];
	  
	  /* Compute the local velocity, taking account of any body force */
	  
	  rrho = 1.0/rho;
	  /* hydrodynamics_get_force_local(index, force_local); */
	  for (i = 0; i < 3; i++) {
	    force_local[i] = force_d[i*nsite+index];
	  }
	  
	  for (i = 0; i < 3; i++) {
	    force[i] = (force_global_d[i] + force_local[i]);
	    u[i] = rrho*(u[i] + 0.5*force[i]);
	  }
	  /* hydrodynamics_set_velocity(index, u); */
	  for (i = 0; i < 3; i++) {
	    velocity_d[i*nsite+index] = u[i];
	  }
	  
	  /* Compute the thermodynamic component of the stress */
	  
	  symmetric_chemical_stress_gpu_d(index, sth, phi_site_d,
					  grad_phi_site_d,
					  delsq_phi_site_d,nsite);

	  /* Relax stress with different shear and bulk viscosity */
	  
	  tr_s   = 0.0;
	  tr_seq = 0.0;
	  
	  for (i = 0; i < 3; i++) {
	    /* Set equilibrium stress, which includes thermodynamic part */
	    for (j = 0; j < 3; j++) {
	      seq[i][j] = rho*u[i]*u[j] + sth[i][j];
	    }
	  /* Compute trace */
	    tr_s   += s[i][i];
	    tr_seq += seq[i][i];
	  }
	  
	  /* Form traceless parts */
	  for (i = 0; i < 3; i++) {
	    s[i][i]   -= r3*tr_s;
	    seq[i][i] -= r3*tr_seq;
	  }
	  
	  /* Relax each mode */
	  tr_s = tr_s - rtau_bulk_d*(tr_s - tr_seq);
	  
	  for (i = 0; i < 3; i++) {
	    for (j = 0; j < 3; j++) {
	      s[i][j] -= rtau_shear_d*(s[i][j] - seq[i][j]);
	      s[i][j] += d_cd[i][j]*r3*tr_s;
	      
	      /* Correction from body force (assumes equal relaxation times) */
	      
	      s[i][j] += (2.0-rtau_shear_d)*(u[i]*force[j] + force[i]*u[j]);
	      shat[i][j] = 0.0;
	    }
	  }
	  
	  //if (isothermal_fluctuations_) fluctuations_on(shat, ghat);
	  
	  /* Now reset the hydrodynamic modes to post-collision values */
	  
	  mode[1] = mode[1] + force[X];    /* Conserved if no force */
	  mode[2] = mode[2] + force[Y];    /* Conserved if no force */
	  mode[3] = mode[3] + force[Z];    /* Conserved if no force */
	  mode[4] = s[X][X] + shat[X][X];
	  mode[5] = s[X][Y] + shat[X][Y];
	  mode[6] = s[X][Z] + shat[X][Z];
	  mode[7] = s[Y][Y] + shat[Y][Y];
	  mode[8] = s[Y][Z] + shat[Y][Z];
	  mode[9] = s[Z][Z] + shat[Z][Z];
	  
	  /* Ghost modes are relaxed toward zero equilibrium. */
	  
	  for (m = NHYDRO; m < NVEL; m++) {
	    mode[m] = mode[m] - rtau_d[m]*(mode[m] - 0.0) + ghat[m];
	  }
	  
	  /* Project post-collision modes back onto the distribution */


  	  double f_tmp;
	  
 	  for (p = 0; p < NVEL; p++) {
 	    f_tmp = 0.0;
 	    for (m = 0; m < NVEL; m++) {
 	      f_tmp += mi_cd[p][m]*mode[m];
 	    }
	    f_d[nsite*NDIST*p + index] = f_tmp;
 	}

	/* Now, the order parameter distribution */

	phi = phi_site_d[index];
	mu = symmetric_chemical_potential_gpu_d(index, phi_site_d,
						delsq_phi_site_d);
	
	jphi[X] = 0.0;
	jphi[Y] = 0.0;
	jphi[Z] = 0.0;
	for (p = 1; p < NVEL; p++) {
	  for (i = 0; i < 3; i++) {
	    jphi[i] += f_loc[NVEL + p]*cv_cd[p][i];
	  }
	}
	
	/* Relax order parameters modes. See the comments above. */
	
	for (i = 0; i < 3; i++) {
	  for (j = 0; j < 3; j++) {
	    sphi[i][j] = phi*u[i]*u[j] + mu*d_cd[i][j];
	  }
	  jphi[i] = jphi[i] - rtau2_d*(jphi[i] - phi*u[i]);
	}
	
	/* Now update the distribution */
	
	for (p = 0; p < NVEL; p++) {
	  
	  int dp0 = (p == 0);
	  jdotc    = 0.0;
	  sphidotq = 0.0;
	  
	  for (i = 0; i < 3; i++) {
	    jdotc += jphi[i]*cv_cd[p][i];
	    for (j = 0; j < 3; j++) {
	      sphidotq += sphi[i][j]*q_cd[p][i][j];
	    }
	  }
	  
	  /* Project all this back to the distributions. The magic
	   * here is to move phi into the non-propagating distribution. */
	  
	  f_d[nsite*NDIST*p+nsite+index]
	    = wv_cd[p]*(jdotc*rcs2_d + sphidotq*r2rcs4) + phi*dp0;
	  
	}
	
	
/* 	} */
      
    }
  
  return;
}

/*****************************************************************************
 *
 *  collision_relaxation_times_set
 *
 *  Note there is an extra normalisation in the lattice fluctuations
 *  which would otherwise give effective kT = cs2
 *
 *****************************************************************************/

void collision_relaxation_times_set_gpu(void) {

  int p;
  double kt;
  double tau_s;
  double tau_b;
  double tau_g;

  extern int is_propagation_ode(void);
 
  if (is_propagation_ode()) {
    rtau_shear = 1.0 / (3.0*get_eta_shear());
    rtau_bulk  = 1.0 / (3.0*get_eta_bulk());
  }
  else {
    rtau_shear = 2.0 / (1.0 + 6.0*get_eta_shear());
    rtau_bulk  = 2.0 / (1.0 + 6.0*get_eta_bulk());
  }

  /* Initialise the relaxation times */

  if (nrelax_ == RELAXATION_M10) {
    for (p = NHYDRO; p < NVEL; p++) {
      rtau_[p] = 1.0;
    }
  }

  if (nrelax_ == RELAXATION_BGK) {
    for (p = 0; p < NVEL; p++) {
      rtau_[p] = rtau_shear;
    }
  }

  if (nrelax_ == RELAXATION_TRT) {

    assert(NVEL != 9);

    tau_g = 2.0/(1.0 + (3.0/8.0)*rtau_shear);

    if (NVEL == 15) {
      rtau_[10] = rtau_shear;
      rtau_[11] = tau_g;
      rtau_[12] = tau_g;
      rtau_[13] = tau_g;
      rtau_[14] = rtau_shear;
    }

    if (NVEL == 19) {
      rtau_[10] = rtau_shear;
      rtau_[14] = rtau_shear;
      rtau_[18] = rtau_shear;

      rtau_[11] = tau_g;
      rtau_[12] = tau_g;
      rtau_[13] = tau_g;
      rtau_[15] = tau_g;
      rtau_[16] = tau_g;
      rtau_[17] = tau_g;
    }
  }

  if (isothermal_fluctuations_) {

    tau_s = 1.0/rtau_shear;
    tau_b = 1.0/rtau_bulk;

    /* Initialise the stress variances */

    kt = fluid_kt();
    kt = kt*rcs2; /* Without normalisation kT = cs^2 */

    var_bulk =
      sqrt(kt)*sqrt(2.0/9.0)*sqrt((tau_b + tau_b - 1.0)/(tau_b*tau_b));
    var_shear =
      sqrt(kt)*sqrt(1.0/9.0)*sqrt((tau_s + tau_s - 1.0)/(tau_s*tau_s));

    /* Noise variances */

    for (p = NHYDRO; p < NVEL; p++) {
      tau_g = 1.0/rtau_[p];
      noise_var[p] =
	sqrt(kt/norm_[p])*sqrt((tau_g + tau_g - 1.0)/(tau_g*tau_g));
    }
  }

  return;
}



/*****************************************************************************
 *
 *  copy_constants_to_gpu
 *
 *  copy constants to accelerator (constant on-chip read-only memory)
 *
 *****************************************************************************/

  void   copy_constants_to_gpu(){

    double a_,b_,kappa_;

    RUN_get_double_parameter("A", &a_);
    RUN_get_double_parameter("B", &b_);
    RUN_get_double_parameter("K", &kappa_);

   /* copy constant values to accelerator (on-chip read-only memory) */
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_shear_d), &rtau_shear, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_bulk_d), &rtau_bulk, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_d), rtau_, NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(wv_cd), wv, NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(ma_cd), ma_, NVEL*NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mi_cd), mi_, NVEL*NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(d_cd), d_, 3*3*sizeof(double), 0,	
    		       hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(cv_cd), cv, NVEL*3*sizeof(int), 0,
			hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(q_cd), q_, NVEL*3*3*sizeof(double), 0,
			hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(rtau2_d), &rtau2, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rcs2_d), &rcs2, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(a_d), &a_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(b_d), &b_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kappa_d), &kappa_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
  }


/*****************************************************************************
 *
 *  fluctuations_off_gpu_d
 *
 *  Return zero fluctuations for stress (shat) and ghost (ghat) modes.
 *
 *****************************************************************************/

__device__ void fluctuations_off_gpu_d(double shat[3][3], double ghat[NVEL]) {

  int ia, ib;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      shat[ia][ib] = 0.0;
    }
  }

  for (ia = NHYDRO; ia < NVEL; ia++) {
    ghat[ia] = 0.0;
  }

  return;
}



/****************************************************************************
 *
 *  symmetric_chemical_stress
 *
 *  Return the chemical stress tensor for given position index.
 *
 *  P_ab = [1/2 A phi^2 + 3/4 B phi^4 - kappa phi \nabla^2 phi
 *       -  1/2 kappa (\nbla phi)^2] \delta_ab
 *       +  kappa \nalba_a phi \nabla_b phi
 *
 ****************************************************************************/
__device__ void symmetric_chemical_stress_gpu_d(const int index, 
						double s[3][3],
						const double* __restrict__ phi_site_d, 
						const double* __restrict__ grad_phi_site_d, 
						const double* __restrict__ delsq_phi_site_d,
						int nsite) {

  int ia, ib;
  double phi;
  double delsq_phi;
  double grad_phi[3];
  double p0;

  phi = phi_site_d[index];
  delsq_phi = delsq_phi_site_d[index];
  for (ia = 0; ia < 3; ia++) grad_phi[ia]=grad_phi_site_d[ia*nsite+index];

  p0 = 0.5*a_d*phi*phi + 0.75*b_d*phi*phi*phi*phi
    - kappa_d*phi*delsq_phi - 
    0.5*kappa_d*dot_product_gpu_d(grad_phi, grad_phi);

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      s[ia][ib] = p0*d_cd[ia][ib]	+ kappa_d*grad_phi[ia]*grad_phi[ib];
    }
  }

  return;
}


/****************************************************************************
 *
 *  symmetric_chemical_potential
 *
 *  The chemical potential \mu = \delta F / \delta \phi
 *                             = a\phi + b\phi^3 - \kappa\nabla^2 \phi
 *
 ****************************************************************************/

__device__ double symmetric_chemical_potential_gpu_d(const int index, 
		const double* __restrict__  phi_site_d, const double* __restrict__  delsq_phi_site_d) {

  double mu, phi;

  phi = phi_site_d[index];
  mu = a_d*phi + b_d*phi*phi*phi - kappa_d*delsq_phi_site_d[index];

  return mu;
}


/*****************************************************************************
 *
 *  dot_product
 *
 *****************************************************************************/

__device__ double dot_product_gpu_d(const double a[3], const double b[3]) {

	return (a[X]*b[X] + a[Y]*b[Y] + a[Z]*b[Z]);
}


/* get linear index from 3d coordinates */
 __device__ static int get_linear_index_gpu_d(int ii,int jj,int kk,int N[3])
{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}
