#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 *  collision_gpu.c
 *
 *  Collision stage routines and associated data.
 *
 *  Isothermal fluctuations following Adhikari et al., Europhys. Lett
 *  (2005).
 *
 *  The relaxation times can be set to give either 'm10', BGK or
 *  'two-relaxation' time (TRT) models.
 *
 *  $Id: collision.c 1728 2012-07-18 08:41:51Z agray3 $
 *
 *  Edinburgh Soft Matter and Statistical Physics Group and
 *  Edinburgh Parallel Computing Centre
 *
 *  Kevin Stratford (kevin@epcc.ed.ac.uk)
 *  (c) 2011 The University of Edinburgh
 *
 *  Adapted to run on GPU: Alan Gray
 * 
 *****************************************************************************/

#include <assert.h>
#include <stdio.h>
#include <math.h>


/* below define needed to stop repeated declaration of distribution_ndist */
#define INCLUDING_FROM_GPU_SOURCE 

#include "model.h"
#include "site_map.h"
#include "collision_gpu.h"
#include "collision.h"
#include "fluctuations.h"

//#include "free_energy.h"
//#include "phi_cahn_hilliard.h"

#include "control.h"
#include "propagation_ode.h"

static int nmodes_ = NVEL;               /* Modes to use in collsion stage */
static int nrelax_ = RELAXATION_M10;     /* [RELAXATION_M10|TRT|BGK] */
                                         /* Default is M10 */
static int isothermal_fluctuations_ = 0; /* Flag for noise. */

static double rtau_shear;       /* Inverse relaxation time for shear modes */
static double rtau_bulk;        /* Inverse relaxation time for bulk modes */
static double var_shear;        /* Variance for shear mode fluctuations */
static double var_bulk;         /* Variance for bulk mode fluctuations */
static double rtau_[NVEL];      /* Inverse relaxation times */
static double noise_var[NVEL];  /* Noise variances */

static fluctuations_t * fl_;

static double rtau2;

/* static void collision_multirelaxation(void); */
/* static void collision_binary_lb(void); */
/* static void collision_bgk(void); */

/* static void fluctuations_off(double shat[3][3], double ghat[NVEL]); */
/*        void collision_fluctuations(int index, double shat[3][3], */
/* 				   double ghat[NVEL]); */


/*****************************************************************************
 *
 *  collide
 *
 *  Driver routine for the collision stage.
 *
 *  Note that the ODE propagation currently uses ndist == 2, as
 *  well as the LB binary, hence the logic.
 *
 *****************************************************************************/

/* void collide(void) { */

/*   int ndist; */

/*   ndist = distribution_ndist(); */
/*   collision_relaxation_times_set(); */

/*   if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_M10) collision_multirelaxation(); */
/*   if  (ndist == 2 && is_propagation_ode() == 0) collision_binary_lb(); */
/*   if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_BGK) collision_bgk(); */

/*   return; */
/* } */

void collide_gpu(void) {

  int ndist,nhalo;
  double mobility;
  int N[3];

  ndist = distribution_ndist();
  nhalo = coords_nhalo();
  coords_nlocal(N); 


  //collision_relaxation_times_set();
  //we need a duplicate GPU copy of this routine, as things are set up, 
  // to set the right copy of static (file scope) variables
  collision_relaxation_times_set_gpu();

  mobility = phi_cahn_hilliard_mobility();
  rtau2 = 2.0 / (1.0 + 2.0*mobility);


/*   if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_M10) collision_multirelaxation(); */
/*   if  (ndist == 2 && is_propagation_ode() == 0) collision_binary_lb(); */
/*   if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_BGK) collision_bgk(); */

/* copy constants to accelerator (constant on-chip read-only memory) */
  copy_constants_to_gpu();
  
  /* set up CUDA grid */
  /* 1D decomposition - use x grid and block dimension only */ 
  int nblocks=(N[X]*N[Y]*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;

  if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_M10)

    {
      
      printf("a\n");
    }

  if  (ndist == 2 && is_propagation_ode() == 0) 
    { 

      collision_binary_lb_gpu_d<<<nblocks,DEFAULT_TPB>>>(ndist, nhalo, N_d, 					      force_global_d, 
					      f_d, 
					      site_map_status_d, 
					       phi_site_d,		
					       grad_phi_site_d,	
					       delsq_phi_site_d,	
					      force_d, 
    			       		      velocity_d, 
					      ma_d, 
					      d_d, 
					      mi_d, 
					      cv_d, 
					       q_d, 
					       wv_d);
   

    }

  if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_BGK)
    {
      printf("c\n");


    }

    hipDeviceSynchronize();

  return;
}



__global__ void collision_binary_lb_gpu_d(int ndist, int nhalo, int N[3], 
					  double* force_global_d, 
					  double* f_d, 
					  char* site_map_status_d, 
					  double* phi_site_d,		
					  double* grad_phi_site_d,	
					  double* delsq_phi_site_d,	
					  double* force_ptr, 
					  double* velocity_ptr, 
					  double* ma_ptr, 
					  double* d_ptr, 
					  double* mi_ptr, 
					  int* cv_ptr, 
					  double* q_ptr, 
					  double* wv_d) 
{

  int       index;                   /* site indices */
  int       p, m;                    /* velocity index */
  int       i, j;                    /* summed over indices ("alphabeta") */

  double    mode[NVEL];              /* Modes; hydrodynamic + ghost */
  double    rho, rrho;               /* Density, reciprocal density */
  double    u[3];                    /* Velocity */
  double    s[3][3];                 /* Stress */
  double    seq[3][3];               /* equilibrium stress */
  double    shat[3][3];              /* random stress */
  double    ghat[NVEL];              /* noise for ghosts */

  double    force[3];                /* External force */
  double    tr_s, tr_seq;

  double    force_local[3];

  double f_loc[2*NVEL]; /* thread local copy of f_ data */

  const double   r3     = (1.0/3.0);

  double    phi, jdotc, sphidotq;    /* modes */
  double    jphi[3];
  double    sth[3][3], sphi[3][3];
  double    mu;                      /* Chemical potential */
  const double r2rcs4 = 4.5;         /* The constant 1 / 2 c_s^4 */

 /* cast dummy gpu memory pointers to pointers of right type (for 
   * multidimensional arrays) */
  double (*force_d)[3] = (double (*)[3]) force_ptr;
  double (*velocity_d)[3] = (double (*)[3]) velocity_ptr;
  double (*ma_d)[NVEL] = (double (*)[NVEL]) ma_ptr;
  double (*mi_d)[NVEL] = (double (*)[NVEL]) mi_ptr;
  double (*d_d)[3] = (double (*)[3]) d_ptr;
  int (*cv_d)[3] = (int (*)[3]) cv_ptr;
  double (*q_d)[3][3] = (double (*)[3][3]) q_ptr;

  int threadIndex, nsite, Nall[3], ii, jj, kk, xfac, yfac;

  /* ndist is always 2 in this routine. Use of hash define may help compiler */
#define NDIST 2

  //chemical_potential = fe_chemical_potential_function();
  // chemical_stress = fe_chemical_stress_function();


   fluctuations_off_gpu_d(shat, ghat); 

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsite = Nall[X]*Nall[Y]*Nall[Z];

  /* CUDA thread index */
  threadIndex = blockIdx.x*blockDim.x+threadIdx.x;

  /* Avoid going beyond problem domain */
  if (threadIndex < N[X]*N[Y]*N[Z])
    {
      
      /* calculate index from CUDA thread index */
      yfac = N[Z];
      xfac = N[Y]*yfac;
      
      ii = threadIndex/xfac;
      jj = ((threadIndex-xfac*ii)/yfac);
      kk = (threadIndex-ii*xfac-jj*yfac);
      
      index = get_linear_index_gpu_d(ii+1,jj+1,kk+1,Nall);
      
/*       if (site_map_status_d[index] == FLUID) */
/* 	{ */
	  
	  
	  /* load data into registers */
	  for(p = 0; p < NVEL; p++) {
	    for(m = 0; m < NDIST; m++) {
	      f_loc[NVEL*m+p] = f_d[nsite*NDIST*p + nsite*m + index];
	    }
	  }
	  
	  
	  /* Compute all the modes */
	  for (m = 0; m < NVEL; m++) {
	    double mode_tmp = 0.0;
	    for (p = 0; p < NVEL; p++) {
	      mode_tmp += f_loc[p]*ma_d[m][p];
	    }
	    mode[m] = mode_tmp;
	  }
	  
	  /* For convenience, write out the physical modes. */
	  
	  rho = mode[0];
	  for (i = 0; i < 3; i++) {
	    u[i] = mode[1 + i];
	  }
	  s[X][X] = mode[4];
	  s[X][Y] = mode[5];
	  s[X][Z] = mode[6];
	  s[Y][X] = s[X][Y];
	  s[Y][Y] = mode[7];
	  s[Y][Z] = mode[8];
	  s[Z][X] = s[X][Z];
	  s[Z][Y] = s[Y][Z];
	  s[Z][Z] = mode[9];
	  
	  /* Compute the local velocity, taking account of any body force */
	  
	  rrho = 1.0/rho;
	  /* hydrodynamics_get_force_local(index, force_local); */
	  for (i = 0; i < 3; i++) {
	    force_local[i] = force_d[index][i];
	  }
	  
	  for (i = 0; i < 3; i++) {
	    force[i] = (force_global_d[i] + force_local[i]);
	    u[i] = rrho*(u[i] + 0.5*force[i]);
	  }
	  /* hydrodynamics_set_velocity(index, u); */
	  for (i = 0; i < 3; i++) {
	    velocity_d[index][i] = u[i];
	  }
	  
	  /* Compute the thermodynamic component of the stress */
	  
	  symmetric_chemical_stress_gpu_d(index, sth, phi_site_d,
					  grad_phi_site_d,
					  delsq_phi_site_d,d_d);

	  /* Relax stress with different shear and bulk viscosity */
	  
	  tr_s   = 0.0;
	  tr_seq = 0.0;
	  
	  for (i = 0; i < 3; i++) {
	    /* Set equilibrium stress, which includes thermodynamic part */
	    for (j = 0; j < 3; j++) {
	      seq[i][j] = rho*u[i]*u[j] + sth[i][j];
	    }
	  /* Compute trace */
	    tr_s   += s[i][i];
	    tr_seq += seq[i][i];
	  }
	  
	  /* Form traceless parts */
	  for (i = 0; i < 3; i++) {
	    s[i][i]   -= r3*tr_s;
	    seq[i][i] -= r3*tr_seq;
	  }
	  
	  /* Relax each mode */
	  tr_s = tr_s - rtau_bulk_d*(tr_s - tr_seq);
	  
	  for (i = 0; i < 3; i++) {
	    for (j = 0; j < 3; j++) {
	      s[i][j] -= rtau_shear_d*(s[i][j] - seq[i][j]);
	      s[i][j] += d_d[i][j]*r3*tr_s;
	      
	      /* Correction from body force (assumes equal relaxation times) */
	      
	      s[i][j] += (2.0-rtau_shear_d)*(u[i]*force[j] + force[i]*u[j]);
	      shat[i][j] = 0.0;
	    }
	  }
	  
	  //if (isothermal_fluctuations_) fluctuations_on(shat, ghat);
	  
	  /* Now reset the hydrodynamic modes to post-collision values */
	  
	  mode[1] = mode[1] + force[X];    /* Conserved if no force */
	  mode[2] = mode[2] + force[Y];    /* Conserved if no force */
	  mode[3] = mode[3] + force[Z];    /* Conserved if no force */
	  mode[4] = s[X][X] + shat[X][X];
	  mode[5] = s[X][Y] + shat[X][Y];
	  mode[6] = s[X][Z] + shat[X][Z];
	  mode[7] = s[Y][Y] + shat[Y][Y];
	  mode[8] = s[Y][Z] + shat[Y][Z];
	  mode[9] = s[Z][Z] + shat[Z][Z];
	  
	  /* Ghost modes are relaxed toward zero equilibrium. */
	  
	  for (m = NHYDRO; m < NVEL; m++) {
	    mode[m] = mode[m] - rtau_d[m]*(mode[m] - 0.0) + ghat[m];
	  }
	  
	  /* Project post-collision modes back onto the distribution */

	  /* the below syncthreads is required, otherwise the above
	     summation goes wrong. This is NOT UNDERSTOOD yet and under
	     investigation - Alan Gray */
	  __syncthreads();

  	  double f_tmp;
	  
 	  for (p = 0; p < NVEL; p++) {
 	    f_tmp = 0.0;
 	    for (m = 0; m < NVEL; m++) {
 	      f_tmp += mi_d[p][m]*mode[m];
 	    }
	    f_d[nsite*NDIST*p + index] = f_tmp;
 	}

	/* Now, the order parameter distribution */

	phi = phi_site_d[index];
	mu = symmetric_chemical_potential_gpu_d(index, phi_site_d,
						delsq_phi_site_d);
	
	jphi[X] = 0.0;
	jphi[Y] = 0.0;
	jphi[Z] = 0.0;
	for (p = 1; p < NVEL; p++) {
	  for (i = 0; i < 3; i++) {
	    jphi[i] += f_loc[NVEL + p]*cv_d[p][i];
	  }
	}
	
	/* Relax order parameters modes. See the comments above. */
	
	for (i = 0; i < 3; i++) {
	  for (j = 0; j < 3; j++) {
	    sphi[i][j] = phi*u[i]*u[j] + mu*d_d[i][j];
	  }
	  jphi[i] = jphi[i] - rtau2_d*(jphi[i] - phi*u[i]);
	}
	
	/* Now update the distribution */
	
	for (p = 0; p < NVEL; p++) {
	  
	  int dp0 = (p == 0);
	  jdotc    = 0.0;
	  sphidotq = 0.0;
	  
	  for (i = 0; i < 3; i++) {
	    jdotc += jphi[i]*cv_d[p][i];
	    for (j = 0; j < 3; j++) {
	      sphidotq += sphi[i][j]*q_d[p][i][j];
	    }
	  }
	  
	  /* Project all this back to the distributions. The magic
	   * here is to move phi into the non-propagating distribution. */
	  
	  f_d[nsite*NDIST*p+nsite+index]
	    = wv_d[p]*(jdotc*rcs2_d + sphidotq*r2rcs4) + phi*dp0;
	  
	}
	
	
/* 	} */
      
    }
  
  return;
}

/*****************************************************************************
 *
 *  collision_relaxation_times_set
 *
 *  Note there is an extra normalisation in the lattice fluctuations
 *  which would otherwise give effective kT = cs2
 *
 *****************************************************************************/

void collision_relaxation_times_set_gpu(void) {

  int p;
  double kt;
  double tau_s;
  double tau_b;
  double tau_g;

  extern int is_propagation_ode(void);
 
  if (is_propagation_ode()) {
    rtau_shear = 1.0 / (3.0*get_eta_shear());
    rtau_bulk  = 1.0 / (3.0*get_eta_bulk());
  }
  else {
    rtau_shear = 2.0 / (1.0 + 6.0*get_eta_shear());
    rtau_bulk  = 2.0 / (1.0 + 6.0*get_eta_bulk());
  }

  /* Initialise the relaxation times */

  if (nrelax_ == RELAXATION_M10) {
    for (p = NHYDRO; p < NVEL; p++) {
      rtau_[p] = 1.0;
    }
  }

  if (nrelax_ == RELAXATION_BGK) {
    for (p = 0; p < NVEL; p++) {
      rtau_[p] = rtau_shear;
    }
  }

  if (nrelax_ == RELAXATION_TRT) {

    assert(NVEL != 9);

    tau_g = 2.0/(1.0 + (3.0/8.0)*rtau_shear);

    if (NVEL == 15) {
      rtau_[10] = rtau_shear;
      rtau_[11] = tau_g;
      rtau_[12] = tau_g;
      rtau_[13] = tau_g;
      rtau_[14] = rtau_shear;
    }

    if (NVEL == 19) {
      rtau_[10] = rtau_shear;
      rtau_[14] = rtau_shear;
      rtau_[18] = rtau_shear;

      rtau_[11] = tau_g;
      rtau_[12] = tau_g;
      rtau_[13] = tau_g;
      rtau_[15] = tau_g;
      rtau_[16] = tau_g;
      rtau_[17] = tau_g;
    }
  }

  if (isothermal_fluctuations_) {

    tau_s = 1.0/rtau_shear;
    tau_b = 1.0/rtau_bulk;

    /* Initialise the stress variances */

    kt = fluid_kt();
    kt = kt*rcs2; /* Without normalisation kT = cs^2 */

    var_bulk =
      sqrt(kt)*sqrt(2.0/9.0)*sqrt((tau_b + tau_b - 1.0)/(tau_b*tau_b));
    var_shear =
      sqrt(kt)*sqrt(1.0/9.0)*sqrt((tau_s + tau_s - 1.0)/(tau_s*tau_s));

    /* Noise variances */

    for (p = NHYDRO; p < NVEL; p++) {
      tau_g = 1.0/rtau_[p];
      noise_var[p] =
	sqrt(kt/norm_[p])*sqrt((tau_g + tau_g - 1.0)/(tau_g*tau_g));
    }
  }

  return;
}



/*****************************************************************************
 *
 *  copy_constants_to_gpu
 *
 *  copy constants to accelerator (constant on-chip read-only memory)
 *
 *****************************************************************************/

  void   copy_constants_to_gpu(){

    double a_,b_,kappa_;

    RUN_get_double_parameter("A", &a_);
    RUN_get_double_parameter("B", &b_);
    RUN_get_double_parameter("K", &kappa_);

   /* copy constant values to accelerator (on-chip read-only memory) */
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_shear_d), &rtau_shear, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_bulk_d), &rtau_bulk, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_d), rtau_, NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rtau2_d), &rtau2, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rcs2_d), &rcs2, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(a_d), &a_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(b_d), &b_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kappa_d), &kappa_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
  }


/*****************************************************************************
 *
 *  fluctuations_off_gpu_d
 *
 *  Return zero fluctuations for stress (shat) and ghost (ghat) modes.
 *
 *****************************************************************************/

__device__ void fluctuations_off_gpu_d(double shat[3][3], double ghat[NVEL]) {

  int ia, ib;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      shat[ia][ib] = 0.0;
    }
  }

  for (ia = NHYDRO; ia < NVEL; ia++) {
    ghat[ia] = 0.0;
  }

  return;
}



/****************************************************************************
 *
 *  symmetric_chemical_stress
 *
 *  Return the chemical stress tensor for given position index.
 *
 *  P_ab = [1/2 A phi^2 + 3/4 B phi^4 - kappa phi \nabla^2 phi
 *       -  1/2 kappa (\nbla phi)^2] \delta_ab
 *       +  kappa \nalba_a phi \nabla_b phi
 *
 ****************************************************************************/
__device__ void symmetric_chemical_stress_gpu_d(const int index, 
						double s[3][3],
						double *phi_site_d, 
						double *grad_phi_site_d, 
						double *delsq_phi_site_d,
						double d_d[3][3]) {

  int ia, ib;
  double phi;
  double delsq_phi;
  double grad_phi[3];
  double p0;

  phi = phi_site_d[index];
  delsq_phi = delsq_phi_site_d[index];
  for (ia = 0; ia < 3; ia++) grad_phi[ia]=grad_phi_site_d[3*index+ia];

  p0 = 0.5*a_d*phi*phi + 0.75*b_d*phi*phi*phi*phi
    - kappa_d*phi*delsq_phi - 
    0.5*kappa_d*dot_product_gpu_d(grad_phi, grad_phi);

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      s[ia][ib] = p0*d_d[ia][ib]	+ kappa_d*grad_phi[ia]*grad_phi[ib];
    }
  }

  return;
}


/****************************************************************************
 *
 *  symmetric_chemical_potential
 *
 *  The chemical potential \mu = \delta F / \delta \phi
 *                             = a\phi + b\phi^3 - \kappa\nabla^2 \phi
 *
 ****************************************************************************/

__device__ double symmetric_chemical_potential_gpu_d(const int index, 
		double *phi_site_d, double *delsq_phi_site_d) {

  double mu, phi;

  phi = phi_site_d[index];
  mu = a_d*phi + b_d*phi*phi*phi - kappa_d*delsq_phi_site_d[index];

  return mu;
}


/*****************************************************************************
 *
 *  dot_product
 *
 *****************************************************************************/

__device__ double dot_product_gpu_d(const double a[3], const double b[3]) {

	return (a[X]*b[X] + a[Y]*b[Y] + a[Z]*b[Z]);
}


/* get linear index from 3d coordinates */
 __device__ static int get_linear_index_gpu_d(int ii,int jj,int kk,int N[3])
{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}
