#include "hip/hip_runtime.h"
 /*****************************************************************************
 *
 *  collision_gpu.c
 *
 *  GPU implementation of collision functionality
 * 
 *  Alan Gray
 * 
 *****************************************************************************/

#include <assert.h>
#include <stdio.h>
#include <math.h>


/* below define needed to stop repeated declaration of distribution_ndist */
#define INCLUDING_FROM_GPU_SOURCE 

#include "model.h"
#include "site_map.h"
#include "comms_gpu.h"
#include "collision_gpu.h"
#include "collision_internal_gpu.h"
#include "collision.h"
#include "fluctuations.h"

//#include "free_energy.h"
//#include "phi_cahn_hilliard.h"

#include "control.h"
#include "propagation_ode.h"

static int nrelax_ = RELAXATION_M10;     /* [RELAXATION_M10|TRT|BGK] */
                                         /* Default is M10 */
static int isothermal_fluctuations_ = 0; /* Flag for noise. */

static double rtau_shear;       /* Inverse relaxation time for shear modes */
static double rtau_bulk;        /* Inverse relaxation time for bulk modes */
static double var_shear;        /* Variance for shear mode fluctuations */
static double var_bulk;         /* Variance for bulk mode fluctuations */
static double rtau_[NVEL];      /* Inverse relaxation times */
static double noise_var[NVEL];  /* Noise variances */

static double rtau2;

extern const double d_[3][3];

/* handles for CUDA streams (for ovelapping)*/
static hipStream_t streamCOLL, streamX, streamY, streamZ;




void collide_gpu(int async=0) {

  int ndist,nhalo;
  double mobility;
  int N[3];

  int Nall[3];

  ndist = distribution_ndist();
  nhalo = coords_nhalo();
  coords_nlocal(N); 

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  collision_relaxation_times_set_gpu();

  mobility = phi_cahn_hilliard_mobility();
  rtau2 = 2.0 / (1.0 + 2.0*mobility);


  /* copy f to ftmp on accelerator */
  //copy_f_to_ftmp_on_gpu();
  
  double *tmpptr=ftmp_d;
  ftmp_d=f_d;
  f_d=tmpptr;


/* copy constants to accelerator (constant on-chip read-only memory) */
  copy_constants_to_gpu();

  hipStreamCreate(&streamCOLL);
  
  int colltype;

  int nblocks;
  /* set up CUDA grid */
  /* 1D decomposition - use x grid and block dimension only */ 
  //int nblocks=(N[X]*N[Y]*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;

  if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_M10)

    {

      colltype=MULTIRELAXATION;
    }
      
  else if  (ndist == 2 && is_propagation_ode() == 0) 
    { 

      colltype=BINARY;
    }

  //if ((ndist == 1 || is_propagation_ode() == 1 ) && nrelax_ == RELAXATION_BGK)
  //  {
  //    printf("Error, KGK collision not supported yet in GPU version\n");
  //    exit(1);

  //   }


  else
    {
      printf("Error, the requested collision is not supported yet in GPU version\n");
      exit(1);

    }




  streamX=getXstream();
  streamY=getYstream();
  streamZ=getZstream();
      


  /* Bulk */
  nblocks=((N[X]-2*nhalo)*(N[Y]-2*nhalo)*(N[Z]-2*nhalo)+DEFAULT_TPB-1)/DEFAULT_TPB;

  collision_lb_gpu_d<<<nblocks,DEFAULT_TPB,0,streamCOLL>>>(ndist, nhalo, N_d, 					      force_global_d,
  					      f_d, ftmp_d,
  					      site_map_status_d,
  					       phi_site_d,
  					       grad_phi_site_d,
  					       delsq_phi_site_d,
  							 force_d,
							   velocity_d,colltype, BULK);


 /* X edges */
 nblocks=(nhalo*N[Y]*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
 collision_edge_gpu_d<<<nblocks,DEFAULT_TPB,0,streamX>>>(nhalo,
 						       N_d,force_global_d,
 					      f_d, ftmp_d,
 					      site_map_status_d,
 					       phi_site_d,
 					       grad_phi_site_d,
 					       delsq_phi_site_d,
 							 force_d,
							 velocity_d,colltype, X);

 /* Y edges */
  nblocks=((N[X]-2*nhalo)*nhalo*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
  collision_edge_gpu_d<<<nblocks,DEFAULT_TPB,0,streamY>>>(nhalo,
 						       N_d,force_global_d,
 					      f_d, ftmp_d,
 					      site_map_status_d,
 					       phi_site_d,
 					       grad_phi_site_d,
 					       delsq_phi_site_d,
 							 force_d,
							  velocity_d,colltype, Y);

 /* Z edges */
  nblocks=((N[X]-2*nhalo)*(N[Y]-2*nhalo)*nhalo+DEFAULT_TPB-1)/DEFAULT_TPB;
  collision_edge_gpu_d<<<nblocks,DEFAULT_TPB,0,streamZ>>>(nhalo,
 						       N_d,force_global_d,
 					      f_d, ftmp_d,
 					      site_map_status_d,
 					       phi_site_d,
 					       grad_phi_site_d,
 					       delsq_phi_site_d,
 							 force_d,
							  velocity_d,colltype, Z);


  //KEEP THESE SYNC POINTS IN JUST NOW UNTL SURE RACE-FREE RE. CORNERS
  hipStreamSynchronize(streamX);
  hipStreamSynchronize(streamY);
  hipStreamSynchronize(streamZ);

  //if (async==1)
  //  hipStreamSynchronize(streamCOLL);



  if (async==0){
    //hipStreamSynchronize(streamX);
    //hipStreamSynchronize(streamY);
    //hipStreamSynchronize(streamZ);
  hipStreamSynchronize(streamCOLL);

  hipStreamDestroy(streamCOLL);
  }

  return;
}

void collide_wait_gpu()
{
   hipStreamSynchronize(streamCOLL);
   hipStreamDestroy(streamCOLL);

   return;
}

__device__ void collision_multirelaxation_site_gpu_d(
					      const double* __restrict__ force_global_d, 
					      double* __restrict__ f_d, 
					      const double* __restrict__ ftmp_d, 

					      const char* __restrict__ site_map_status_d, 
					      const double* __restrict__ force_d, 
    			       		      double* __restrict__ velocity_d, const int nsite, const int index
					      )
{

  int       p, m;                    /* velocity index */
  int       ia, ib;                  /* indices ("alphabeta") */

  double    mode[NVEL];              /* Modes; hydrodynamic + ghost */
  double    rho, rrho;               /* Density, reciprocal density */
  double    u[3];                    /* Velocity */
  double    s[3][3];                 /* Stress */
  double    seq[3][3];               /* Equilibrium stress */
  double    shat[3][3];              /* random stress */
  double    ghat[NVEL];              /* noise for ghosts */
  double    rdim;                    /* 1 / dimension */

  double    force[3];                /* External force */
  double    tr_s, tr_seq;

  double    force_local[3];

  /* int threadIndex, nsite, Nall[3], ii, jj, kk, xfac, yfac; */

  
	  
	  
	  /* Compute all the modes */
	  
	  for (m = 0; m < NVEL; m++) {
	   double dtmp = 0.;
	  for (p = 0; p < NVEL; p++) {
	    dtmp += ftmp_d[nsite*p + index]*ma_cd[m][p];
	  }
	  mode[m] = dtmp;
	  }
	  

	  /* For convenience, write out the physical modes, that is,
	   * rho, NDIM components of velocity, independent components
	   * of stress (upper triangle), and lower triangle. */
	  
	  rho = mode[0];
	  for (ia = 0; ia < NDIM; ia++) {
	    u[ia] = mode[1 + ia];
	  }
	  
	  
	  m = 0;
	  for (ia = 0; ia < NDIM; ia++) {
	    for (ib = ia; ib < NDIM; ib++) {
	      s[ia][ib] = mode[1 + NDIM + m++];
	    }
	  }
	  
	  for (ia = 1; ia < NDIM; ia++) {
	    for (ib = 0; ib < ia; ib++) {
	      s[ia][ib] = s[ib][ia];
	    }
	  }
	  
	  
	  
	  /* Compute the local velocity, taking account of any body force */
	  
	  rrho = 1.0/rho;
	  /* hydrodynamics_get_force_local(index, force_local); */
	  for (ia = 0; ia < 3; ia++) {
	    force_local[ia] = force_d[ia*nsite+index];
	  }
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    force[ia] = (force_global_d[ia] + force_local[ia]);
	    u[ia] = rrho*(u[ia] + 0.5*force[ia]);
	  }
	  
	  /* hydrodynamics_set_velocity(index, u); */
	  for (ia = 0; ia < 3; ia++) {
	    velocity_d[ia*nsite+index] = u[ia];
	  }
	  
	  /* Relax stress with different shear and bulk viscosity */
	  
	  tr_s   = 0.0;
	  tr_seq = 0.0;
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    /* Set equilibrium stress */
	    for (ib = 0; ib < NDIM; ib++) {
	      seq[ia][ib] = rho*u[ia]*u[ib];
	    }
	    /* Compute trace */
	    tr_s   += s[ia][ia];
	    tr_seq += seq[ia][ia];
	  }
	  
	  /* Form traceless parts */
	  for (ia = 0; ia < NDIM; ia++) {
	    s[ia][ia]   -= rdim*tr_s;
	    seq[ia][ia] -= rdim*tr_seq;
	  }
	  
	  /* Relax each mode */
	  tr_s = tr_s - rtau_bulk_d*(tr_s - tr_seq);
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    for (ib = 0; ib < NDIM; ib++) {
	      s[ia][ib] -= rtau_shear_d*(s[ia][ib] - seq[ia][ib]);
	      s[ia][ib] += d_cd[ia][ib]*rdim*tr_s;
	      
	      /* Correction from body force (assumes equal relaxation times) */
	      
	      s[ia][ib] += (2.0-rtau_shear_d)*(u[ia]*force[ib] + force[ia]*u[ib]);
	    }
	  }
	  
	  //if (isothermal_fluctuations_) 	    collision_fluctuations(index, shat, ghat);
	  
	  /* Now reset the hydrodynamic modes to post-collision values:
	   * rho is unchanged, velocity unchanged if no force,
	   * independent components of stress, and ghosts. */
	  
	  for (ia = 0; ia < NDIM; ia++) {
	    mode[1 + ia] += force[ia];
	  }

	  
	  m = 0;
	  for (ia = 0; ia < NDIM; ia++) {
	    for (ib = ia; ib < NDIM; ib++) {
	      mode[1 + NDIM + m++] = s[ia][ib] + shat[ia][ib];
	    }
	  }
	  
	  /* Ghost modes are relaxed toward zero equilibrium. */
	  
	  for (m = NHYDRO; m < NVEL; m++) {
	    mode[m] = mode[m] - rtau_d[m]*(mode[m] - 0.0) + ghat[m];
	  }
	  
	  /* Project post-collision modes back onto the distribution */
	  
	  for (p = 0; p < NVEL; p++) {
	   double dtmp = 0.;
	  for (m = 0; m < NVEL; m++) {
	    dtmp += mi_cd[p][m]*mode[m];
	  }
	  f_d[nsite*p + index] = dtmp;
	  }
  
  
  return;
}



__device__ void collision_binary_lb_site_gpu_d(const double* __restrict__ force_global_d, 
					  double* __restrict__ f_d, 
					  const double* __restrict__ ftmp_d, 
					  const char* __restrict__ site_map_status_d, 
					  const double* __restrict__ phi_site_d,		
					  const double* __restrict__ grad_phi_site_d,	
					  const double* __restrict__ delsq_phi_site_d,	
					  const double* __restrict__ force_d, 
					       double* __restrict__ velocity_d, const int nsite, const int index) 
{
  int       p, m;                    /* velocity index */
  int       i, j;                    /* summed over indices ("alphabeta") */

  double    mode[NVEL];              /* Modes; hydrodynamic + ghost */
  double    rho, rrho;               /* Density, reciprocal density */
  double    u[3];                    /* Velocity */
  double    s[3][3];                 /* Stress */
  double    seq[3][3];               /* equilibrium stress */
  double    shat[3][3];              /* random stress */
  double    ghat[NVEL];              /* noise for ghosts */

  double    force[3];                /* External force */
  double    tr_s, tr_seq;

  double    force_local[3];

  double f_loc[2*NVEL]; /* thread local copy of f_ data */

  const double   r3     = (1.0/3.0);

  double    phi, jdotc, sphidotq;    /* modes */
  double    jphi[3];
  double    sth[3][3], sphi[3][3];
  double    mu;                      /* Chemical potential */
  const double r2rcs4 = 4.5;         /* The constant 1 / 2 c_s^4 */


  /* ndist is always 2 in this routine. Use of hash define may help compiler */
#define NDIST 2



  /* load data into registers */
  for(p = 0; p < NVEL; p++) {
    for(m = 0; m < NDIST; m++) {
      f_loc[NVEL*m+p] = ftmp_d[nsite*NDIST*p + nsite*m + index];
    }
  }
  
  
  /* Compute all the modes */
  for (m = 0; m < NVEL; m++) {
    double mode_tmp = 0.0;
    for (p = 0; p < NVEL; p++) {
      mode_tmp += f_loc[p]*ma_cd[m][p];
    }
    mode[m] = mode_tmp;
  }
  
  
  /* For convenience, write out the physical modes. */
  
  rho = mode[0];
  for (i = 0; i < 3; i++) {
    u[i] = mode[1 + i];
  }
  s[X][X] = mode[4];
  s[X][Y] = mode[5];
  s[X][Z] = mode[6];
  s[Y][X] = s[X][Y];
  s[Y][Y] = mode[7];
  s[Y][Z] = mode[8];
  s[Z][X] = s[X][Z];
  s[Z][Y] = s[Y][Z];
  s[Z][Z] = mode[9];
  
  /* Compute the local velocity, taking account of any body force */
  
  rrho = 1.0/rho;
  /* hydrodynamics_get_force_local(index, force_local); */
  for (i = 0; i < 3; i++) {
    force_local[i] = force_d[i*nsite+index];
  }
  
  for (i = 0; i < 3; i++) {
    force[i] = (force_global_d[i] + force_local[i]);
    u[i] = rrho*(u[i] + 0.5*force[i]);
  }
  /* hydrodynamics_set_velocity(index, u); */
  for (i = 0; i < 3; i++) {
    velocity_d[i*nsite+index] = u[i];
  }
  
  /* Compute the thermodynamic component of the stress */
  
  symmetric_chemical_stress_gpu_d(index, sth, phi_site_d,
				  grad_phi_site_d,
				  delsq_phi_site_d,nsite);
  
  /* Relax stress with different shear and bulk viscosity */
  
  tr_s   = 0.0;
  tr_seq = 0.0;
  
  for (i = 0; i < 3; i++) {
    /* Set equilibrium stress, which includes thermodynamic part */
    for (j = 0; j < 3; j++) {
      seq[i][j] = rho*u[i]*u[j] + sth[i][j];
    }
    /* Compute trace */
    tr_s   += s[i][i];
    tr_seq += seq[i][i];
  }
  
  /* Form traceless parts */
  for (i = 0; i < 3; i++) {
    s[i][i]   -= r3*tr_s;
    seq[i][i] -= r3*tr_seq;
  }
  
  /* Relax each mode */
  tr_s = tr_s - rtau_bulk_d*(tr_s - tr_seq);
  
  for (i = 0; i < 3; i++) {
    for (j = 0; j < 3; j++) {
      s[i][j] -= rtau_shear_d*(s[i][j] - seq[i][j]);
      s[i][j] += d_cd[i][j]*r3*tr_s;
      
      /* Correction from body force (assumes equal relaxation times) */
      
      s[i][j] += (2.0-rtau_shear_d)*(u[i]*force[j] + force[i]*u[j]);
      shat[i][j] = 0.0;
    }
  }
  
  //if (isothermal_fluctuations_) fluctuations_on(shat, ghat);
  
  /* Now reset the hydrodynamic modes to post-collision values */
  
  mode[1] = mode[1] + force[X];    /* Conserved if no force */
  mode[2] = mode[2] + force[Y];    /* Conserved if no force */
  mode[3] = mode[3] + force[Z];    /* Conserved if no force */
  mode[4] = s[X][X] + shat[X][X];
  mode[5] = s[X][Y] + shat[X][Y];
  mode[6] = s[X][Z] + shat[X][Z];
  mode[7] = s[Y][Y] + shat[Y][Y];
  mode[8] = s[Y][Z] + shat[Y][Z];
  mode[9] = s[Z][Z] + shat[Z][Z];
  
  /* Ghost modes are relaxed toward zero equilibrium. */
  
  for (m = NHYDRO; m < NVEL; m++) {
    mode[m] = mode[m] - rtau_d[m]*(mode[m] - 0.0) + ghat[m];
  }
  
  /* Project post-collision modes back onto the distribution */
  
  
  double f_tmp;
  
  for (p = 0; p < NVEL; p++) {
    f_tmp = 0.0;
    for (m = 0; m < NVEL; m++) {
      f_tmp += mi_cd[p][m]*mode[m];
    }
    f_d[nsite*NDIST*p + index] = f_tmp;
  }
  
  /* Now, the order parameter distribution */
  
  phi = phi_site_d[index];
  mu = symmetric_chemical_potential_gpu_d(index, phi_site_d,
					  delsq_phi_site_d);
  
  jphi[X] = 0.0;
  jphi[Y] = 0.0;
  jphi[Z] = 0.0;
  for (p = 1; p < NVEL; p++) {
    for (i = 0; i < 3; i++) {
      jphi[i] += f_loc[NVEL + p]*cv_cd[p][i];
    }
  }
  
  /* Relax order parameters modes. See the comments above. */
  
  for (i = 0; i < 3; i++) {
    for (j = 0; j < 3; j++) {
      sphi[i][j] = phi*u[i]*u[j] + mu*d_cd[i][j];
    }
    jphi[i] = jphi[i] - rtau2_d*(jphi[i] - phi*u[i]);
  }
  
  /* Now update the distribution */
	
  for (p = 0; p < NVEL; p++) {
    
    int dp0 = (p == 0);
    jdotc    = 0.0;
    sphidotq = 0.0;
    
    for (i = 0; i < 3; i++) {
      jdotc += jphi[i]*cv_cd[p][i];
      for (j = 0; j < 3; j++) {
	sphidotq += sphi[i][j]*q_cd[p][i][j];
      }
    }
    
    /* Project all this back to the distributions. The magic
     * here is to move phi into the non-propagating distribution. */
    
    f_d[nsite*NDIST*p+nsite+index]
      = wv_cd[p]*(jdotc*rcs2_d + sphidotq*r2rcs4) + phi*dp0;
    
  }
  
  


  return;
}


/* pack edges on the accelerator */
__global__ static void collision_edge_gpu_d(int nhalo, 
						   int N[3],
						   const double* __restrict__ force_global_d, 
						   double* __restrict__ f_d, 
					    const double* __restrict__ ftmp_d, 
						   const char* __restrict__ site_map_status_d, 
						   const double* __restrict__ phi_site_d,		
						   const double* __restrict__ grad_phi_site_d,	
						   const double* __restrict__ delsq_phi_site_d,	
						   const double* __restrict__ force_d, 
					    double* __restrict__ velocity_d,int colltype, int dirn)
{



  int Nall[3];
  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;
  int nsite = Nall[X]*Nall[Y]*Nall[Z];
 
  int Nedge[3];

  if (dirn == X){
    Nedge[X]=nhalo;
    Nedge[Y]=N[Y];
    Nedge[Z]=N[Z];
  }
  else if (dirn == Y){
    //Nedge[X]=Nall[X];
    Nedge[X]=N[X]-2*nhalo;
    Nedge[Y]=nhalo;
    Nedge[Z]=N[Z];
  }
  else if (dirn == Z){
    Nedge[X]=N[X]-2*nhalo;
    Nedge[Y]=N[Y]-2*nhalo;
    Nedge[Z]=nhalo;
  }


  int p,m, index,ii,jj,kk,ii_,jj_,kk_;
 
  int npackedsite = Nedge[X]*Nedge[Y]*Nedge[Z];
  

  int threadIndex = blockIdx.x*blockDim.x+threadIdx.x;
  
  if (threadIndex < npackedsite)
    {
      
      get_coords_from_index_gpu_d(&ii,&jj,&kk,threadIndex,Nedge);
      
      /* LOW EDGE */
      if (dirn == X){
	index = get_linear_index_gpu_d(ii+nhalo,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn == Y){
	index = get_linear_index_gpu_d(ii+2*nhalo,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn == Z){
	index = get_linear_index_gpu_d(ii+2*nhalo,jj+2*nhalo,kk+nhalo,Nall);
      }


      get_coords_from_index_gpu_d(&ii_,&jj_,&kk_,index,Nall);
      /* printf("low dir=%d %d %d %d\n",dirn,ii_,jj_,kk_); */

      if (colltype==BINARY){
      collision_binary_lb_site_gpu_d(force_global_d,
      					      f_d, ftmp_d,
      					      site_map_status_d,
      					       phi_site_d,
      					       grad_phi_site_d,
      					       delsq_phi_site_d,
      							 force_d,
      				     velocity_d, nsite, index);
	}
      else if (colltype==MULTIRELAXATION)
	{

	  
	  collision_multirelaxation_site_gpu_d(force_global_d, f_d, ftmp_d, 
					       site_map_status_d, 
					       force_d, velocity_d, nsite, 
					       index);

	}

 
      /* printf("%f\n",f_d[index]); */

      //if (jj_==1 && kk_==1)printf("X low %d %d %d %f\n",ii_,jj_,kk_,f_d[index]);


        
      /* HIGH EDGE */
      if (dirn == X){
	index = get_linear_index_gpu_d(Nall[X]-2*nhalo+ii,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn == Y){
	index = get_linear_index_gpu_d(ii+2*nhalo,Nall[Y]-2*nhalo+jj,kk+nhalo,Nall);
      }
      else if (dirn == Z){
	index = get_linear_index_gpu_d(ii+2*nhalo,jj+2*nhalo,Nall[Z]-2*nhalo+kk,Nall);
      }

      //printf("high dir=%d %d %d %d\n",dirn,ii,jj,kk);

      if (colltype==BINARY){
      collision_binary_lb_site_gpu_d(force_global_d,
      					      f_d, ftmp_d,
      					      site_map_status_d,
      					       phi_site_d,
      					       grad_phi_site_d,
      					       delsq_phi_site_d,
      							 force_d,
      				     velocity_d, nsite, index);
	}
      else if (colltype==MULTIRELAXATION)
	{

	  
	  collision_multirelaxation_site_gpu_d(force_global_d, f_d, ftmp_d, 
					       site_map_status_d, 
					       force_d, velocity_d, nsite, 
					       index);

	}


    }
  
  
}



__global__ void collision_lb_gpu_d(int ndist, int nhalo, int N[3], 
					  const double* __restrict__ force_global_d, 
					  double* __restrict__ f_d, 
					  const double* __restrict__ ftmp_d, 
					  const char* __restrict__ site_map_status_d, 
					  const double* __restrict__ phi_site_d,		
					  const double* __restrict__ grad_phi_site_d,	
					  const double* __restrict__ delsq_phi_site_d,	
					  const double* __restrict__ force_d, 
				   double* __restrict__ velocity_d, int colltype, int latchunk) 
{

  int       index;                   /* site indices */
  int       p, m;                    /* velocity index */
  int       i, j;                    /* summed over indices ("alphabeta") */

  double    mode[NVEL];              /* Modes; hydrodynamic + ghost */
  double    rho, rrho;               /* Density, reciprocal density */
  double    u[3];                    /* Velocity */
  double    s[3][3];                 /* Stress */
  double    seq[3][3];               /* equilibrium stress */
  double    shat[3][3];              /* random stress */
  double    ghat[NVEL];              /* noise for ghosts */

  double    force[3];                /* External force */
  double    tr_s, tr_seq;

  double    force_local[3];

  double f_loc[2*NVEL]; /* thread local copy of f_ data */

  const double   r3     = (1.0/3.0);

  double    phi, jdotc, sphidotq;    /* modes */
  double    jphi[3];
  double    sth[3][3], sphi[3][3];
  double    mu;                      /* Chemical potential */
  const double r2rcs4 = 4.5;         /* The constant 1 / 2 c_s^4 */


  int threadIndex, nsite, Nall[3], ii, jj, kk, xfac, yfac, ii_,jj_,kk_;

  /* ndist is always 2 in this routine. Use of hash define may help compiler */
#define NDIST 2


  fluctuations_off_gpu_d(shat, ghat); 

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsite = Nall[X]*Nall[Y]*Nall[Z];

  int N_[3];



  if (latchunk==BULK){
    N_[X]=N[X]-2*nhalo;  N_[Y]=N[Y]-2*nhalo;  N_[Z]=N[Z]-2*nhalo;    
  }
  else{
    N_[X]=N[X];  N_[Y]=N[Y];  N_[Z]=N[Z];
  }

  /* CUDA thread index */
  threadIndex = blockIdx.x*blockDim.x+threadIdx.x;

  /* Avoid going beyond problem domain */
  if (threadIndex < N_[X]*N_[Y]*N_[Z])
    {


	get_coords_from_index_gpu_d(&ii,&jj,&kk,threadIndex,N_);


      if (latchunk==BULK){
	index = get_linear_index_gpu_d(ii+2*nhalo,jj+2*nhalo,kk+2*nhalo,Nall);
      }
      else{
	index = get_linear_index_gpu_d(ii+nhalo,jj+nhalo,kk+nhalo,Nall);
      }


      if (colltype==BINARY){
      collision_binary_lb_site_gpu_d(force_global_d,
      					      f_d, ftmp_d,
      					      site_map_status_d,
      					       phi_site_d,
      					       grad_phi_site_d,
      					       delsq_phi_site_d,
      							 force_d,
      				     velocity_d, nsite, index);
      }
      else if (colltype==MULTIRELAXATION)
	{

	  
	  collision_multirelaxation_site_gpu_d(force_global_d, f_d, ftmp_d, 
					       site_map_status_d, 
					       force_d, velocity_d, nsite, 
					       index);

	}



      //      } 
      
    }
  
  return;
}

/*****************************************************************************
 *
 *  collision_relaxation_times_set
 *
 *  Note there is an extra normalisation in the lattice fluctuations
 *  which would otherwise give effective kT = cs2
 *
 *****************************************************************************/

void collision_relaxation_times_set_gpu(void) {

  int p;
  double kt;
  double tau_s;
  double tau_b;
  double tau_g;

  extern int is_propagation_ode(void);
 
  if (is_propagation_ode()) {
    rtau_shear = 1.0 / (3.0*get_eta_shear());
    rtau_bulk  = 1.0 / (3.0*get_eta_bulk());
  }
  else {
    rtau_shear = 2.0 / (1.0 + 6.0*get_eta_shear());
    rtau_bulk  = 2.0 / (1.0 + 6.0*get_eta_bulk());
  }

  /* Initialise the relaxation times */

  if (nrelax_ == RELAXATION_M10) {
    for (p = NHYDRO; p < NVEL; p++) {
      rtau_[p] = 1.0;
    }
  }

  if (nrelax_ == RELAXATION_BGK) {
    for (p = 0; p < NVEL; p++) {
      rtau_[p] = rtau_shear;
    }
  }

  if (nrelax_ == RELAXATION_TRT) {

    assert(NVEL != 9);

    tau_g = 2.0/(1.0 + (3.0/8.0)*rtau_shear);

    if (NVEL == 15) {
      rtau_[10] = rtau_shear;
      rtau_[11] = tau_g;
      rtau_[12] = tau_g;
      rtau_[13] = tau_g;
      rtau_[14] = rtau_shear;
    }

    if (NVEL == 19) {
      rtau_[10] = rtau_shear;
      rtau_[14] = rtau_shear;
      rtau_[18] = rtau_shear;

      rtau_[11] = tau_g;
      rtau_[12] = tau_g;
      rtau_[13] = tau_g;
      rtau_[15] = tau_g;
      rtau_[16] = tau_g;
      rtau_[17] = tau_g;
    }
  }

  if (isothermal_fluctuations_) {

    tau_s = 1.0/rtau_shear;
    tau_b = 1.0/rtau_bulk;

    /* Initialise the stress variances */

    kt = fluid_kt();
    kt = kt*rcs2; /* Without normalisation kT = cs^2 */

    var_bulk =
      sqrt(kt)*sqrt(2.0/9.0)*sqrt((tau_b + tau_b - 1.0)/(tau_b*tau_b));
    var_shear =
      sqrt(kt)*sqrt(1.0/9.0)*sqrt((tau_s + tau_s - 1.0)/(tau_s*tau_s));

    /* Noise variances */

    for (p = NHYDRO; p < NVEL; p++) {
      tau_g = 1.0/rtau_[p];
      noise_var[p] =
	sqrt(kt/norm_[p])*sqrt((tau_g + tau_g - 1.0)/(tau_g*tau_g));
    }
  }

  return;
}



/*****************************************************************************
 *
 *  copy_constants_to_gpu
 *
 *  copy constants to accelerator (constant on-chip read-only memory)
 *
 *****************************************************************************/

  void   copy_constants_to_gpu(){

    double a_,b_,kappa_;

    RUN_get_double_parameter("A", &a_);
    RUN_get_double_parameter("B", &b_);
    RUN_get_double_parameter("K", &kappa_);

   /* copy constant values to accelerator (on-chip read-only memory) */
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_shear_d), &rtau_shear, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_bulk_d), &rtau_bulk, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rtau_d), rtau_, NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(wv_cd), wv, NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);

    hipMemcpyToSymbol(HIP_SYMBOL(ma_cd), ma_, NVEL*NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(mi_cd), mi_, NVEL*NVEL*sizeof(double), 0,	
		       hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(d_cd), d_, 3*3*sizeof(double), 0,	
    		       hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(cv_cd), cv, NVEL*3*sizeof(int), 0,
			hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(q_cd), q_, NVEL*3*3*sizeof(double), 0,
			hipMemcpyHostToDevice);
     hipMemcpyToSymbol(HIP_SYMBOL(rtau2_d), &rtau2, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(rcs2_d), &rcs2, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(a_d), &a_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(b_d), &b_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kappa_d), &kappa_, sizeof(double), 0,	
		       hipMemcpyHostToDevice);
  }


/*****************************************************************************
 *
 *  fluctuations_off_gpu_d
 *
 *  Return zero fluctuations for stress (shat) and ghost (ghat) modes.
 *
 *****************************************************************************/

__device__ void fluctuations_off_gpu_d(double shat[3][3], double ghat[NVEL]) {

  int ia, ib;

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      shat[ia][ib] = 0.0;
    }
  }

  for (ia = NHYDRO; ia < NVEL; ia++) {
    ghat[ia] = 0.0;
  }

  return;
}



/****************************************************************************
 *
 *  symmetric_chemical_stress
 *
 *  Return the chemical stress tensor for given position index.
 *
 *  P_ab = [1/2 A phi^2 + 3/4 B phi^4 - kappa phi \nabla^2 phi
 *       -  1/2 kappa (\nbla phi)^2] \delta_ab
 *       +  kappa \nalba_a phi \nabla_b phi
 *
 ****************************************************************************/
__device__ void symmetric_chemical_stress_gpu_d(const int index, 
						double s[3][3],
						const double* __restrict__ phi_site_d, 
						const double* __restrict__ grad_phi_site_d, 
						const double* __restrict__ delsq_phi_site_d,
						int nsite) {

  int ia, ib;
  double phi;
  double delsq_phi;
  double grad_phi[3];
  double p0;

  phi = phi_site_d[index];
  delsq_phi = delsq_phi_site_d[index];
  for (ia = 0; ia < 3; ia++) grad_phi[ia]=grad_phi_site_d[ia*nsite+index];

  p0 = 0.5*a_d*phi*phi + 0.75*b_d*phi*phi*phi*phi
    - kappa_d*phi*delsq_phi - 
    0.5*kappa_d*dot_product_gpu_d(grad_phi, grad_phi);

  for (ia = 0; ia < 3; ia++) {
    for (ib = 0; ib < 3; ib++) {
      s[ia][ib] = p0*d_cd[ia][ib]	+ kappa_d*grad_phi[ia]*grad_phi[ib];
    }
  }

  return;
}


/****************************************************************************
 *
 *  symmetric_chemical_potential
 *
 *  The chemical potential \mu = \delta F / \delta \phi
 *                             = a\phi + b\phi^3 - \kappa\nabla^2 \phi
 *
 ****************************************************************************/

__device__ double symmetric_chemical_potential_gpu_d(const int index, 
		const double* __restrict__  phi_site_d, const double* __restrict__  delsq_phi_site_d) {

  double mu, phi;

  phi = phi_site_d[index];
  mu = a_d*phi + b_d*phi*phi*phi - kappa_d*delsq_phi_site_d[index];

  return mu;
}


/*****************************************************************************
 *
 *  dot_product
 *
 *****************************************************************************/

__device__ double dot_product_gpu_d(const double a[3], const double b[3]) {

	return (a[X]*b[X] + a[Y]*b[Y] + a[Z]*b[Z]);
}


/* get 3d coordinates from the index on the accelerator */
__device__ static void get_coords_from_index_gpu_d(int *ii,int *jj,int *kk,int index,int N[3])

{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;
  
  *ii = index/xfac;
  *jj = ((index-xfac*(*ii))/yfac);
  *kk = (index-(*ii)*xfac-(*jj)*yfac);

  return;

}


/* get linear index from 3d coordinates */
 __device__ static int get_linear_index_gpu_d(int ii,int jj,int kk,int N[3])
{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}
