#include "hip/hip_runtime.h"
/*****************************************************************************
 *
 * comms_gpu.cu
 * 
 * Alan Gray
 *
 * 
 *****************************************************************************/

#include <stdio.h>

#include "comms_gpu.h"
#include "comms_internal_gpu.h"
#include "utilities_gpu.h"
#include "common_gpu.h"
#include "model.h" 
extern "C" int  RUN_get_string_parameter(const char *, char *, const int);

/* external pointers to data on host*/
extern double * f_;
extern double * ftmp;
extern double * colloid_force_tmp;
extern double * velocity_d;

/* external pointers to data on accelerator*/
extern int * cv_d;
extern int * N_d;
extern double * f_d;
extern double * ftmp_d;

int *packedindex_d;
char *mask_d;
char *mask_;
char *mask_with_neighbours;


/* edge and halo buffers on accelerator */
static double * edgeXLOW_d;
static double * edgeXHIGH_d;
static double * edgeYLOW_d;
static double * edgeYHIGH_d;
static double * edgeZLOW_d;
static double * edgeZHIGH_d;
static double * haloXLOW_d;
static double * haloXHIGH_d;
static double * haloYLOW_d;
static double * haloYHIGH_d;
static double * haloZLOW_d;
static double * haloZHIGH_d;


/* edge and halo buffers on host */
static double * edgeXLOW;
static double * edgeXHIGH;
static double * edgeYLOW;
static double * edgeYHIGH;
static double * edgeZLOW;
static double * edgeZHIGH;
static double * haloXLOW;
static double * haloXHIGH;
static double * haloYLOW;
static double * haloYHIGH;
static double * haloZLOW;
static double * haloZHIGH;




static int * packedindex;


/* data size variables */
static int nhalo;
static int nsites;
static int ndist;
static int nop;
static  int N[3];
static  int Nall[3];
static int npvel; /* number of velocity components when packed */
static int nhalodataX;
static int nhalodataY;
static int nhalodataZ;

/* handles for CUDA streams (for ovelapping)*/
static hipStream_t streamX,streamY, streamZ;


static int reduced_halo=0;

/* constant memory symbols internal to this module */
__constant__ int cv_cd[NVEL][3];


/* Perform tasks necessary to initialise accelerator */
void init_comms_gpu()
{

  calculate_comms_data_sizes();
  allocate_comms_memory_on_gpu();


  char string[FILENAME_MAX];

  RUN_get_string_parameter("reduced_halo", string, FILENAME_MAX);
  if (strcmp(string, "yes") == 0) reduced_halo = 1;
  
  /* create CUDA streams (for ovelapping)*/
  hipStreamCreate(&streamX);
  hipStreamCreate(&streamY);
  hipStreamCreate(&streamZ);


  hipMemcpyToSymbol(HIP_SYMBOL(cv_cd), cv, NVEL*3*sizeof(int), 0, hipMemcpyHostToDevice); 
 
  //checkCUDAError("Init GPU");  


}

void finalise_comms_gpu()
{
  free_comms_memory_on_gpu();

  hipStreamDestroy(streamX);
  hipStreamDestroy(streamY);
  hipStreamDestroy(streamZ);

}


/* calculate sizes of data - needed for memory copies to accelerator */
static void calculate_comms_data_sizes()
{
  coords_nlocal(N);  
  nhalo = coords_nhalo();  
  ndist = distribution_ndist();
  nop = phi_nop();

  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsites = Nall[X]*Nall[Y]*Nall[Z];

  

  /* calculate number of velocity components when packed */
  int p;
  npvel=0;
  for (p=0; p<NVEL; p++)
    {
      if (cv[p][0] == 1 || !reduced_halo) npvel++; 
    }

  int n1=ndist*npvel;
  if (nop > n1) n1=nop;

  nhalodataX = N[Y] * N[Z] * nhalo * n1;
  nhalodataY = Nall[X] * N[Z] * nhalo * n1;
  nhalodataZ = Nall[X] * Nall[Y] * nhalo * n1;



}



/* Allocate memory on accelerator */
static void allocate_comms_memory_on_gpu()
{

  
  hipHostAlloc( (void **)&packedindex, nsites*sizeof(int), 
		 hipHostMallocDefault);

  hipHostAlloc( (void **)&mask_, nsites*sizeof(char), 
		 hipHostMallocDefault);

  hipHostAlloc( (void **)&mask_with_neighbours, nsites*sizeof(char), 
		 hipHostMallocDefault);

  hipHostAlloc( (void **)&edgeXLOW, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeXHIGH, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeYLOW, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeYHIGH, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeZLOW, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&edgeZHIGH, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);


  hipHostAlloc( (void **)&haloXLOW, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloXHIGH, nhalodataX*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloYLOW, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloYHIGH, nhalodataY*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloZLOW, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);
  hipHostAlloc( (void **)&haloZHIGH, nhalodataZ*sizeof(double), 
		 hipHostMallocDefault);




  
  
  hipMalloc((void **) &edgeXLOW_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &edgeXHIGH_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &edgeYLOW_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &edgeYHIGH_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &edgeZLOW_d, nhalodataZ*sizeof(double));
  hipMalloc((void **) &edgeZHIGH_d, nhalodataZ*sizeof(double));
  
  hipMalloc((void **) &haloXLOW_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &haloXHIGH_d, nhalodataX*sizeof(double));
  hipMalloc((void **) &haloYLOW_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &haloYHIGH_d, nhalodataY*sizeof(double));
  hipMalloc((void **) &haloZLOW_d, nhalodataZ*sizeof(double));
  hipMalloc((void **) &haloZHIGH_d, nhalodataZ*sizeof(double));

  hipMalloc((void **) &mask_d, nsites*sizeof(char));
  hipMalloc((void **) &packedindex_d, nsites*sizeof(int));




  //   checkCUDAError("allocate_memory_on_gpu");

}


/* Free memory on accelerator */
static void free_comms_memory_on_gpu()
{



  hipHostFree(packedindex);
  hipHostFree(mask_);
  hipHostFree(mask_with_neighbours);

  hipHostFree(edgeXLOW);
  hipHostFree(edgeXHIGH);
  hipHostFree(edgeYLOW);
  hipHostFree(edgeYHIGH);
  hipHostFree(edgeZLOW);
  hipHostFree(edgeZHIGH);

  hipHostFree(haloXLOW);
  hipHostFree(haloXHIGH);
  hipHostFree(haloYLOW);
  hipHostFree(haloYHIGH);
  hipHostFree(haloZLOW);
  hipHostFree(haloZHIGH);

  /* free memory on accelerator */

  hipFree(mask_d);
  hipFree(packedindex_d);

  hipFree(edgeXLOW_d);
  hipFree(edgeXHIGH_d);
  hipFree(edgeYLOW_d);
  hipFree(edgeYHIGH_d);
  hipFree(edgeZLOW_d);
  hipFree(edgeZHIGH_d);

  hipFree(haloXLOW_d);
  hipFree(haloXHIGH_d);
  hipFree(haloYLOW_d);
  hipFree(haloYHIGH_d);
  hipFree(haloZLOW_d);
  hipFree(haloZHIGH_d);

}







void fill_mask_with_neighbours(char *mask)
{

  int i, ib[3], p;

  for (i=0; i<nsites; i++)
    mask_with_neighbours[i]=0;


  for (i=0; i<nsites; i++){
    if(mask[i]){
      mask_with_neighbours[i]=1;
      coords_index_to_ijk(i, ib);
      /* if not a halo */
      int halo = (ib[X] < 1 || ib[Y] < 1 || ib[Z] < 1 ||
		  ib[X] > N[X] || ib[Y] > N[Y] || ib[Z] > N[Z]);
      
      if (!halo){
	
	for (p=1; p<NVEL; p++){
	  int indexn = coords_index(ib[X] + cv[p][X], ib[Y] + cv[p][Y],
				    ib[Z] + cv[p][Z]);
	  mask_with_neighbours[indexn]=1;
	}
      }
    }
    
  }
  
  

}




void put_field_partial_on_gpu(int nfields1, int nfields2, int include_neighbours,double *data_d, void (* access_function)(const int, double *)){

  char *mask;
  int i;
  int index;
  double field_tmp[50];
  
  if(include_neighbours){
    fill_mask_with_neighbours(mask_);
    mask=mask_with_neighbours;
  }
  else{
    mask=mask_;
  }



  int packedsize=0;
  for (index=0; index<nsites; index++){
    if(mask[index]) packedsize++;
  }


  int j=0;
  for (index=0; index<nsites; index++){
    
    if(mask[index]){
 
      access_function(index,field_tmp);
      
      for (i=0;i<(nfields1*nfields2);i++)
	{
	  ftmp[i*packedsize+j]=field_tmp[i];
	}
      
      packedindex[index]=j;
      j++;

    }

  }

  hipMemcpy(ftmp_d, ftmp, packedsize*nfields1*nfields2*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(mask_d, mask, nsites*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(packedindex_d, packedindex, nsites*sizeof(int), hipMemcpyHostToDevice);

  /* run the GPU kernel */

  int nblocks=(Nall[X]*Nall[Y]*Nall[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
  copy_field_partial_gpu_d<<<nblocks,DEFAULT_TPB>>>(nfields1*nfields2, nhalo, N_d,
  						data_d, ftmp_d, mask_d,
  						packedindex_d, packedsize, 1);
  hipDeviceSynchronize();
  checkCUDAError("put_partial_field_on_gpu");

}


/* copy part of velocity_ from accelerator to host, using mask structure */
void get_field_partial_from_gpu(int nfields1, int nfields2, int include_neighbours,double *data_d, void (* access_function)(const int, double *))
{


  char *mask;
  int i;
  int index;
  double field_tmp[50];

  if(include_neighbours){
    fill_mask_with_neighbours(mask_);
    mask=mask_with_neighbours;
  }
  else{
    mask=mask_;
  }

  int j=0;
  for (i=0; i<nsites; i++){
    if(mask[i]){
      packedindex[i]=j;
      j++;
    }
    
  }

  int packedsize=j;

  hipMemcpy(mask_d, mask, nsites*sizeof(char), hipMemcpyHostToDevice);
  hipMemcpy(packedindex_d, packedindex, nsites*sizeof(int), hipMemcpyHostToDevice);

  int nblocks=(Nall[X]*Nall[Y]*Nall[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
 copy_field_partial_gpu_d<<<nblocks,DEFAULT_TPB>>>(nfields1*nfields2, nhalo, N_d,
  						ftmp_d, data_d, mask_d,
  						packedindex_d, packedsize, 0);
  hipDeviceSynchronize();

  hipMemcpy(ftmp, ftmp_d, packedsize*nfields1*nfields2*sizeof(double), hipMemcpyDeviceToHost); 

  j=0;
  for (index=0; index<nsites; index++){
    
    if(mask[index]){
 
      for (i=0;i<nfields1*nfields2;i++)
	{
	  field_tmp[i]=ftmp[i*packedsize+j];
	}
      access_function(index,field_tmp);       
      j++;

    }

  }



  /* run the GPU kernel */

  checkCUDAError("get_field_partial_from_gpu");

}


__global__ static void copy_field_partial_gpu_d(int nPerSite, int nhalo, int N[3],
					    double* f_out, double* f_in, char *mask_d, int *packedindex_d, int packedsize, int inpack) {

  int threadIndex, nsite, Nall[3];
  int i;


  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;

  nsite = Nall[X]*Nall[Y]*Nall[Z];


  /* CUDA thread index */
  threadIndex = blockIdx.x*blockDim.x+threadIdx.x;

  //Avoid going beyond problem domain
  if ((threadIndex < Nall[X]*Nall[Y]*Nall[Z]) && mask_d[threadIndex])
    {

      for (i=0;i<nPerSite;i++)
	{
	    
	  if (inpack)
	    f_out[i*nsite+threadIndex]
	    =f_in[i*packedsize+packedindex_d[threadIndex]];
	  else
	   f_out[i*packedsize+packedindex_d[threadIndex]]
	      =f_in[i*nsite+threadIndex];
	  
	}
    }


  return;
}



void halo_gpu(int nfields1, int nfields2, int packablefield1, double * data_d)
{


  int pack_field1=packablefield1*reduced_halo;
  int nfields1packed;
  if (packablefield1){
    /* calculate number of velocity components when packed */
    int p;
    nfields1packed=0;
    for (p=0; p<NVEL; p++)
      {
	if (cv[p][0] == 1 || !reduced_halo) nfields1packed++; 
      }
  }
  else{
    nfields1packed=nfields1;
  }


  int NedgeX[3], NedgeY[3], NedgeZ[3];

  int ii,jj,kk,m,index_source,index_target;

  int nblocks;

#define OVERLAP

  const int tagf = 903;
  const int tagb = 904;
  
  MPI_Request request[4];
  MPI_Status status[4];
  MPI_Comm comm = cart_comm();


  /* the sizes of the packed structures */
  NedgeX[X]=nhalo;
  NedgeX[Y]=N[Y];
  NedgeX[Z]=N[Z];

  NedgeY[X]=Nall[X];
  NedgeY[Y]=nhalo;
  NedgeY[Z]=N[Z];

  NedgeZ[X]=Nall[X];
  NedgeZ[Y]=Nall[Y];
  NedgeZ[Z]=nhalo;

  int npackedsiteX=NedgeX[X]*NedgeX[Y]*NedgeX[Z];
  int npackedsiteY=NedgeY[X]*NedgeY[Y]*NedgeY[Z];
  int npackedsiteZ=NedgeZ[X]*NedgeZ[Y]*NedgeZ[Z];


  /* the below code is structured to overlap packing, CPU-GPU comms and MPI
   as and where possible */

 /* pack X edges on accelerator */
 nblocks=(nhalo*N[Y]*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
 pack_edge_gpu_d<<<nblocks,DEFAULT_TPB,0,streamX>>>(nfields1,nfields2,nhalo,
						pack_field1, N_d,edgeXLOW_d,
						     edgeXHIGH_d,data_d,X);


 /* pack Y edges on accelerator */
  nblocks=(Nall[X]*nhalo*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
  pack_edge_gpu_d<<<nblocks,DEFAULT_TPB,0,streamY>>>(nfields1,nfields2,nhalo,
						pack_field1, N_d,edgeYLOW_d,
						     edgeYHIGH_d,data_d,Y);

 /* pack Z edges on accelerator */
    nblocks=(Nall[X]*Nall[Y]*nhalo+DEFAULT_TPB-1)/DEFAULT_TPB;
  pack_edge_gpu_d<<<nblocks,DEFAULT_TPB,0,streamZ>>>(nfields1,nfields2,nhalo,
  						pack_field1, N_d,edgeZLOW_d,
						     edgeZHIGH_d,data_d,Z);


  /* get X low edges */
  hipMemcpyAsync(edgeXLOW, edgeXLOW_d, nhalodataX*sizeof(double),
		  hipMemcpyDeviceToHost,streamX);
 /* get X high edges */
  hipMemcpyAsync(edgeXHIGH, edgeXHIGH_d, nhalodataX*sizeof(double),
		  hipMemcpyDeviceToHost,streamX);


#ifndef OVERLAP
  hipStreamSynchronize(streamX);
#endif

 /* get Y low edges */
  hipMemcpyAsync(edgeYLOW, edgeYLOW_d, nhalodataY*sizeof(double),
		  hipMemcpyDeviceToHost,streamY);
 /* get Y high edges */
  hipMemcpyAsync(edgeYHIGH, edgeYHIGH_d, nhalodataY*sizeof(double),
		  hipMemcpyDeviceToHost,streamY);


#ifndef OVERLAP
  hipStreamSynchronize(streamY);
#endif

  /* get Z low edges */
  hipMemcpyAsync(edgeZLOW, edgeZLOW_d, nhalodataZ*sizeof(double),
		  hipMemcpyDeviceToHost,streamZ);
  /* get Z high edges */
  hipMemcpyAsync(edgeZHIGH, edgeZHIGH_d, nhalodataZ*sizeof(double),
		  hipMemcpyDeviceToHost,streamZ);


#ifndef OVERLAP
  hipStreamSynchronize(streamZ);
#endif


 /* wait for X data from accelerator*/
  hipStreamSynchronize(streamX);



   if (cart_size(X) == 1) {
     /* x up */
     memcpy(haloXLOW,edgeXHIGH,nhalodataX*sizeof(double));
     
     /* x down */
     memcpy(haloXHIGH,edgeXLOW,nhalodataX*sizeof(double));
     
      }
  else
    {
      /* initiate transfers */
      MPI_Irecv(haloXLOW, nhalodataX, MPI_DOUBLE,
	      cart_neighb(BACKWARD,X), tagf, comm, &request[0]);
      MPI_Irecv(haloXHIGH, nhalodataX, MPI_DOUBLE,
	      cart_neighb(FORWARD,X), tagb, comm, &request[1]);
      MPI_Isend(edgeXHIGH, nhalodataX, MPI_DOUBLE,
	      cart_neighb(FORWARD,X), tagf, comm, &request[2]);
      MPI_Isend(edgeXLOW,  nhalodataX, MPI_DOUBLE,
	      cart_neighb(BACKWARD,X), tagb, comm, &request[3]);
     }


 /* wait for X halo swaps to finish */
   if (cart_size(X) > 1)       MPI_Waitall(4, request, status);


 /* put X halos back on device, and unpack */
  hipMemcpyAsync(haloXLOW_d, haloXLOW, nhalodataX*sizeof(double),
		  hipMemcpyHostToDevice,streamX);
  hipMemcpyAsync(haloXHIGH_d, haloXHIGH, nhalodataX*sizeof(double),
		  hipMemcpyHostToDevice,streamX);
  nblocks=(nhalo*N[Y]*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
     unpack_halo_gpu_d<<<nblocks,DEFAULT_TPB,0,streamX>>>(nfields1,nfields2,nhalo,
  						  pack_field1, N_d,data_d,haloXLOW_d,
							      haloXHIGH_d,X);



#ifndef OVERLAP
  hipStreamSynchronize(streamX);
#endif

  /* wait for Y data from accelerator*/
  hipStreamSynchronize(streamY);


  /* fill in corners of Y edge data  */

  for (m=0;m<(nfields1packed*nfields2);m++)
    {
      
      
      for (ii = 0; ii < nhalo; ii++) {
	for (jj = 0; jj < nhalo; jj++) {
	  for (kk = 0; kk < N[Z]; kk++) {
	    
	    
	    
	    /* xlow part of ylow */
	    index_source = get_linear_index(ii,jj,kk,NedgeX);
	    index_target = get_linear_index(ii,jj,kk,NedgeY);
	    
	    edgeYLOW[npackedsiteY*m+index_target] =
	      haloXLOW[npackedsiteX*m+index_source];
	    
	    /* xlow part of yhigh */
	    index_source = get_linear_index(ii,NedgeX[Y]-nhalo+jj,kk,NedgeX);
	    index_target = get_linear_index(ii,jj,kk,NedgeY);
	    
	    edgeYHIGH[npackedsiteY*m+index_target] =
	      haloXLOW[npackedsiteX*m+index_source];
	    
	    
	    /* get high X data */
	    
	    /* xhigh part of ylow */
	    index_source = get_linear_index(ii,jj,kk,NedgeX);
	    index_target = get_linear_index(NedgeY[X]-nhalo+ii,jj,kk,NedgeY);
	    
	    edgeYLOW[npackedsiteY*m+index_target] =
	      haloXHIGH[npackedsiteX*m+index_source];
	    
	    /* xhigh part of yhigh */
	    
	    index_source = get_linear_index(ii,NedgeX[Y]-nhalo+jj,kk,NedgeX);			index_target = get_linear_index(NedgeY[X]-nhalo+ii,jj,kk,NedgeY);
	    
	    edgeYHIGH[npackedsiteY*m+index_target] =
	      haloXHIGH[npackedsiteX*m+index_source];
	    
	    
	    
	  }
	}
	
      }
    }
  


  /* The y-direction (XZ plane) */
   if (cart_size(Y) == 1) {
  /* y up */
  memcpy(haloYLOW,edgeYHIGH,nhalodataY*sizeof(double));
  
  /* y down */
  memcpy(haloYHIGH,edgeYLOW,nhalodataY*sizeof(double));
  
      }
  else
    {
      /* initiate transfers */
      MPI_Irecv(haloYLOW, nhalodataY, MPI_DOUBLE,
	      cart_neighb(BACKWARD,Y), tagf, comm, &request[0]);
      MPI_Irecv(haloYHIGH, nhalodataY, MPI_DOUBLE,
	      cart_neighb(FORWARD,Y), tagb, comm, &request[1]);
      MPI_Isend(edgeYHIGH, nhalodataY, MPI_DOUBLE,
	      cart_neighb(FORWARD,Y), tagf, comm, &request[2]);
      MPI_Isend(edgeYLOW,  nhalodataY, MPI_DOUBLE,
	      cart_neighb(BACKWARD,Y), tagb, comm, &request[3]);
    }


 /* wait for Y halo swaps to finish */
    if (cart_size(Y) > 1)       MPI_Waitall(4, request, status);

 /* put Y halos back on device, and unpack */
  hipMemcpyAsync(haloYLOW_d, haloYLOW, nhalodataY*sizeof(double),
		  hipMemcpyHostToDevice,streamY);
  hipMemcpyAsync(haloYHIGH_d, haloYHIGH, nhalodataY*sizeof(double),
		  hipMemcpyHostToDevice,streamY);
  nblocks=(Nall[X]*nhalo*N[Z]+DEFAULT_TPB-1)/DEFAULT_TPB;
    unpack_halo_gpu_d<<<nblocks,DEFAULT_TPB,0,streamY>>>(nfields1,nfields2,nhalo,
  						  pack_field1, N_d,data_d,haloYLOW_d,
							 haloYHIGH_d,Y);



#ifndef OVERLAP
  hipStreamSynchronize(streamY);
#endif

 
  /* wait for Z data from accelerator*/
  hipStreamSynchronize(streamZ);

  /* fill in corners of Z edge data: from Xhalo  */
    
  for (m=0;m<(nfields1packed*nfields2);m++)
    {
      
      for (ii = 0; ii < nhalo; ii++) {
	for (jj = 0; jj < N[Y]; jj++) {
	  for (kk = 0; kk < nhalo; kk++) {
	    
	    
	    
	    /* xlow part of zlow */
	    index_source = get_linear_index(ii,jj,kk,NedgeX);
	    index_target = get_linear_index(ii,jj+nhalo,kk,NedgeZ);
	    
	    edgeZLOW[npackedsiteZ*m+index_target] =
	      haloXLOW[npackedsiteX*m+index_source];
	    
	    
	    /* xlow part of zhigh */
	    index_source = get_linear_index(ii,jj,NedgeX[Z]-nhalo+kk,NedgeX);
	    index_target = get_linear_index(ii,jj+nhalo,kk,NedgeZ);
	    
	    edgeZHIGH[npackedsiteZ*m+index_target] =
	      haloXLOW[npackedsiteX*m+index_source];
	    
	    
	    
	    /* xhigh part of zlow */
	    index_source = get_linear_index(ii,jj,kk,NedgeX);
	    index_target = get_linear_index(NedgeZ[X]-nhalo+ii,jj+nhalo,kk,
					    NedgeZ);
	    
	    edgeZLOW[npackedsiteZ*m+index_target] =
	      haloXHIGH[npackedsiteX*m+index_source];
	    
	    
	    /* xhigh part of zhigh */
	    index_source = get_linear_index(ii,jj,NedgeX[Z]-nhalo+kk,NedgeX);
	    index_target = get_linear_index(NedgeZ[X]-nhalo+ii,jj+nhalo,kk,
					    NedgeZ);
	    
	    edgeZHIGH[npackedsiteZ*m+index_target] =
	      haloXHIGH[npackedsiteX*m+index_source];
	    
	    
	  }
	}
	
	
      }
    }
  
  /* fill in corners of Z edge data: from Yhalo  */
  
  
  
  for (m=0;m<(nfields1packed*nfields2);m++)
    {
      
      
      
      for (ii = 0; ii < Nall[X]; ii++) {
	for (jj = 0; jj < nhalo; jj++) {
	  for (kk = 0; kk < nhalo; kk++) {
	    
	    
	    
	    /* ylow part of zlow */
	    index_source = get_linear_index(ii,jj,kk,NedgeY);
	    index_target = get_linear_index(ii,jj,kk,NedgeZ);
	    
	    edgeZLOW[npackedsiteZ*m+index_target] =
	      haloYLOW[npackedsiteY*m+index_source];
	    
	    
	    /* ylow part of zhigh */
	    index_source = get_linear_index(ii,jj,NedgeY[Z]-nhalo+kk,NedgeY);
	    index_target = get_linear_index(ii,jj,kk,NedgeZ);
	    
	    edgeZHIGH[npackedsiteZ*m+index_target] =
	      haloYLOW[npackedsiteY*m+index_source];
	    
	    
	    
	    /* yhigh part of zlow */
	    index_source = get_linear_index(ii,jj,kk,NedgeY);
	    index_target = get_linear_index(ii,NedgeZ[Y]-nhalo+jj,kk,NedgeZ);
	    
	    edgeZLOW[npackedsiteZ*m+index_target] =
	      haloYHIGH[npackedsiteY*m+index_source];
	    
	    
	    /* yhigh part of zhigh */
	    
	    index_source = get_linear_index(ii,jj,NedgeY[Z]-nhalo+kk,NedgeY);
	    index_target = get_linear_index(ii,NedgeZ[Y]-nhalo+jj,kk,NedgeZ);
	    
	    edgeZHIGH[npackedsiteZ*m+index_target] =
	      haloYHIGH[npackedsiteY*m+index_source];
	    
	    
	  }
	}
	
      }
    }
  


  /* The z-direction (xy plane) */
   if (cart_size(Z) == 1) {
  /* z up */
  memcpy(haloZLOW,edgeZHIGH,nhalodataZ*sizeof(double));

  /* z down */
  memcpy(haloZHIGH,edgeZLOW,nhalodataZ*sizeof(double));
      }
  else
    {
      MPI_Irecv(haloZLOW, nhalodataZ, MPI_DOUBLE,
	      cart_neighb(BACKWARD,Z), tagf, comm, &request[0]);
      MPI_Irecv(haloZHIGH, nhalodataZ, MPI_DOUBLE,
	      cart_neighb(FORWARD,Z), tagb, comm, &request[1]);
      MPI_Isend(edgeZHIGH, nhalodataZ, MPI_DOUBLE,
	      cart_neighb(FORWARD,Z), tagf, comm, &request[2]);
      MPI_Isend(edgeZLOW,  nhalodataZ, MPI_DOUBLE,
	      cart_neighb(BACKWARD,Z), tagb, comm, &request[3]);
      MPI_Waitall(4, request, status);

    }

 /* put Z halos back on device, and unpack */
  hipMemcpyAsync(haloZLOW_d, haloZLOW, nhalodataZ*sizeof(double),
		  hipMemcpyHostToDevice,streamZ);
  hipMemcpyAsync(haloZHIGH_d, haloZHIGH, nhalodataZ*sizeof(double),
		  hipMemcpyHostToDevice,streamZ);

    nblocks=(Nall[X]*Nall[Y]*nhalo+DEFAULT_TPB-1)/DEFAULT_TPB;
     unpack_halo_gpu_d<<<nblocks,DEFAULT_TPB,0,streamZ>>>(nfields1,nfields2,nhalo,
							  pack_field1, N_d,data_d,haloZLOW_d,
							  haloZHIGH_d,Z);


  /* wait for all streams to complete */
  hipStreamSynchronize(streamX);
  hipStreamSynchronize(streamY);
  hipStreamSynchronize(streamZ);
  

}



/* pack edges on the accelerator */
__global__ static void pack_edge_gpu_d(int nfields1, int nfields2,
				       int nhalo, int pack_field1,
					  int N[3],
					 double* edgeLOW_d,
				       double* edgeHIGH_d, 
				       double* f_d, int dirn)
{


  /* variables to determine how vel packing is done from cv array */
  int ud=-1; /* up or down */
  int pn=-1; /* positive 1 or negative 1 factor */


  int Nall[3];
  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;
  int nsite = Nall[X]*Nall[Y]*Nall[Z];
 
  int Nedge[3];

  if (dirn == X){
    Nedge[X]=nhalo;
    Nedge[Y]=N[Y];
    Nedge[Z]=N[Z];
  }
  else if (dirn == Y){
    Nedge[X]=Nall[X];
    Nedge[Y]=nhalo;
    Nedge[Z]=N[Z];
  }
  else if (dirn == Z){
    Nedge[X]=Nall[X];
    Nedge[Y]=Nall[Y];
    Nedge[Z]=nhalo;
  }


  int p,m, index,ii,jj,kk;
  int packed_index,packedp;
 
 
  int npackedsite = Nedge[X]*Nedge[Y]*Nedge[Z];
  
  int threadIndex = blockIdx.x*blockDim.x+threadIdx.x;
  
  if (threadIndex < npackedsite)
    {
      
      if (threadIndex==0){
	//for (p=0;p<NVEL;p++)
	//for (m=0;m<3;m++)
	  //p=0;m=1;
 	  //printf("TT3 %d %d %d %d %d\n",p,m,cv_cd[p][m],cv_d[p][m], cv_ptr[p*3+m]);

      }      
      packed_index = threadIndex;
      
      get_coords_from_index_gpu_d(&ii,&jj,&kk,threadIndex,Nedge);
      
      /* LOW EDGE */
      if (dirn == X){
	index = get_linear_index_gpu_d(ii+nhalo,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn == Y){
	index = get_linear_index_gpu_d(ii,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn == Z){
	index = get_linear_index_gpu_d(ii,jj,kk+nhalo,Nall);
      }
 
      /* copy data to packed structure */
      packedp=0;
      for (p = 0; p < nfields1; p++) {
	if (cv_cd[p][dirn] == ud || !pack_field1)
	  {
	    for (m = 0; m < nfields2; m++) {
	      edgeLOW_d[nfields2*npackedsite*packedp+m*npackedsite
	      	  +packed_index]
	      	= f_d[nfields2*nsite*p+nsite*m+index];
	    }
	    packedp++;
	  }
      }
      
  
      /* HIGH EDGE */
      if (dirn == X){
	index = get_linear_index_gpu_d(Nall[X]-2*nhalo+ii,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn == Y){
        index = get_linear_index_gpu_d(ii,Nall[Y]-2*nhalo+jj,kk+nhalo,Nall);
      }
      else if (dirn == Z){
	index = get_linear_index_gpu_d(ii,jj,Nall[Z]-2*nhalo+kk,Nall);
      }

      /* copy data to packed structure */
      packedp=0;
      for (p = 0; p < nfields1; p++) {
	if (cv_cd[p][dirn] == ud*pn || !pack_field1 )
	  {
	    for (m = 0; m < nfields2; m++) {
	      
	      edgeHIGH_d[nfields2*npackedsite*packedp+m*npackedsite
			   +packed_index]
		= f_d[nfields2*nsite*p+nsite*m+index];
	      
	    }
	    packedp++;
	  }
      }
    }
  
  
}



/* unpack halos on the accelerator */
__global__ static void unpack_halo_gpu_d(int nfields1, int nfields2,
					 int nhalo, int pack_field1,
					   int N[3],
					   double* f_d, double* haloLOW_d,
					 double* haloHIGH_d, int dirn)
{


  int dirn_save=dirn;

  /* variables to determine how vel packing is done from cv array */
  int ud=1; /* up or down */
  int pn=-1; /* positive 1 or negative 1 factor */

  int Nall[3];
  Nall[X]=N[X]+2*nhalo;
  Nall[Y]=N[Y]+2*nhalo;
  Nall[Z]=N[Z]+2*nhalo;
  int nsite = Nall[X]*Nall[Y]*Nall[Z];
 
  int Nedge[3];

  if (dirn == X){
    Nedge[X]=nhalo;
    Nedge[Y]=N[Y];
    Nedge[Z]=N[Z];
  }
  else if (dirn == Y){
    Nedge[X]=Nall[X];
    Nedge[Y]=nhalo;
    Nedge[Z]=N[Z];
  }
  else if (dirn == Z){
    Nedge[X]=Nall[X];
    Nedge[Y]=Nall[Y];
    Nedge[Z]=nhalo;
  }


  int p,m, index,ii,jj,kk;
  int packed_index, packedp;
 
  int npackedsite = Nedge[X]*Nedge[Y]*Nedge[Z];
  
  int threadIndex = blockIdx.x*blockDim.x+threadIdx.x;
  
  if (threadIndex < npackedsite)
    {
      
      packed_index = threadIndex;
      
      get_coords_from_index_gpu_d(&ii,&jj,&kk,threadIndex,Nedge);

      /* LOW HALO */
      if (dirn == X){
	index = get_linear_index_gpu_d(ii,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn == Y){
	index = get_linear_index_gpu_d(ii,jj,kk+nhalo,Nall);
      }
      else if (dirn == Z){
	index = get_linear_index_gpu_d(ii,jj,kk,Nall);
      }

      if (dirn==Y || dirn==Z){
	/* correct for diagonal data that was packed by X packing subroutine */
	if (ii < nhalo) 
	  { 
	    dirn = X;
	    ud=1;
	    pn=1;
	  }
	if (ii >= Nall[X]-nhalo)
	  { 
	    dirn = X;
	    ud=-1;
	    pn=1;
	  }
      }

      if (dirn==Z){
	/* correct for diagonal data that was packed by Y packing subroutine */
	if (jj < nhalo)
	  { 
	    dirn = Y;
	    ud=1;
	    pn=1;
	  }
	
	if (jj >= Nall[Y]-nhalo)
	  { 
	    dirn = Y;
	    ud=-1;
	    pn=1;
	  }
      }

      
      
      /* copy packed structure data to original array */
      packedp=0;
      for (p = 0; p < nfields1; p++) {
	if (cv_cd[p][dirn] == ud || !pack_field1)
	  {
	    for (m = 0; m < nfields2; m++) {
	  
	      f_d[nfields2*nsite*p+nsite*m+index] =
	      haloLOW_d[nfields2*npackedsite*packedp+m*npackedsite
	      	    +packed_index];

	    }
	    packedp++;
	  }
      }
           
      /* HIGH HALO */
      if (dirn_save == X){
	index = get_linear_index_gpu_d(Nall[X]-nhalo+ii,jj+nhalo,kk+nhalo,Nall);
      }
      else if (dirn_save == Y){
	index = get_linear_index_gpu_d(ii,Nall[Y]-nhalo+jj,kk+nhalo,Nall);	
      }
      else if (dirn_save == Z){
	index = get_linear_index_gpu_d(ii,jj,Nall[Z]-nhalo+kk,Nall);
      }

      /* copy packed structure data to original array */
      packedp=0;
      for (p = 0; p < nfields1; p++) {
	if (cv_cd[p][dirn] == ud*pn || !pack_field1 )
	  {
	    for (m = 0; m < nfields2; m++) {
	      
	      f_d[nfields2*nsite*p+nsite*m+index] =
	      haloHIGH_d[nfields2*npackedsite*packedp+m*npackedsite
	           +packed_index];
	      
	    }
	    packedp++;
	  }
      }
    }
  
  
}

/* get 3d coordinates from the index on the accelerator */
__device__ static void get_coords_from_index_gpu_d(int *ii,int *jj,int *kk,int index,int N[3])

{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;
  
  *ii = index/xfac;
  *jj = ((index-xfac*(*ii))/yfac);
  *kk = (index-(*ii)*xfac-(*jj)*yfac);

  return;

}

/* get linear index from 3d coordinates (device) */
 __device__ static int get_linear_index_gpu_d(int ii,int jj,int kk,int N[3])

{
  
  int yfac = N[Z];
  int xfac = N[Y]*yfac;

  return ii*xfac + jj*yfac + kk;

}

