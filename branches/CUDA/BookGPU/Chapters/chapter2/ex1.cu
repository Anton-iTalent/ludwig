#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <assert.h>
#include "cutil_inline.h"

const int nbThreadsPerBloc=256;

__global__ 
void addition(int size, int *d_C, int *d_A, int *d_B) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid<size) {
		d_C[tid]=d_A[tid]+d_B[tid];
	}
}


int main( int argc, char** argv) 
{
	if(argc!=2) { 
		printf("usage: ex1 nb_components\n");
		exit(0);
	}

	int size=atoi(argv[1]);
	int i;
	int *h_arrayA=(int*)malloc(size*sizeof(int));
	int *h_arrayB=(int*)malloc(size*sizeof(int));
	int *h_arrayC=(int*)malloc(size*sizeof(int));
	int *h_arrayCgpu=(int*)malloc(size*sizeof(int));
	int *d_arrayA, *d_arrayB, *d_arrayC;

	hipMalloc((void**)&d_arrayA,size*sizeof(int));
	hipMalloc((void**)&d_arrayB,size*sizeof(int));
	hipMalloc((void**)&d_arrayC,size*sizeof(int));

	for(i=0;i<size;i++) {
		h_arrayA[i]=i;
		h_arrayB[i]=2*i;
	}

	unsigned int timer_cpu = 0;
	cutilCheckError(cutCreateTimer(&timer_cpu));
  cutilCheckError(cutStartTimer(timer_cpu));
	for(i=0;i<size;i++) {
		h_arrayC[i]=h_arrayA[i]+h_arrayB[i];
	}
	cutilCheckError(cutStopTimer(timer_cpu));
	printf("CPU processing time : %f (ms) \n", cutGetTimerValue(timer_cpu));
	cutDeleteTimer(timer_cpu);

	unsigned int timer_gpu = 0;
	cutilCheckError(cutCreateTimer(&timer_gpu));
  cutilCheckError(cutStartTimer(timer_gpu));
	hipMemcpy(d_arrayA,h_arrayA, size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_arrayB,h_arrayB, size * sizeof(int), hipMemcpyHostToDevice);
	
	int nbBlocs=(size+nbThreadsPerBloc-1)/nbThreadsPerBloc;
	addition<<<nbBlocs,nbThreadsPerBloc>>>(size,d_arrayC,d_arrayA,d_arrayB);
	hipMemcpy(h_arrayCgpu,d_arrayC, size * sizeof(int), hipMemcpyDeviceToHost);

	cutilCheckError(cutStopTimer(timer_gpu));
	printf("GPU processing time : %f (ms) \n", cutGetTimerValue(timer_gpu));
	cutDeleteTimer(timer_gpu);

	for(i=0;i<size;i++)
		assert(h_arrayC[i]==h_arrayCgpu[i]);

	hipFree(d_arrayA);
	hipFree(d_arrayB);
	hipFree(d_arrayC);
	free(h_arrayA);
	free(h_arrayB);
	free(h_arrayC);
	return 0;
}
